#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <general_buffer2.hpp>
#include <optimizers/nesterov_optimizer.hpp>
#include <utils.cuh>
#include <utils.hpp>

namespace HugeCTR {

namespace {

template <typename T>
__global__ void nesterov_update_kernel(int len, float* weight, T* accum, const T* wgrad, float lr,
                                       float mu, float scaler) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) {
    float accum_old = TypeConvertFunc<float, T>::convert(accum[i]);
    float accum_new = mu * accum_old - lr * TypeConvertFunc<float, T>::convert(wgrad[i]) / scaler;
    accum[i] = TypeConvertFunc<T, float>::convert(accum_new);
    weight[i] += (-mu * accum_old + (1.f + mu) * accum_new);
  }
}

}  // namespace

template <typename T>
NesterovOptimizer<T>::NesterovOptimizer(const Tensor2<float>& weight_main, const Tensor2<T>& wgrad,
                                        const std::shared_ptr<BufferBlock2<T>>& opt_buf,
                                        const std::shared_ptr<GPUResource>& gpu_resource,
                                        float learning_rate, float momentum_factor, float scaler)
    : Optimizer(weight_main, gpu_resource, learning_rate, scaler),
      wgrad_(wgrad),
      mu_(momentum_factor) {
  if (weight_main_.get_num_elements() != wgrad_.get_num_elements()) {
    HCTR_OWN_THROW(Error_t::WrongInput, "weight->get_num_elements() != wgrad->get_num_elements()");
  }
  opt_buf->reserve({weight_main.get_num_elements()}, &accum_);
}

template <typename T>
void NesterovOptimizer<T>::initialize() {
  HCTR_LIB_THROW(hipMemsetAsync(accum_.get_ptr(), 0, accum_.get_size_in_bytes(),
                                 gpu_resource_->get_stream()));
}

template <typename T>
void NesterovOptimizer<T>::update() {
  CudaDeviceContext context(get_device_id());

  const size_t len = weight_main_.get_num_elements();
  constexpr size_t block_dim = 256;
  const size_t grid_dim = (len - 1) / block_dim + 1;

  float* weight = weight_main_.get_ptr();
  T* accum = accum_.get_ptr();
  T* wgrad = wgrad_.get_ptr();
  nesterov_update_kernel<<<grid_dim, block_dim, 0, gpu_resource_->get_stream()>>>(
      len, weight, accum, wgrad, lr_, mu_, scaler_);

#ifndef NDEBUG
  HCTR_LIB_THROW(hipDeviceSynchronize());
  HCTR_LIB_THROW(hipGetLastError());
#endif
}

template class NesterovOptimizer<float>;
template class NesterovOptimizer<__half>;

}  // namespace HugeCTR
