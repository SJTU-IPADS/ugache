#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <general_buffer2.hpp>
#include <optimizers/adam_optimizer.hpp>
#include <utils.cuh>
#include <utils.hpp>

namespace HugeCTR {

namespace {

template <typename T>
__global__ void adam_update_kernel(int len, float* weight, T* m, T* v, const T* wgrad,
                                   float alpha_t, float beta1, float beta2, float epsilon,
                                   float scaler) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) {
    float gi = TypeConvertFunc<float, T>::convert(wgrad[i]) / scaler;
    float mi = beta1 * TypeConvertFunc<float, T>::convert(m[i]) + (1.f - beta1) * gi;
    float vi = beta2 * TypeConvertFunc<float, T>::convert(v[i]) + (1.f - beta2) * gi * gi;
    m[i] = TypeConvertFunc<T, float>::convert(mi);
    v[i] = TypeConvertFunc<T, float>::convert(vi);
    weight[i] -= alpha_t * mi / (sqrt(vi) + epsilon);
  }
}

}  // namespace

template <typename T>
AdamOptimizer<T>::AdamOptimizer(const Tensor2<float>& weight_main, const Tensor2<T>& wgrad,
                                const std::shared_ptr<BufferBlock2<T>>& opt_buf,
                                const std::shared_ptr<GPUResource>& gpu_resource,
                                float learning_rate, float beta1, float beta2, float epsilon,
                                float scaler)
    : Optimizer(weight_main, gpu_resource, learning_rate, scaler),
      wgrad_(wgrad),
      t_(0),
      beta1_(beta1),
      beta2_(beta2),
      epsilon_(epsilon) {
  if (weight_main_.get_num_elements() != wgrad_.get_num_elements()) {
    HCTR_OWN_THROW(Error_t::WrongInput, "weight->get_num_elements() != wgrad->get_num_elements()");
  }
  opt_buf->reserve({weight_main.get_num_elements()}, &m_);
  opt_buf->reserve({weight_main.get_num_elements()}, &v_);
}

template <typename T>
void AdamOptimizer<T>::initialize() {
  HCTR_LIB_THROW(
      hipMemsetAsync(m_.get_ptr(), 0, m_.get_size_in_bytes(), gpu_resource_->get_stream()));
  HCTR_LIB_THROW(
      hipMemsetAsync(v_.get_ptr(), 0, v_.get_size_in_bytes(), gpu_resource_->get_stream()));
}

template <typename T>
void AdamOptimizer<T>::update() {
  CudaDeviceContext context(get_device_id());

  const size_t len = weight_main_.get_num_elements();
  constexpr size_t block_dim = 256;
  const size_t grid_dim = (len - 1) / block_dim + 1;

  ++t_;
  const float alpha_t = lr_ * sqrt(1 - pow(beta2_, t_)) / (1 - pow(beta1_, t_));

  float* weight = weight_main_.get_ptr();

  T* m = m_.get_ptr();
  T* v = v_.get_ptr();
  const T* wgrad = wgrad_.get_ptr();
  adam_update_kernel<<<grid_dim, block_dim, 0, gpu_resource_->get_stream()>>>(
      len, weight, m, v, wgrad, alpha_t, beta1_, beta2_, epsilon_, scaler_);
#ifndef NDEBUG
  HCTR_LIB_THROW(hipDeviceSynchronize());
  HCTR_LIB_THROW(hipGetLastError());
#endif
}

template class AdamOptimizer<float>;
template class AdamOptimizer<__half>;

}  // namespace HugeCTR
