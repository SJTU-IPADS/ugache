/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include <functional>
#include <layers/matrix_multiply_layer.hpp>
#include <utils.cuh>
#include <utils.hpp>

#ifndef NDEBUG
#include <iostream>
#endif

namespace HugeCTR {

template <typename T>
MatrixMultiplyLayer<T>::MatrixMultiplyLayer(
    const Tensors2<T>& in_tensors, Tensor2<T>& out_tensor,
    const std::shared_ptr<GeneralBuffer2<CudaAllocator>>& blobs_buff,
    const std::shared_ptr<GPUResource>& gpu_resource)
    : Layer(gpu_resource) {
  try {
    num_ = in_tensors.size();

    // error input checking
    dims_ = in_tensors[0].get_dimensions().size();
    if (num_ < 2) {
      HCTR_OWN_THROW(Error_t::WrongInput, "MatrixMultiplyLayer needs at least 2 input tensors");
    }
    if (in_tensors[1].get_dimensions().size() != dims_) {
      HCTR_OWN_THROW(Error_t::WrongInput, "All the input tensors must have the same num of dims");
    }
    if (in_tensors[1].get_dimensions()[dims_ - 2] != in_tensors[0].get_dimensions()[dims_ - 1]) {
      HCTR_OWN_THROW(Error_t::WrongInput,
                     "The last two dimension of the input tensors should be m x n, n x k");
    }

    for (size_t i = 0; i < num_; i++) {
      in_tensors_.push_back(in_tensors[i]);
    }

    size_t m = in_tensors[0].get_dimensions()[dims_ - 2];
    size_t k = in_tensors[1].get_dimensions()[dims_ - 1];

    if (dims_ == 2) {
      std::vector<size_t> out_dim = {m, k};
      blobs_buff->reserve(out_dim, &out_tensor);
    } else if (dims_ == 3) {  // dims_ == 3
      if (in_tensors[0].get_dimensions()[0] != in_tensors[1].get_dimensions()[0]) {
        HCTR_OWN_THROW(Error_t::WrongInput, "3D input tensors must have the same batch size");
      }
      size_t b = in_tensors[0].get_dimensions()[0];
      std::vector<size_t> out_dim = {b, m, k};
      blobs_buff->reserve(out_dim, &out_tensor);
    } else if (dims_ == 4) {
      if (in_tensors[0].get_dimensions()[0] != in_tensors[1].get_dimensions()[0]) {
        HCTR_OWN_THROW(Error_t::WrongInput, "4D input tensors must have the same batch size");
      }
      if (in_tensors[0].get_dimensions()[1] != in_tensors[1].get_dimensions()[1]) {
        HCTR_OWN_THROW(Error_t::WrongInput, "4D input tensors must have the same second dim");
      }
      size_t b = in_tensors[0].get_dimensions()[0];
      size_t num_head = in_tensors[0].get_dimensions()[1];
      std::vector<size_t> out_dim = {b, num_head, m, k};
      blobs_buff->reserve(out_dim, &out_tensor);
    }

    out_tensors_.push_back(out_tensor);

    blobs_buff->reserve(in_tensors[0].get_dimensions(), &fprop_inputA_);

  } catch (const std::runtime_error& rt_err) {
    HCTR_LOG_S(ERROR, WORLD) << rt_err.what() << std::endl;
    throw;
  }
}

template <typename T>
void MatrixMultiplyLayer<T>::fprop(bool is_train) {
  CudaDeviceContext context(get_device_id());

  T* in1 = in_tensors_[0].get_ptr();
  T* in2 = in_tensors_[1].get_ptr();
  T* out = out_tensors_[0].get_ptr();

  const auto& in_tensor_dim = in_tensors_[0].get_dimensions();
  const auto& out_tensor_dim = out_tensors_[0].get_dimensions();

  size_t m, n, k, b = 1;

  b = dims_ == 3 ? in_tensor_dim[0] : 1;
  b = dims_ == 4 ? in_tensor_dim[0] * in_tensor_dim[1] : b;
  m = in_tensor_dim[dims_ - 2];
  n = in_tensor_dim[dims_ - 1];
  k = out_tensor_dim[dims_ - 1];
  float alpha = 1.0f, beta = 0.0f;
  hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_32F;

  for (size_t i = 0; i < b; i++) {
    T* cur_in1 = in1 + i * m * n;
    T* cur_in2 = in2 + i * n * k;
    T* cur_out = out + i * m * k;
    HCTR_LIB_THROW(hipblasGemmEx(get_gpu().get_cublas_handle(), HIPBLAS_OP_N, HIPBLAS_OP_N, k, m, n,
                                &alpha, cur_in2, HIP_R_32F, k, cur_in1, HIP_R_32F, n, &beta,
                                cur_out, HIP_R_32F, k, compute_type, HIPBLAS_GEMM_DEFAULT));
  }

  HCTR_LIB_THROW(hipMemcpyAsync((void*)fprop_inputA_.get_ptr(), (void*)in1,
                                 in_tensors_[0].get_size_in_bytes(), hipMemcpyDeviceToDevice,
                                 get_gpu().get_stream()));
#ifndef NDEBUG
  hipDeviceSynchronize();
  HCTR_LIB_THROW(hipGetLastError());
#endif
}

template <typename T>
void MatrixMultiplyLayer<T>::bprop() {
  CudaDeviceContext context(get_device_id());

  T* in1 = in_tensors_[0].get_ptr();
  T* in2 = in_tensors_[1].get_ptr();
  T* out = out_tensors_[0].get_ptr();

  const auto& in_tensor_dim = in_tensors_[0].get_dimensions();
  const auto& out_tensor_dim = out_tensors_[0].get_dimensions();

  size_t m, n, k, b = 1;

  b = dims_ == 3 ? in_tensor_dim[0] : 1;
  b = dims_ == 4 ? in_tensor_dim[0] * in_tensor_dim[1] : b;
  m = in_tensor_dim[dims_ - 2];
  n = in_tensor_dim[dims_ - 1];
  k = out_tensor_dim[dims_ - 1];
  float alpha = 1.0f, beta = 0.0f;
  hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_32F;

  for (size_t i = 0; i < b; i++) {
    T* cur_in1 = in1 + i * m * n;
    T* cur_in2 = in2 + i * n * k;
    T* cur_out = out + i * m * k;
    // gradient respect to A
    HCTR_LIB_THROW(hipblasGemmEx(get_gpu().get_cublas_handle(), HIPBLAS_OP_T, HIPBLAS_OP_N, n, m, k,
                                &alpha, cur_in2, HIP_R_32F, k, cur_out, HIP_R_32F, k, &beta,
                                cur_in1, HIP_R_32F, n, compute_type, HIPBLAS_GEMM_DEFAULT));

    cur_in1 = fprop_inputA_.get_ptr() + i * m * n;
    // gradient respect to B
    HCTR_LIB_THROW(hipblasGemmEx(get_gpu().get_cublas_handle(), HIPBLAS_OP_N, HIPBLAS_OP_T, k, n, m,
                                &alpha, cur_out, HIP_R_32F, k, cur_in1, HIP_R_32F, n, &beta,
                                cur_in2, HIP_R_32F, k, compute_type, HIPBLAS_GEMM_DEFAULT));
  }
#ifndef NDEBUG
  hipDeviceSynchronize();
  HCTR_LIB_THROW(hipGetLastError());
#endif
}

template class MatrixMultiplyLayer<float>;

}  // namespace HugeCTR
