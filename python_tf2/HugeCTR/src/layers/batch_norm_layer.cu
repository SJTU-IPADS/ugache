#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include <functional>
#include <layers/batch_norm_layer.hpp>
#include <string>
#include <utils.hpp>

#ifndef NDEBUG
#include <iostream>
#endif

namespace HugeCTR {

namespace {

template <typename T>
using ToStringType = typename std::conditional<std::is_same<T, __half>::value, float, T>::type;
}

template <typename T>
BatchNormLayer<T>::BatchNormLayer(const std::shared_ptr<BufferBlock2<float>>& weight_buff,
                                  const std::shared_ptr<BufferBlock2<float>>& wgrad_buff,
                                  const std::shared_ptr<GeneralBuffer2<CudaAllocator>>& blob_buff,
                                  const Tensor2<T>& in_tensor, const Tensor2<T>& out_tensor,
                                  const Params& params,
                                  const std::shared_ptr<GPUResource>& gpu_resource,
                                  std::vector<Initializer_t> initializer_types)
    : Layer(gpu_resource, initializer_types),
      params_(params),
      mode_(HIPDNN_BATCHNORM_PER_ACTIVATION) {
  CudaDeviceContext context(get_device_id());
  const auto& in_tensor_dim = in_tensor.get_dimensions();
  const auto& out_tensor_dim = out_tensor.get_dimensions();

  assert(get_size_from_dims(in_tensor_dim) == get_size_from_dims(out_tensor_dim));
  assert(in_tensor_dim.size() == 2 && out_tensor_dim.size() == 2);
  assert(in_tensor_dim[0] == out_tensor_dim[0]);
  assert(in_tensor_dim[1] == out_tensor_dim[1]);

  HCTR_LIB_THROW(hipdnnCreateTensorDescriptor(&in_out_desc_));

  size_t num_feature = in_tensor_dim[1];
  int batch_size = in_tensor_dim[0];

  hipdnnDataType_t data_type = std::is_same<T, __half>::value ? HIPDNN_DATA_HALF : HIPDNN_DATA_FLOAT;
  int n_stride = num_feature;
  int w_stride = 1;

  HCTR_LIB_THROW(hipdnnSetTensor4dDescriptorEx(in_out_desc_, data_type, batch_size, 1, 1,
                                              num_feature, n_stride, 1, 1, w_stride));

  in_tensors_.push_back(in_tensor);
  out_tensors_.push_back(out_tensor);

  HCTR_LIB_THROW(hipdnnCreateTensorDescriptor(&gamma_beta_desc_));

  HCTR_LIB_THROW(hipdnnDeriveBNTensorDescriptor(gamma_beta_desc_, in_out_desc_, mode_));

  std::vector<size_t> gamma_dim = {num_feature, 1};

  // gamma & beta
  weight_buff->reserve(gamma_dim, &gamma_);
  weight_buff->reserve(gamma_dim, &beta_);
  weights_.push_back(gamma_);
  weights_.push_back(beta_);

  // gamma grad & beta grad
  wgrad_buff->reserve(gamma_dim, &gamma_grad_);
  wgrad_buff->reserve(gamma_dim, &beta_grad_);
  wgrad_.push_back(gamma_grad_);
  wgrad_.push_back(beta_grad_);

  // result running mean & var
  blob_buff->reserve(gamma_dim, &result_running_mean_);
  blob_buff->reserve(gamma_dim, &result_running_var_);

  // save running mean & var (cache)
  blob_buff->reserve(gamma_dim, &result_save_mean_);
  blob_buff->reserve(gamma_dim, &result_save_inv_var_);
}

template <typename T>
BatchNormLayer<T>::~BatchNormLayer() {
  try {
    HCTR_LIB_THROW(hipdnnDestroyTensorDescriptor(in_out_desc_));
    HCTR_LIB_THROW(hipdnnDestroyTensorDescriptor(gamma_beta_desc_));
  } catch (const std::runtime_error& rt_err) {
    HCTR_LOG_S(ERROR, WORLD) << rt_err.what() << std::endl;
  }
}

template <typename T>
void BatchNormLayer<T>::initialize() {
  // host array to get running mean & var

  size_t num_feature = in_tensors_[0].get_dimensions()[1];

  std::shared_ptr<GeneralBuffer2<HostAllocator>> internal_host_buf =
      GeneralBuffer2<HostAllocator>::create();

  internal_host_buf->reserve({num_feature}, &h_result_running_mean_);
  internal_host_buf->reserve({num_feature}, &h_result_running_var_);

  internal_host_buf->allocate();
}

template <typename T>
void BatchNormLayer<T>::fprop(bool is_train) {
  CudaDeviceContext context(get_device_id());
  float one = 1.0f, zero = 0.0f;

  Tensor2<T>& in_tensor = in_tensors_[0];
  Tensor2<T>& out_tensor = out_tensors_[0];
  T* in = in_tensor.get_ptr();
  T* out = out_tensor.get_ptr();

  float* gamma = gamma_.get_ptr();
  float* beta = beta_.get_ptr();

  float* result_running_mean = result_running_mean_.get_ptr();
  float* result_running_var = result_running_var_.get_ptr();
  float* result_save_mean = result_save_mean_.get_ptr();
  float* result_save_inv_var = result_save_inv_var_.get_ptr();

  if (is_train) {
    HCTR_LIB_THROW(hipdnnBatchNormalizationForwardTraining(
        get_gpu().get_cudnn_handle(), mode_, &one, &zero, in_out_desc_, in, in_out_desc_, out,
        gamma_beta_desc_, gamma, beta, params_.factor, result_running_mean, result_running_var,
        params_.eps, result_save_mean, result_save_inv_var));
  } else {
    HCTR_LIB_THROW(hipdnnBatchNormalizationForwardInference(
        get_gpu().get_cudnn_handle(), mode_, &one, &zero, in_out_desc_, in, in_out_desc_, out,
        gamma_beta_desc_, gamma, beta, result_running_mean, result_running_var, params_.eps));
  }
}

template <typename T>
void BatchNormLayer<T>::bprop() {
  CudaDeviceContext context(get_device_id());

  float one = 1.0f, zero = 0.0f;

  Tensor2<T>& in_tensor = in_tensors_[0];
  Tensor2<T>& out_tensor = out_tensors_[0];
  T* in = in_tensor.get_ptr();
  T* out = out_tensor.get_ptr();

  float* gamma = gamma_.get_ptr();

  float* gamma_grad = gamma_grad_.get_ptr();
  float* beta_grad = beta_grad_.get_ptr();

  float* result_save_mean = result_save_mean_.get_ptr();
  float* result_save_inv_var = result_save_inv_var_.get_ptr();

  HCTR_LIB_THROW(hipdnnBatchNormalizationBackward(
      get_gpu().get_cudnn_handle(), mode_, &one, &zero, &one, &zero, in_out_desc_, in, in_out_desc_,
      out, in_out_desc_, in, gamma_beta_desc_, gamma, gamma_grad, beta_grad, params_.eps,
      result_save_mean, result_save_inv_var));
}

template <typename T>
std::string BatchNormLayer<T>::get_no_trained_params_in_string() {
  float* d_result_running_mean = result_running_mean_.get_ptr();
  float* d_result_running_var = result_running_var_.get_ptr();
  size_t n_byte = result_running_mean_.get_size_in_bytes();
  size_t n_elem = n_byte / sizeof(T);

  HCTR_LIB_THROW(hipMemcpy(h_result_running_mean_.get_ptr(), d_result_running_mean, n_byte,
                            hipMemcpyDeviceToHost));
  HCTR_LIB_THROW(hipMemcpy(h_result_running_var_.get_ptr(), d_result_running_var, n_byte,
                            hipMemcpyDeviceToHost));

  std::string result = "      \"type\": \"BatchNorm\",\n";
  result += "      \"mean\": [";
  for (size_t i = 0; i < n_elem; i++) {
    result += std::to_string(ToStringType<T>(h_result_running_mean_.get_ptr()[i]));
    if (i != (n_elem - 1)) result += ", ";
  }
  result += "],\n";

  result += "      \"var\": [";
  for (size_t i = 0; i < n_elem; i++) {
    result += std::to_string(ToStringType<T>(h_result_running_var_.get_ptr()[i]));
    if (i != (n_elem - 1)) result += ", ";
  }
  result += "]";

  return result;
}

template <typename T>
std::vector<TensorBag2> BatchNormLayer<T>::get_tensors_for_non_trainable_params() {
  std::vector<TensorBag2> tensors;
  tensors.push_back(result_running_mean_.shrink());
  tensors.push_back(result_running_var_.shrink());
  return tensors;
}

template <typename T>
std::unique_ptr<DataSimulator> BatchNormLayer<T>::get_default_initializer(const int index) {
  std::unique_ptr<DataSimulator> simu;
  if (0 == index) {
    simu.reset(new ConstantDataSimulator(1.0f));
  } else if (1 == index) {
    simu.reset(new ConstantDataSimulator(0.0f));
  } else {
    HCTR_OWN_THROW(Error_t::OutOfBound, "index != {0, 1}.");
  }
  return simu;
}

template class BatchNormLayer<float>;
template class BatchNormLayer<__half>;

}  // namespace HugeCTR
