#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include <functional>
#include <include/utils.cuh>
#include <layers/element_wise_function.hpp>
#include <layers/softmax_layer.hpp>
#include <linalg/binary_op.cuh>
#include <linalg/reduce.cuh>
#include <linalg/unary_op.cuh>
#include <utils.hpp>

#ifndef NDEBUG
#include <iostream>
#endif

namespace HugeCTR {

template <typename T>
SoftmaxLayer<T>::SoftmaxLayer(const Tensor2<T>& in_tensor, const Tensor2<T>& out_tensor,
                              const std::shared_ptr<GeneralBuffer2<CudaAllocator>>& blobs_buff,
                              const std::shared_ptr<GPUResource>& gpu_resource)
    : Layer(gpu_resource) {
  assert(in_tensor.get_num_elements() == out_tensor.get_num_elements());
  in_tensors_.push_back(in_tensor);
  out_tensors_.push_back(out_tensor);

  len_ = in_tensors_[0].get_num_elements();
  dims_ = in_tensor.get_dimensions().size();
  hidden_size_ = in_tensor.get_dimensions()[dims_ - 1];
  n_rows_ = len_ / hidden_size_;
  blobs_buff->reserve({n_rows_}, &workspace_);
  blobs_buff->reserve({hidden_size_}, &identity_);
  blobs_buff->reserve(in_tensor.get_dimensions(), &softmax_out_);
}

template <typename T>
void SoftmaxLayer<T>::initialize() {
  CudaDeviceContext context(get_device_id());
  initialize_array<<<(hidden_size_ - 1) / 1024 + 1, 1024, 0, get_gpu().get_stream()>>>(
      identity_.get_ptr(), hidden_size_, 1.0f);
}

template <>
void SoftmaxLayer<__half>::initialize() {
  CudaDeviceContext context(get_device_id());
  initialize_array<<<(hidden_size_ - 1) / 1024 + 1, 1024, 0, get_gpu().get_stream()>>>(
      identity_.get_ptr(), hidden_size_, __float2half(1.0f));
}

template <typename T>
void __global__ Softmax_fprop_kernel(T* out, T* workspace, int m, int n) {
  int offset = blockIdx.x * n;
  for (int tid = threadIdx.x; tid < n; tid += blockDim.x) {
    out[offset + tid] = out[offset + tid] / workspace[blockIdx.x];
  }
}

template <>
void __global__ Softmax_fprop_kernel(__half* out, __half* workspace, int m, int n) {
  int offset = blockIdx.x * n;
  for (int tid = threadIdx.x; tid < n; tid += blockDim.x) {
    out[offset + tid] = __hdiv(out[offset + tid], workspace[blockIdx.x]);
  }
}

template <typename T>
void Softmax_fprop(T* out, T* workspace, int m, int n, hipStream_t stream) {
  dim3 grid(m);
  dim3 block(min(n, 1024));
  Softmax_fprop_kernel<<<grid, block, 0, stream>>>(out, workspace, m, n);
#ifndef NDEBUG
  hipDeviceSynchronize();
  HCTR_LIB_THROW(hipGetLastError());
#endif
}

template <typename T>
void __global__ Softmax_bprop_kernel(T* top, T* bottom, T* softmax, int m, int n) {
  int offset = blockIdx.x * n;
  float grad_softmax = static_cast<float>(0.0f);
  __shared__ float grad_sum;
  for (int tid = threadIdx.x; tid < n; tid += blockDim.x) {
    int idx = offset + tid;
    grad_softmax += top[idx] * softmax[idx];
  }
  float tmp = blockReduceSum<T>(grad_softmax);
  if (threadIdx.x == 0) {
    grad_sum = tmp;
  }
  __syncthreads();

  for (int tid = threadIdx.x; tid < n; tid += blockDim.x) {
    int idx = offset + tid;
    bottom[idx] = softmax[idx] * top[idx] - softmax[idx] * grad_sum;
  }
}

template <>
void __global__ Softmax_bprop_kernel(__half* top, __half* bottom, __half* softmax, int m, int n) {
  int offset = blockIdx.x * n;
  float grad_softmax = static_cast<float>(0.0f);
  __shared__ __half grad_sum;

  for (int tid = threadIdx.x; tid < n; tid += blockDim.x) {
    int idx = offset + tid;
    grad_softmax += static_cast<float>(top[idx] * softmax[idx]);
  }

  float tmp = blockReduceSum<float>(grad_softmax);
  if (threadIdx.x == 0) {
    grad_sum = static_cast<__half>(tmp);
  }
  __syncthreads();
  for (int tid = threadIdx.x; tid < n; tid += blockDim.x) {
    int idx = offset + tid;
    __half tmp = __hsub(top[idx], grad_sum);
    bottom[idx] = __hmul(bottom[idx], tmp);
  }
}

template <typename T>
void Softmax_bprop(T* top, T* bottom, T* softmax_out, int m, int n, hipStream_t stream) {
  dim3 grid(m);
  dim3 block(min(n, 1024));
  Softmax_bprop_kernel<<<grid, block, 0, stream>>>(top, bottom, softmax_out, m, n);
#ifndef NDEBUG
  hipDeviceSynchronize();
  HCTR_LIB_THROW(hipGetLastError());
#endif
}

template <typename T>
void SoftmaxLayer<T>::fprop(bool is_train) {
  CudaDeviceContext context(get_device_id());
  Tensor2<T>& in_tensor = in_tensors_[0];
  Tensor2<T>& out_tensor = out_tensors_[0];
  const auto& in_tensor_dim = in_tensor.get_dimensions();
  // exp(x_i)
  MLCommon::LinAlg::unaryOp(
      out_tensor.get_ptr(), in_tensor.get_ptr(), len_, [] __device__(T in) { return expf(in); },
      get_gpu().get_stream());
  // Get sum of exp(x_i) i=[0, embedding_vector_size-1].
  MLCommon::LinAlg::reduce(workspace_.get_ptr(), out_tensor.get_ptr(), hidden_size_, n_rows_, T(0),
                           true, true, get_gpu().get_stream());
  // Softmax exp(x_i) / sum(exp)(x_i)) i=[0, embedding_vector_size-1].
  Softmax_fprop(out_tensor.get_ptr(), workspace_.get_ptr(), n_rows_, hidden_size_,
                get_gpu().get_stream());
  HCTR_LIB_THROW(hipMemcpyAsync((void*)softmax_out_.get_ptr(), (void*)out_tensor.get_ptr(),
                                 out_tensor.get_size_in_bytes(), hipMemcpyDeviceToDevice,
                                 get_gpu().get_stream()));
#ifndef NDEBUG
  hipDeviceSynchronize();
  HCTR_LIB_THROW(hipGetLastError());
#endif
}

template <>
void SoftmaxLayer<__half>::fprop(bool is_train) {
  CudaDeviceContext context(get_device_id());
  Tensor2<__half>& in_tensor = in_tensors_[0];
  Tensor2<__half>& out_tensor = out_tensors_[0];
  const auto& in_tensor_dim = in_tensor.get_dimensions();

  const __half alpha = __float2half(1.0f);
  const __half beta = __float2half(0.0f);
  // exp(x_i)
  MLCommon::LinAlg::unaryOp(
      out_tensor.get_ptr(), in_tensor.get_ptr(), len_,
      [] __device__(__half in) { return hexp(in); }, get_gpu().get_stream());
  // Get sum of exp(x_i) i=[0, embedding_vector_size-1]
  HCTR_LIB_THROW(hipblasGemmEx(
      get_gpu().get_cublas_handle(), HIPBLAS_OP_T, HIPBLAS_OP_N, n_rows_, 1, hidden_size_, &alpha,
      out_tensor.get_ptr(), HIP_R_16F, hidden_size_, identity_.get_ptr(), HIP_R_16F, hidden_size_,
      &beta, workspace_.get_ptr(), HIP_R_16F, n_rows_, HIP_R_16F, HIPBLAS_GEMM_DEFAULT));
  // Softmax exp(x_i) / sum(exp)(x_i)) i=[0, embedding_vector_size-1]
  Softmax_fprop(out_tensor.get_ptr(), workspace_.get_ptr(), n_rows_, hidden_size_,
                get_gpu().get_stream());
  HCTR_LIB_THROW(hipMemcpyAsync((void*)softmax_out_.get_ptr(), (void*)out_tensor.get_ptr(),
                                 out_tensor.get_size_in_bytes(), hipMemcpyDeviceToDevice,
                                 get_gpu().get_stream()));
#ifndef NDEBUG
  hipDeviceSynchronize();
  HCTR_LIB_THROW(hipGetLastError());
#endif
}

template <typename T>
void SoftmaxLayer<T>::bprop() {
  CudaDeviceContext context(get_device_id());
  Tensor2<T>& bottom_tensor = in_tensors_[0];
  Tensor2<T>& top_tensor = out_tensors_[0];
  const auto& in_tensor_dim = bottom_tensor.get_dimensions();

  const size_t len = bottom_tensor.get_num_elements();

  Softmax_bprop(top_tensor.get_ptr(), bottom_tensor.get_ptr(), softmax_out_.get_ptr(), n_rows_,
                hidden_size_, get_gpu().get_stream());

#ifndef NDEBUG
  hipDeviceSynchronize();
  HCTR_LIB_THROW(hipGetLastError());
#endif
}

template <>
void SoftmaxLayer<__half>::bprop() {
  CudaDeviceContext context(get_device_id());
  Tensor2<__half>& bottom_tensor = in_tensors_[0];
  Tensor2<__half>& top_tensor = out_tensors_[0];
  const auto& in_tensor_dim = bottom_tensor.get_dimensions();

  Softmax_bprop(top_tensor.get_ptr(), bottom_tensor.get_ptr(), softmax_out_.get_ptr(), n_rows_,
                hidden_size_, get_gpu().get_stream());

#ifndef NDEBUG
  hipDeviceSynchronize();
  HCTR_LIB_THROW(hipGetLastError());
#endif
}

template class SoftmaxLayer<float>;
template class SoftmaxLayer<__half>;

}  // namespace HugeCTR
