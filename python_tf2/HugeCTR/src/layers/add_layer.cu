#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include <functional>
#include <layers/add_layer.hpp>
#include <utils.cuh>
#include <utils.hpp>

#ifndef NDEBUG
#include <iostream>
#endif

namespace HugeCTR {

namespace {

#define BLOCK_DIM_SIZE 32
template <typename T>
__global__ void add_kernel(T** inputs, T* output, int size, int num) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid < size) {
    T tmp = 0;
    for (int i = 0; i < num; i++) {
      tmp += inputs[i][tid];
    }
    output[tid] = tmp;
  }
}

template <typename T>
__global__ void add_dgrad_kernel(const T* top_grad, T** dgrads, int size, int num) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid < size) {
    for (int i = 0; i < num; i++) {
      dgrads[i][tid] = top_grad[tid];
    }
  }
}

template <>
__global__ void add_kernel<__half>(__half** inputs, __half* output, int size, int num) {
  const __half2** inputs2 = (const __half2**)(inputs);
  __half2* output2 = (__half2*)(output);
  int size2 = size / 2;

  const __half2 zero = __half2half2(__float2half(0.f));
  int start = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = start; i < size2; i += stride) {
    __half2 tmp = zero;
    for (int j = 0; j < num; ++j) {
      tmp += inputs2[j][i];
    }
    output2[i] = tmp;
  }
  if (start == 0 && size % 2 > 0) {
    __half tmp = __float2half(0.0f);
    for (int j = 0; j < num; ++j) {
      tmp += inputs[j][size - 1];
    }
    output[size - 1] = tmp;
  }
}

template <>
__global__ void add_dgrad_kernel<__half>(const __half* top_grad, __half** dgrads, int size,
                                         int num) {
  const __half2* top_grad2 = (const __half2*)(top_grad);
  __half2** dgrads2 = (__half2**)(dgrads);
  int size2 = size / 2;

  int start = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = start; i < size2; i += stride) {
    for (int j = 0; j < num; ++j) {
      dgrads2[j][i] = top_grad2[i];
    }
  }
  if (start == 0 && size % 2 > 0) {
    for (int j = 0; j < num; ++j) {
      dgrads[j][size - 1] = top_grad[size - 1];
    }
  }
}

}  // end of namespace

template <typename T>
AddLayer<T>::AddLayer(const Tensors2<T>& in_tensors, const Tensor2<T>& out_tensor,
                      const std::shared_ptr<GeneralBuffer2<CudaAllocator>>& blobs_buff,
                      const std::shared_ptr<GPUResource>& gpu_resource)
    : Layer(gpu_resource) {
  try {
    size_ = in_tensors[0].get_num_elements();
    num_ = in_tensors.size();

    // error input checking
    auto dims = in_tensors[0].get_dimensions();
    if (num_ < 2) {
      HCTR_OWN_THROW(Error_t::WrongInput, "AddLayer needs at least 2 input tensors");
    }
    for (size_t i = 1; i < num_; i++) {
      if (in_tensors[i].get_dimensions().size() != dims.size()) {
        HCTR_OWN_THROW(Error_t::WrongInput, "All the input tensors must have the same num of dims");
      }
      for (unsigned int j = 0; j < dims.size(); j++) {
        if (in_tensors[i].get_dimensions()[j] != dims[j]) {
          HCTR_OWN_THROW(Error_t::WrongInput, "All the input tensors must have the same dims");
        }
      }
    }

    for (size_t i = 0; i < num_; i++) {
      in_tensors_.push_back(in_tensors[i]);
    }
    out_tensors_.push_back(out_tensor);

    blobs_buff->reserve({num_}, &d_inputs_);

  } catch (const std::runtime_error& rt_err) {
    HCTR_LOG_S(ERROR, WORLD) << rt_err.what() << std::endl;
    throw;
  }
}

template <typename T>
void AddLayer<T>::initialize() {
  std::shared_ptr<GeneralBuffer2<CudaHostAllocator>> pinned_host_buf =
      GeneralBuffer2<CudaHostAllocator>::create();
  pinned_host_buf->reserve({num_}, &h_inputs_);
  pinned_host_buf->allocate();

  for (size_t i = 0; i < num_; i++) {
    h_inputs_.get_ptr()[i] = in_tensors_[i].get_ptr();
  }

  HCTR_LIB_THROW(hipMemcpyAsync((void*)d_inputs_.get_ptr(), (void*)h_inputs_.get_ptr(),
                                 num_ * sizeof(T*), hipMemcpyHostToDevice,
                                 get_gpu().get_stream()));
}

template <typename T>
void AddLayer<T>::fprop(bool is_train) {
  CudaDeviceContext context(get_device_id());

  T* output = out_tensors_[0].get_ptr();

  dim3 block_size(256, 1, 1);
  dim3 grid_size((size_ + block_size.x - 1) / block_size.x, 1, 1);
  add_kernel<<<grid_size, block_size, 0, get_gpu().get_stream()>>>(d_inputs_.get_ptr(), output,
                                                                   size_, num_);
}

template <typename T>
void AddLayer<T>::bprop() {
  CudaDeviceContext context(get_device_id());

  T* output = out_tensors_[0].get_ptr();

  dim3 blockSize(256, 1, 1);
  dim3 gridSize((size_ + blockSize.x - 1) / blockSize.x, 1, 1);
  add_dgrad_kernel<<<gridSize, blockSize, 0, get_gpu().get_stream()>>>(output, d_inputs_.get_ptr(),
                                                                       size_, num_);
}

template <>
void AddLayer<__half>::fprop(bool is_train) {
  CudaDeviceContext context(get_device_id());

  __half* output = out_tensors_[0].get_ptr();

  dim3 block_size(256, 1, 1);
  dim3 grid_size((size_ / 2 + block_size.x - 1) / block_size.x, 1, 1);
  add_kernel<<<grid_size, block_size, 0, get_gpu().get_stream()>>>(d_inputs_.get_ptr(), output,
                                                                   size_, num_);
}

template <>
void AddLayer<__half>::bprop() {
  CudaDeviceContext context(get_device_id());

  __half* output = out_tensors_[0].get_ptr();

  dim3 blockSize(256, 1, 1);
  dim3 gridSize((size_ / 2 + blockSize.x - 1) / blockSize.x, 1, 1);
  add_dgrad_kernel<<<gridSize, blockSize, 0, get_gpu().get_stream()>>>(output, d_inputs_.get_ptr(),
                                                                       size_, num_);
}

template class AddLayer<float>;
template class AddLayer<__half>;

}  // namespace HugeCTR
