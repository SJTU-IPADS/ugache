#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include <functional>
#include <layers/fused_reshape_concat_layer.hpp>
#include <utils.cuh>
#include <utils.hpp>

#ifndef NDEBUG
#include <iostream>
#endif

namespace HugeCTR {

namespace {

#define BLOCK_DIM_SIZE 32
template <typename T>
__global__ void fused_reshape_concat_kernel(bool forward, T** inputs, T* output_item, T* output_ad,
                                            int batch_size, int slot_num, size_t* vecs_size,
                                            int output_width, int num) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int threads_num = blockDim.x * gridDim.x;
  int total_size = batch_size * slot_num * output_width;

  for (int index = tid; index < total_size; index += threads_num) {
    int row = index / output_width;
    int out_col = index % output_width;

    int in_no = 0;
    int in_col = out_col;
    int accum_width = 0;
    for (int k = 0; k < num; k++) {
      if (out_col < accum_width + vecs_size[k]) {
        in_no = k;
        in_col -= accum_width;
        break;
      }
      accum_width += vecs_size[k];
    }
    T* in = inputs[in_no];
    int in_idx = row * vecs_size[in_no] + in_col;
    int out_row = ((row + 1) % slot_num == 0) ? (row / slot_num) : (row - (row / slot_num));
    int out_idx = out_row * output_width + out_col;
    T* output = (row + 1) % slot_num == 0 ? output_ad : output_item;
    if (forward) {
      output[out_idx] = in[in_idx];
    } else {
      in[in_idx] = output[out_idx];
    }
  }
}

}  // end of namespace

template <typename T>
FusedReshapeConcatLayer<T>::FusedReshapeConcatLayer(
    const Tensors2<T>& in_tensors, Tensors2<T>& out_tensors,
    const std::shared_ptr<GeneralBuffer2<CudaAllocator>>& blobs_buff,
    const std::shared_ptr<GPUResource>& gpu_resource)
    : Layer(gpu_resource) {
  try {
    if (in_tensors.empty()) {
      HCTR_OWN_THROW(Error_t::WrongInput, "Empty input tensors");
    }

    num_ = in_tensors.size();
    for (size_t i = 0; i < num_; i++) {
      auto cur_in_dims = in_tensors[i].get_dimensions();
      if (i != 0) {
        auto first_in_dims = in_tensors[0].get_dimensions();
        if (cur_in_dims[0] != first_in_dims[0]) {
          HCTR_OWN_THROW(Error_t::WrongInput,
                         "All the input tensors must have the same batch_size");
        }
        if (cur_in_dims[1] != first_in_dims[1]) {
          HCTR_OWN_THROW(Error_t::WrongInput, "All the input tensors must have the same slot_num");
        }
      }
      if (cur_in_dims.size() != 3) {
        HCTR_OWN_THROW(Error_t::WrongInput, "All the input tensors must be 3D");
      }
      if (i == 0) {
        batch_size_ = cur_in_dims[0];
        slot_num_ = cur_in_dims[1];
      }
      new_width_ += cur_in_dims[2];
      h_vecs_size_.push_back(cur_in_dims[2]);
    }

    {
      std::vector<size_t> out_dims_item = {batch_size_ * (slot_num_ - 1), new_width_};
      Tensor2<T> tensor_item;
      blobs_buff->reserve(out_dims_item, &tensor_item);
      out_tensors.push_back(tensor_item);

      std::vector<size_t> out_dims_ad = {batch_size_, new_width_};
      Tensor2<T> tensor_ad;
      blobs_buff->reserve(out_dims_ad, &tensor_ad);
      out_tensors.push_back(tensor_ad);
    }

    for (const Tensor2<T>& in_tensor : in_tensors) {
      in_tensors_.push_back(in_tensor);
    }

    blobs_buff->reserve({num_}, &d_inputs_);
    blobs_buff->reserve({num_}, &vecs_size_);

    for (auto& out_tensor : out_tensors) {
      out_tensors_.push_back(out_tensor);
    }

  } catch (const std::runtime_error& rt_err) {
    HCTR_LOG_S(ERROR, WORLD) << rt_err.what() << std::endl;
    throw;
  }
}

template <typename T>
void FusedReshapeConcatLayer<T>::initialize() {
  std::shared_ptr<GeneralBuffer2<CudaHostAllocator>> pinned_host_buf =
      GeneralBuffer2<CudaHostAllocator>::create();
  pinned_host_buf->reserve({num_}, &h_inputs_);
  pinned_host_buf->allocate();

  for (size_t i = 0; i < num_; i++) {
    h_inputs_.get_ptr()[i] = in_tensors_[i].get_ptr();
  }
  HCTR_LIB_THROW(hipMemcpyAsync((void*)vecs_size_.get_ptr(), (void*)h_vecs_size_.data(),
                                 num_ * sizeof(size_t), hipMemcpyHostToDevice,
                                 get_gpu().get_stream()));

  HCTR_LIB_THROW(hipMemcpyAsync((void*)d_inputs_.get_ptr(), (void*)h_inputs_.get_ptr(),
                                 num_ * sizeof(T*), hipMemcpyHostToDevice,
                                 get_gpu().get_stream()));
}

template <typename T>
void FusedReshapeConcatLayer<T>::fprop(bool is_train) {
  CudaDeviceContext context(get_device_id());
  Tensors2<T>& out_tensors = out_tensors_;
  T* output_item = out_tensors[0].get_ptr();
  T* output_ad = out_tensors[1].get_ptr();
  dim3 block_size(256, 1, 1);
  size_t n_sms = get_gpu().get_sm_count();
  dim3 grid_size(n_sms * 8, 1, 1);
  fused_reshape_concat_kernel<<<grid_size, block_size, 0, get_gpu().get_stream()>>>(
      true, d_inputs_.get_ptr(), output_item, output_ad, batch_size_, slot_num_,
      vecs_size_.get_ptr(), new_width_, num_);
#ifndef NDEBUG
  hipDeviceSynchronize();
  HCTR_LIB_THROW(hipGetLastError());
#endif
}

template <typename T>
void FusedReshapeConcatLayer<T>::bprop() {
  CudaDeviceContext context(get_device_id());
  Tensors2<T>& out_tensors = out_tensors_;
  T* output_item = out_tensors[0].get_ptr();
  T* output_ad = out_tensors[1].get_ptr();
  dim3 block_size(256, 1, 1);
  size_t n_sms = get_gpu().get_sm_count();
  dim3 grid_size(n_sms * 8, 1, 1);
  fused_reshape_concat_kernel<<<grid_size, block_size, 0, get_gpu().get_stream()>>>(
      false, d_inputs_.get_ptr(), output_item, output_ad, batch_size_, slot_num_,
      vecs_size_.get_ptr(), new_width_, num_);
#ifndef NDEBUG
  hipDeviceSynchronize();
  HCTR_LIB_THROW(hipGetLastError());
#endif
}

template class FusedReshapeConcatLayer<float>;

}  // namespace HugeCTR
