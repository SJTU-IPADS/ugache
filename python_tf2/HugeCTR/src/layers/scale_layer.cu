#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include <cuda_utils.cuh>
#include <functional>
#include <include/utils.cuh>
#include <layers/element_wise_function.hpp>
#include <layers/scale_layer.hpp>
#include <linalg/binary_op.cuh>
#include <linalg/reduce.cuh>
#include <linalg/unary_op.cuh>
#include <utils.hpp>

#ifndef NDEBUG
#include <iostream>
#endif

namespace HugeCTR {

template <typename T>
ScaleLayer<T>::ScaleLayer(const Tensor2<T>& in_tensor, Tensor2<T>& out_tensor,
                          const std::shared_ptr<GeneralBuffer2<CudaAllocator>>& blobs_buff,
                          int axis, int factor, const std::shared_ptr<GPUResource>& gpu_resource)
    : Layer(gpu_resource) {
  assert(axis < 2);
  size_t out_y = axis == 1 ? in_tensor.get_dimensions()[0] * factor : in_tensor.get_dimensions()[0];
  size_t out_x = axis == 0 ? in_tensor.get_dimensions()[1] * factor : in_tensor.get_dimensions()[1];
  std::vector<size_t> out_dims = {out_y, out_x};
  blobs_buff->reserve(out_dims, &out_tensor);

  in_tensors_.push_back(in_tensor);
  out_tensors_.push_back(out_tensor);
  axis_ = axis;
  factor_ = factor;
}

template <typename T>
void __global__ upscale_kernel(T* out, T* in, int batchsize, int num_elems, int axis, int factor) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int threads_num = blockDim.x * gridDim.x;
  int len = batchsize * num_elems;
  if (axis == 0) {
    for (int index = tid; index < len; index += threads_num) {
      for (int i = 0; i < factor; i++) {
        out[index * factor + i] = in[index];
      }
    }
  } else {
    for (int index = threadIdx.x; index < num_elems; index += blockDim.x) {
      for (int i = 0; i < factor; i++) {
        out[blockIdx.x * factor * num_elems + i * num_elems + index] =
            in[index + blockIdx.x * num_elems];
      }
    }
  }
}

template <typename T>
void __global__ downscale_kernel(T* out, T* in, int batchsize, int num_elems, int axis,
                                 int factor) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int threads_num = blockDim.x * gridDim.x;
  int len = batchsize * num_elems;
  if (axis == 0) {
    for (int index = tid; index < len; index += threads_num) {
      out[index] = in[index * factor];
    }
  } else {
    for (int index = threadIdx.x; index < num_elems; index += blockDim.x) {
      out[blockIdx.x * num_elems + index] = in[index + blockIdx.x * num_elems * factor];
    }
  }
}

template <typename T>
void scale(T* out, T* in, int batchsize, int num_elems, int axis, int factor, hipStream_t stream,
           bool forward) {
  dim3 grid(batchsize);
  dim3 block(min(num_elems, 1024));

  if (forward)
    upscale_kernel<<<grid, block, 0, stream>>>(out, in, batchsize, num_elems, axis, factor);
  else
    downscale_kernel<<<grid, block, 0, stream>>>(out, in, batchsize, num_elems, axis, factor);
#ifndef NDEBUG
  hipDeviceSynchronize();
  HCTR_LIB_THROW(hipGetLastError());
#endif
}

template <typename T>
void ScaleLayer<T>::fprop(bool is_train) {
  CudaDeviceContext context(get_device_id());
  Tensor2<T>& in_tensor = in_tensors_[0];
  Tensor2<T>& out_tensor = out_tensors_[0];
  const auto& in_tensor_dim = in_tensor.get_dimensions();
  int axis = axis_;
  int factor = factor_;

  scale(out_tensor.get_ptr(), in_tensor.get_ptr(), in_tensor_dim[0], in_tensor_dim[1], axis, factor,
        get_gpu().get_stream(), true);
}

template <typename T>
void ScaleLayer<T>::bprop() {
  CudaDeviceContext context(get_device_id());
  Tensor2<T>& bottom_tensor = in_tensors_[0];
  Tensor2<T>& top_tensor = out_tensors_[0];
  const auto& bottom_tensor_dim = bottom_tensor.get_dimensions();
  int axis = axis_;
  int factor = factor_;

  scale(bottom_tensor.get_ptr(), top_tensor.get_ptr(), bottom_tensor_dim[0], bottom_tensor_dim[1],
        axis, factor, get_gpu().get_stream(), false);
}

template class ScaleLayer<float>;
// template class ScaleLayer<__half>;

}  // namespace HugeCTR
