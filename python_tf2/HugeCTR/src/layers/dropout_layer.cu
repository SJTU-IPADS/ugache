#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <HugeCTR/include/utils.hpp>
#include <algorithm>
#include <cstdio>
#include <ctime>
#include <functional>
#include <layers/dropout_layer.hpp>
#include <prims/linalg/binary_op.cuh>
#include <utils.cuh>
#include <utils.hpp>

#ifndef NDEBUG
#include <iostream>
#endif

namespace HugeCTR {

template <typename T>
DropoutLayer<T>::DropoutLayer(const Tensor2<T>& in_tensor, const Tensor2<T>& out_tensor,
                              const std::shared_ptr<GeneralBuffer2<CudaAllocator>> blobs_buff,
                              float rate, const std::shared_ptr<GPUResource>& gpu_resource)

    : Layer(gpu_resource), rate_(rate), scale_(1.0 / (1.0 - rate)) {
  assert(in_tensor.get_num_elements() == out_tensor.get_num_elements());
  assert(rate_ > 0.f && rate_ < 1.f);

  const auto& in_tensor_dim = in_tensor.get_dimensions();
  in_tensors_.emplace_back(in_tensor);
  out_tensors_.emplace_back(out_tensor);

  CudaDeviceContext context(get_device_id());

  size_t num_feature = in_tensor_dim[1];
  int batch_size = in_tensor_dim[0];
  hipdnnDataType_t data_type = CudnnDataType<T>::getType();
  int n_stride = num_feature;
  int w_stride = 1;
  HCTR_LIB_THROW(hipdnnCreateTensorDescriptor(&in_out_desc_));
  HCTR_LIB_THROW(hipdnnSetTensor4dDescriptorEx(in_out_desc_, data_type, batch_size, 1, 1,
                                              num_feature, n_stride, 1, 1, w_stride));

  HCTR_LIB_THROW(hipdnnCreateDropoutDescriptor(&dropout_descriptor_));

  size_t sizeInBytes = 0;

  HCTR_LIB_THROW(hipdnnDropoutGetStatesSize(gpu_resource->get_cudnn_handle(), &sizeInBytes));

  assert(sizeInBytes != 0);

  HCTR_LIB_THROW(cudnnDropoutGetReserveSpaceSize(in_out_desc_, &reserveSpaceSizeInBytes_));

  blobs_buff->reserve({1, reserveSpaceSizeInBytes_}, &mask_);

  HCTR_LIB_THROW(hipMalloc(&cudnn_status_, sizeInBytes));

  HCTR_LIB_THROW(hipdnnSetDropoutDescriptor(dropout_descriptor_, gpu_resource->get_cudnn_handle(),
                                           rate, cudnn_status_, sizeInBytes, 0));
}

template <typename T>
DropoutLayer<T>::~DropoutLayer() {
  try {
    HCTR_LIB_THROW(hipdnnDestroyDropoutDescriptor(dropout_descriptor_));
    HCTR_LIB_THROW(hipFree(cudnn_status_));
    HCTR_LIB_THROW(hipdnnDestroyTensorDescriptor(in_out_desc_));
  } catch (const std::runtime_error& rt_err) {
    HCTR_LOG_S(ERROR, WORLD) << rt_err.what() << std::endl;
  }
}

template <typename T>
void DropoutLayer<T>::fprop(bool is_train) {
  CudaDeviceContext context(get_device_id());

  if (is_train) {
    HCTR_LIB_THROW(cudnnDropoutForward(
        get_gpu().get_cudnn_handle(), dropout_descriptor_, in_out_desc_, in_tensors_[0].get_ptr(),
        in_out_desc_, out_tensors_[0].get_ptr(), mask_.get_ptr(), reserveSpaceSizeInBytes_));
  } else {
    HCTR_LIB_THROW(hipMemcpyAsync(out_tensors_[0].get_ptr(), in_tensors_[0].get_ptr(),
                                   in_tensors_[0].get_size_in_bytes(), hipMemcpyDeviceToDevice,
                                   get_gpu().get_stream()));
  }

#ifndef NDEBUG
  hipDeviceSynchronize();
  HCTR_LIB_THROW(hipGetLastError());
#endif
}

template <typename T>
void DropoutLayer<T>::bprop() {
  CudaDeviceContext context(get_device_id());
  HCTR_LIB_THROW(cudnnDropoutBackward(
      get_gpu().get_cudnn_handle(), dropout_descriptor_, in_out_desc_, out_tensors_[0].get_ptr(),
      in_out_desc_, in_tensors_[0].get_ptr(), mask_.get_ptr(), reserveSpaceSizeInBytes_));

#ifndef NDEBUG
  hipDeviceSynchronize();
  HCTR_LIB_THROW(hipGetLastError());
#endif
}

template class DropoutLayer<float>;
template class DropoutLayer<__half>;

}  // namespace HugeCTR
