#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include <functional>
#include <include/utils.cuh>
#include <layers/element_wise_function.hpp>
#include <layers/masked_softmax_layer.hpp>
#include <linalg/binary_op.cuh>
#include <linalg/reduce.cuh>
#include <linalg/unary_op.cuh>
#include <utils.hpp>

#ifndef NDEBUG
#include <iostream>
#endif

namespace HugeCTR {
#define MAX_NUM_STRIDE 64

template <typename T>
MaskedSoftmaxLayer<T>::MaskedSoftmaxLayer(
    const Tensors2<T>& in_tensors, const Tensor2<T>& out_tensor, float scalar,
    const std::shared_ptr<GeneralBuffer2<CudaAllocator>>& blobs_buff,
    const std::shared_ptr<GPUResource>& gpu_resource)
    : Layer(gpu_resource), scalar_(scalar) {
  // Input 0: input data [batch_size, head, seq_len, seq_len]
  // Input 1: mask [batch_size, 1, 1, seq_len]
  assert(in_tensors[0].get_num_elements() == out_tensor.get_num_elements());
  size_t num_ = in_tensors.size();

  size_t dims_ = in_tensors[0].get_dimensions().size();
  if (num_ < 2) {
    HCTR_OWN_THROW(Error_t::WrongInput, "MaskedSoftmaxLayer needs at least 2 input tensors");
  }
  if (in_tensors[1].get_dimensions().size() != dims_) {
    HCTR_OWN_THROW(Error_t::WrongInput, "All the input tensors must have the same num of dims");
  }
  if (in_tensors[1].get_dimensions()[dims_ - 1] != in_tensors[0].get_dimensions()[dims_ - 1]) {
    HCTR_OWN_THROW(Error_t::WrongInput,
                   "The last dimension of the input tensors should be the same");
  }

  for (size_t i = 0; i < num_; i++) {
    in_tensors_.push_back(in_tensors[i]);
  }
  out_tensors_.push_back(out_tensor);

  blobs_buff->reserve(in_tensors[0].get_dimensions(), &softmax_out_);
}

// grid = (seq_len, head_num, batch_size)
// block.x = max(32, (seq_len + 31)/32*32)
template <typename T>
void __global__ mask_softmax_fprop_kernel(T* out, T* in, const T* mask, const int batch_size,
                                          const int head_num, const int seq_len,
                                          const float scalar) {
  float data[MAX_NUM_STRIDE];
  float local_max = -1e20f;
  float local_sum = 0.0f;
  int input_offset;
  __shared__ float s_rsum, s_max;
  for (int idx = 0; blockDim.x * idx + threadIdx.x < seq_len; idx++) {
    input_offset = ((blockIdx.z * head_num + blockIdx.y) * seq_len + blockIdx.x) * seq_len +
                   blockDim.x * idx + threadIdx.x;
    int mask_offset = blockIdx.z * seq_len + blockDim.x * idx + threadIdx.x;

    float in_val = static_cast<float>(in[input_offset]);
    float mask_val = (float)mask[mask_offset];
    mask_val = (1.0f - mask_val) * 10000.0f;
    data[idx] = in_val * scalar - (float)mask_val;
    local_max = fmax(local_max, data[idx]);
  }
  float max_val = blockReduceMax<float>(local_max);
  if (threadIdx.x == 0) {
    s_max = max_val;
  }
  __syncthreads();
  for (int idx = 0; blockDim.x * idx + threadIdx.x < seq_len; idx++) {
    data[idx] = __expf(data[idx] - s_max);
    local_sum += data[idx];
  }
  float sum_val = blockReduceSum<float>(local_sum);
  if (threadIdx.x == 0) {
    s_rsum = sum_val + 1e-6f;
    s_rsum = __fdividef(1.0f, s_rsum);
  }
  __syncthreads();

  for (int idx = 0; blockDim.x * idx + threadIdx.x < seq_len; idx++) {
    input_offset = ((blockIdx.z * head_num + blockIdx.y) * seq_len + blockIdx.x) * seq_len +
                   blockDim.x * idx + threadIdx.x;

    out[input_offset] = static_cast<T>(data[idx] * s_rsum);
  }
}

template <typename T>
void mask_softmax_fprop(T* out, T* in, T* mask, int batch_size, int head_num, int seq_len,
                        float scalar, hipStream_t stream) {
  dim3 grid(seq_len, head_num, batch_size);
  int block_len = max(32, (seq_len + 31) / 32 * 32);
  dim3 block(min(block_len, 1024));
  mask_softmax_fprop_kernel<<<grid, block, 0, stream>>>(out, in, mask, batch_size, head_num,
                                                        seq_len, scalar);
#ifndef NDEBUG
  hipDeviceSynchronize();
  HCTR_LIB_THROW(hipGetLastError());
#endif
}

template <typename T>
void __global__ mask_softmax_bprop_kernel(T* top, T* bottom, T* softmax, int m, int n,
                                          float scalar) {
  int offset = blockIdx.x * n;
  float grad_softmax = static_cast<float>(0.0f);
  __shared__ float grad_sum;
  for (int tid = threadIdx.x; tid < n; tid += blockDim.x) {
    int idx = offset + tid;
    grad_softmax += top[idx] * softmax[idx];
  }
  float tmp = blockReduceSum<T>(grad_softmax);
  if (threadIdx.x == 0) {
    grad_sum = tmp;
  }
  __syncthreads();

  for (int tid = threadIdx.x; tid < n; tid += blockDim.x) {
    int idx = offset + tid;
    bottom[idx] = (softmax[idx] * top[idx] - softmax[idx] * grad_sum) * scalar;
  }
}

template <>
void __global__ mask_softmax_bprop_kernel(__half* top, __half* bottom, __half* softmax, int m,
                                          int n, float scalar) {
  int offset = blockIdx.x * n;
  float grad_softmax = static_cast<float>(0.0f);
  __shared__ __half grad_sum;

  for (int tid = threadIdx.x; tid < n; tid += blockDim.x) {
    int idx = offset + tid;
    grad_softmax += static_cast<float>(top[idx] * softmax[idx]);
  }

  float tmp = blockReduceSum<float>(grad_softmax);
  if (threadIdx.x == 0) {
    grad_sum = static_cast<__half>(tmp);
  }

  __syncthreads();
  for (int tid = threadIdx.x; tid < n; tid += blockDim.x) {
    int idx = offset + tid;
    __half tmp = __hsub(top[idx], grad_sum);
    bottom[idx] = __hmul(bottom[idx], tmp);
    bottom[idx] = __hdiv(bottom[idx], scalar);
  }
}

template <typename T>
void mask_softmax_bprop(T* top, T* bottom, T* softmax, int m, int n, float scalar,
                        hipStream_t stream) {
  dim3 grid(m);
  dim3 block(min(n, 1024));
  mask_softmax_bprop_kernel<<<grid, block, 0, stream>>>(top, bottom, softmax, m, n, scalar);
#ifndef NDEBUG
  hipDeviceSynchronize();
  HCTR_LIB_THROW(hipGetLastError());
#endif
}

template <typename T>
void MaskedSoftmaxLayer<T>::fprop(bool is_train) {
  CudaDeviceContext context(get_device_id());
  Tensor2<T>& in_tensor = in_tensors_[0];
  Tensor2<T>& mask_tensor = in_tensors_[1];
  Tensor2<T>& out_tensor = out_tensors_[0];
  const auto& in_tensor_dim = in_tensor.get_dimensions();

  mask_softmax_fprop(out_tensor.get_ptr(), in_tensor.get_ptr(), mask_tensor.get_ptr(),
                     in_tensor_dim[0], in_tensor_dim[1], in_tensor_dim[2], scalar_,
                     get_gpu().get_stream());
  HCTR_LIB_THROW(hipMemcpyAsync((void*)softmax_out_.get_ptr(), (void*)out_tensor.get_ptr(),
                                 out_tensor.get_size_in_bytes(), hipMemcpyDeviceToDevice,
                                 get_gpu().get_stream()));
#ifndef NDEBUG
  hipDeviceSynchronize();
  HCTR_LIB_THROW(hipGetLastError());
#endif
}

template <>
void MaskedSoftmaxLayer<__half>::fprop(bool is_train) {
  CudaDeviceContext context(get_device_id());
  Tensor2<__half>& in_tensor = in_tensors_[0];
  Tensor2<__half>& mask_tensor = in_tensors_[1];
  Tensor2<__half>& out_tensor = out_tensors_[0];
  const auto& in_tensor_dim = in_tensor.get_dimensions();
  mask_softmax_fprop(out_tensor.get_ptr(), in_tensor.get_ptr(), mask_tensor.get_ptr(),
                     in_tensor_dim[0], in_tensor_dim[1], in_tensor_dim[2], scalar_,
                     get_gpu().get_stream());
  HCTR_LIB_THROW(hipMemcpyAsync((void*)softmax_out_.get_ptr(), (void*)out_tensor.get_ptr(),
                                 out_tensor.get_size_in_bytes(), hipMemcpyDeviceToDevice,
                                 get_gpu().get_stream()));
#ifndef NDEBUG
  hipDeviceSynchronize();
  HCTR_LIB_THROW(hipGetLastError());
#endif
}

template <typename T>
void MaskedSoftmaxLayer<T>::bprop() {
  CudaDeviceContext context(get_device_id());
  Tensor2<T>& bottom_tensor = in_tensors_[0];
  Tensor2<T>& top_tensor = out_tensors_[0];
  const auto& in_tensor_dim = bottom_tensor.get_dimensions();
  int hidden_size = in_tensor_dim[in_tensor_dim.size() - 1];
  int batch = bottom_tensor.get_num_elements() / hidden_size;

  mask_softmax_bprop(top_tensor.get_ptr(), bottom_tensor.get_ptr(), softmax_out_.get_ptr(), batch,
                     hidden_size, scalar_, get_gpu().get_stream());

#ifndef NDEBUG
  hipDeviceSynchronize();
  HCTR_LIB_THROW(hipGetLastError());
#endif
}

template <>
void MaskedSoftmaxLayer<__half>::bprop() {
  CudaDeviceContext context(get_device_id());
  Tensor2<__half>& bottom_tensor = in_tensors_[0];
  Tensor2<__half>& top_tensor = out_tensors_[0];
  const auto& in_tensor_dim = bottom_tensor.get_dimensions();

  int hidden_size = in_tensor_dim[in_tensor_dim.size() - 1];
  int n_rows = bottom_tensor.get_num_elements() / hidden_size;

  mask_softmax_bprop(top_tensor.get_ptr(), bottom_tensor.get_ptr(), softmax_out_.get_ptr(), n_rows,
                     hidden_size, scalar_, get_gpu().get_stream());

#ifndef NDEBUG
  hipDeviceSynchronize();
  HCTR_LIB_THROW(hipGetLastError());
#endif
}

template class MaskedSoftmaxLayer<float>;
template class MaskedSoftmaxLayer<__half>;

}  // namespace HugeCTR
