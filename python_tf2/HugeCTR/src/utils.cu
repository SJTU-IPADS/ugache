#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <utils.cuh>
#include <utils.hpp>

namespace HugeCTR {

#ifndef NUMA_NODE_MAP
#define NUMA_NODE_MAP
std::unordered_map<int, int> CudaCPUDeviceContext::device_id_to_numa_node_;
#endif

template <typename TIN, typename TOUT>
void convert_array_on_device(TOUT *out, const TIN *in, size_t num_elements,
                             const hipStream_t &stream) {
  if (num_elements > 0) {
    convert_array<<<(num_elements - 1) / 1024 + 1, 1024, 0, stream>>>(out, in, num_elements);
  }
}

template void convert_array_on_device<long long, int>(int *, const long long *, size_t,
                                                      const hipStream_t &);
template void convert_array_on_device<unsigned int, int>(int *, const unsigned int *, size_t,
                                                         const hipStream_t &);
template void convert_array_on_device<float, float>(float *, const float *, size_t,
                                                    const hipStream_t &);
template void convert_array_on_device<float, __half>(__half *, const float *, size_t,
                                                     const hipStream_t &);
template void convert_array_on_device<__half, float>(float *, const __half *, size_t,
                                                     const hipStream_t &);

template <typename TypeKey>
void data_to_unique_categories(TypeKey *value, const TypeKey *rowoffset,
                               const TypeKey *emmbedding_offsets, int num_tables,
                               int num_rowoffsets, const hipStream_t &stream) {
  constexpr size_t block_size = 256;
  size_t grid_size = (num_rowoffsets - 1) / block_size + 1;
  unique_key_kernels::data_to_unique_categories_kernel<<<grid_size, block_size, 0, stream>>>(
      value, rowoffset, emmbedding_offsets, num_tables, num_rowoffsets);
}

template void data_to_unique_categories<long long>(long long *, const long long *,
                                                   const long long *, int, int,
                                                   const hipStream_t &);

template void data_to_unique_categories<unsigned int>(unsigned int *, const unsigned int *,
                                                      const unsigned int *, int, int,
                                                      const hipStream_t &);

template <typename TypeKey>
void data_to_unique_categories(TypeKey *value, const TypeKey *emmbedding_offsets, int num_tables,
                               int nnz, const hipStream_t &stream) {
  constexpr size_t block_size = 256;
  size_t grid_size = std::min(4096ul, (nnz - 1) / block_size + 1);
  if (num_tables % 2 == 0 && sizeof(TypeKey) == 4) {
    unique_key_kernels::
        data_to_unique_categories_align2_kernel<<<grid_size, block_size, 0, stream>>>(
            value, emmbedding_offsets, num_tables / 2, nnz / 2);
  } else {
    unique_key_kernels::data_to_unique_categories_kernel<<<grid_size, block_size, 0, stream>>>(
        value, emmbedding_offsets, num_tables, nnz);
  }
}

template void data_to_unique_categories<long long>(long long *, const long long *, int, int,
                                                   const hipStream_t &);

template void data_to_unique_categories<unsigned int>(unsigned int *, const unsigned int *, int,
                                                      int, const hipStream_t &);

template <typename T>
__global__ void inc_var_cuda(T *x) {
  if (blockIdx.x == 0 and threadIdx.x == 0) {
    (*x)++;
  }
}

template <typename T>
void inc_var(volatile T *x, hipStream_t stream) {
  inc_var_cuda<<<1, 32, 0, stream>>>(x);
}

template void inc_var<size_t>(volatile size_t *x, hipStream_t stream);

}  // namespace HugeCTR
