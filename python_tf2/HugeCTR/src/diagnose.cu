#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <diagnose.hpp>
#include <fstream>
#include <limits>
#include <utils.cuh>

namespace HugeCTR {

namespace diagnose {

__device__ float atomicMin(float* address, float val) {
  float old = val;
  do {
    val = old;
    old = atomicExch(address, val);
  } while (old < val);
  return old;
}

__device__ float atomicMax(float* address, float val) {
  float old = val;
  do {
    val = old;
    old = atomicExch(address, val);
  } while (old > val);
  return old;
}

template <typename T>
__global__ void histogram_kernel(const T* arr, size_t len, float* range) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < len; i += blockDim.x * gridDim.x) {
    float val = TypeConvertFunc<float, T>::convert(arr[i]);
    if (val <= 0) {
      atomicMin(range + 0, val);
      atomicMax(range + 1, val);
    }
    if (val >= 0) {
      atomicMin(range + 2, val);
      atomicMax(range + 3, val);
    }
  }
}

template <typename T>
__global__ void verify_kernel(const T* arr, size_t len, int* flag);

template <>
__global__ void verify_kernel<float>(const float* arr, size_t len, int* flag) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < len; i += blockDim.x * gridDim.x) {
    if (isnan(arr[i])) atomicAdd(flag, 1);
  }
}

template <>
__global__ void verify_kernel(const __half* arr, size_t len, int* flag) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < len; i += blockDim.x * gridDim.x) {
    if (__hisnan(arr[i])) {
      atomicAdd(flag, 1);
    }
  }
}

template <typename T>
__global__ void sample_kernel(const T* arr, int len, float* arr_sample, int stride,
                              int max_sample_len) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < len; i += blockDim.x * gridDim.x) {
    if (i % stride == 0) {
      int j = i / stride;
      if (j < max_sample_len) {
        arr_sample[j] = TypeConvertFunc<float, T>::convert(arr[i]);
      }
    }
  }
}

template <typename T>
void verify_and_histogram(const char* category, const Tensor2<T>& tensor,
                          const hipStream_t& stream) {
  float h_array[4]{0.0f, -std::numeric_limits<float>::infinity(),
                   std::numeric_limits<float>::infinity(), 0.0f};
  int h_flag;
  float* d_array;
  int* d_flag;
  HCTR_LIB_THROW(hipMalloc(&d_array, sizeof(h_array)));
  HCTR_LIB_THROW(hipMalloc(&d_flag, sizeof(int)));
  HCTR_LIB_THROW(
      hipMemcpyAsync(d_array, h_array, sizeof(h_array), hipMemcpyHostToDevice, stream));
  HCTR_LIB_THROW(hipMemsetAsync(d_flag, 0, sizeof(int), stream));
  histogram_kernel<<<160, 1024, 0, stream>>>(tensor.get_ptr(), tensor.get_num_elements(), d_array);
  verify_kernel<<<160, 1024, 0, stream>>>(tensor.get_ptr(), tensor.get_num_elements(), d_flag);
  HCTR_LIB_THROW(
      hipMemcpyAsync(h_array, d_array, sizeof(h_array), hipMemcpyDeviceToHost, stream));
  HCTR_LIB_THROW(hipMemcpyAsync(&h_flag, d_flag, sizeof(int), hipMemcpyDeviceToHost, stream));
  HCTR_LIB_THROW(hipStreamSynchronize(stream));

  HCTR_LOG_S(INFO, ROOT) << "Diagnose for (" << category << "), Histogram [" << h_array[0] << ", "
                         << h_array[1] << "]"
                         << ", [" << h_array[2] << ", " << h_array[3] << "]" << std::endl;

  if (h_flag != 0) {
    std::ostringstream os;
    os << "Nan assert for " << category << " failed(" << h_flag << ").";
    HCTR_OWN_THROW(Error_t::DataCheckError, os.str());
  }
  HCTR_LIB_THROW(hipFree(d_array));
  HCTR_LIB_THROW(hipFree(d_flag));
}

template <typename T>
void sample_and_print(const char* category, const Tensor2<T>& tensor, size_t sample_count,
                      const hipStream_t& stream) {
  if (sample_count == 0) return;

  std::unique_ptr<float[]> h_array(new float[sample_count]);

  float* d_array;
  HCTR_LIB_THROW(hipMalloc(&d_array, sample_count * sizeof(float)));
  HCTR_LIB_THROW(hipMemsetAsync(d_array, 0, sample_count * sizeof(float), stream));
  sample_kernel<<<160, 1024, 0, stream>>>(tensor.get_ptr(), tensor.get_num_elements(), d_array,
                                          tensor.get_num_elements() / sample_count, sample_count);
  HCTR_LIB_THROW(hipMemcpyAsync(h_array.get(), d_array, sample_count * sizeof(float),
                                 hipMemcpyDeviceToHost, stream));
  HCTR_LIB_THROW(hipStreamSynchronize(stream));

  {
    auto log = HCTR_LOG_S(INFO, ROOT);
    log << "Diagnose for (" << category << "), Sampling [";
    for (size_t i = 0; i < min(sample_count, tensor.get_num_elements()); i++) {
      if (i != 0) log << ",";
      log << h_array[i];
    }
    log << "]" << std::endl;
  }

  HCTR_LIB_THROW(hipFree(d_array));
}

template <typename T>
void sample_and_print(const char* category, const Tensor2<T>& tensor, int begin, int end,
                      const hipStream_t& stream) {
  if (begin >= 0 && end <= static_cast<int>(tensor.get_num_elements()) && end > begin) {
  } else if (end < 0 && begin >= -static_cast<int>(tensor.get_num_elements()) && end > begin) {
    begin += tensor.get_num_elements();
    end += tensor.get_num_elements();
  } else {
    return;
  }

  std::unique_ptr<T[]> h_array(new T[end - begin]);
  HCTR_LIB_THROW(hipMemcpyAsync(h_array.get(), tensor.get_ptr() + begin,
                                 (begin - end) * sizeof(float), hipMemcpyDeviceToHost, stream));
  HCTR_LIB_THROW(hipStreamSynchronize(stream));

  {
    auto log = HCTR_LOG_S(INFO, ROOT);
    log << "Diagnose for (" << category << "), Sampling [";
    for (size_t i = 0; i < end - begin; i++) {
      if (i != 0) log << ",";
      log << h_array[i];
    }
    log << "]" << std::endl;
  }
}

template <typename T>
void dump(const char* filename, const Tensor2<T>& tensor, const hipStream_t& stream) {
  std::unique_ptr<T[]> h_array(new T[tensor.get_num_elements()]);
  HCTR_LIB_THROW(hipMemcpyAsync(h_array.get(), tensor.get_ptr(), tensor.get_size_in_bytes(),
                                 hipMemcpyDeviceToHost, stream));
  HCTR_LIB_THROW(hipStreamSynchronize(stream));

  std::ofstream s(filename, std::ios::out | std::ios::binary);
  s.write(reinterpret_cast<const char*>(h_array.get()), tensor.get_size_in_bytes());
  s.close();
}

template void verify_and_histogram<float>(const char* category, const Tensor2<float>& tensor,
                                          const hipStream_t& stream);

template void dump<unsigned int>(const char* filename, const Tensor2<unsigned int>& tensor,
                                 const hipStream_t& stream);
template void dump<unsigned long>(const char* filename, const Tensor2<unsigned long>& tensor,
                                  const hipStream_t& stream);
template void dump<long long>(const char* filename, const Tensor2<long long>& tensor,
                              const hipStream_t& stream);
template void dump<float>(const char* filename, const Tensor2<float>& tensor,
                          const hipStream_t& stream);
template void dump<__half>(const char* filename, const Tensor2<__half>& tensor,
                           const hipStream_t& stream);

}  // namespace diagnose

}  // namespace HugeCTR
