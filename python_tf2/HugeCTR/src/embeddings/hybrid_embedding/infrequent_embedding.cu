#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_runtime.h>

#include <algorithm>
#include <hipcub/hipcub.hpp>
#include <iostream>
#include <utility>
#include <vector>

#include "HugeCTR/include/common.hpp"
#include "HugeCTR/include/data_simulator.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/data.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/infrequent_embedding.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/model.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/update.cuh"
#include "HugeCTR/include/embeddings/hybrid_embedding/utils.cuh"
#include "HugeCTR/include/embeddings/hybrid_embedding/utils.hpp"
#include "HugeCTR/include/shuffle/shuffle.cuh"
#include "HugeCTR/include/tensor2.hpp"
#include "HugeCTR/include/utils.hpp"

namespace HugeCTR {

namespace hybrid_embedding {

namespace infrequent_embedding_kernels {

template <typename dtype, typename emtype>
__global__ void hier_update_model(InfrequentEmbeddingSelectionView<dtype>* indices,
                                  const dtype* __restrict__ category_location,
                                  const emtype* __restrict__ gradients,
                                  float* __restrict__ embedding_vectors,
                                  uint32_t embedding_vec_size, uint32_t num_instances,
                                  uint32_t local_samples_size, uint32_t local_comm_buff_size,
                                  const float* __restrict__ lr_ptr, const float scale) {
  float lr = __ldg(lr_ptr) / scale;
  const uint32_t num_indices = indices->model_indices_offsets[num_instances];

  // Load offset only when the network_id changes
  uint32_t previous_network_id = 0;
  uint32_t offset = 0;

  for (uint32_t i = blockIdx.x; i < num_indices; i += gridDim.x) {
    uint32_t index = indices->model_indices[i];
    dtype category = indices->samples[index];
    dtype location = category_location[2 * category + 1];
    uint32_t network_id = index / local_samples_size;
    if (network_id != previous_network_id) {
      offset = indices->model_indices_offsets[network_id];
      previous_network_id = network_id;
    }
    atomicAdd(
        embedding_vectors + location * embedding_vec_size + threadIdx.x,
        -lr * TypeConvertFunc<float, emtype>::convert(
                  gradients[embedding_vec_size * (network_id * local_comm_buff_size + i - offset) +
                            threadIdx.x]));
  }
}

template <typename dtype, typename emtype>
__global__ void infrequent_update_model_direct(
    const emtype* const* __restrict__ gradients_pointers, float* embedding_vectors,
    InfrequentEmbeddingSelectionView<dtype>* indices, const dtype* __restrict__ category_location,
    uint32_t num_instances, uint32_t model_id, uint32_t embedding_vec_size,
    uint32_t local_samples_size, const float* __restrict__ lr_ptr, const float scale) {
  float lr = __ldg(lr_ptr) / scale;
  // Shift pattern
  const uint32_t offset = indices->model_indices_offsets[model_id + 1];
  const uint32_t num_model_indices = indices->model_indices_offsets[num_instances];

  for (uint32_t i = blockIdx.x; i < num_model_indices; i += gridDim.x) {
    uint32_t vid = (i + offset) % num_model_indices;

    uint32_t index = indices->model_indices[vid];
    uint32_t network_id = index / local_samples_size;
    uint32_t local_index = index % local_samples_size;
    dtype category = indices->samples[index];
    uint32_t location = category_location[2 * category + 1];

    const emtype* gradients = gradients_pointers[network_id];

    atomicAdd(embedding_vectors + location * embedding_vec_size + threadIdx.x,
              -lr * TypeConvertFunc<float, emtype>::convert(
                        gradients[local_index * embedding_vec_size + threadIdx.x]));
  }
}

// template <typename dtype>
// __global__ void calculate_network_indices_mask(const dtype* __restrict__ local_samples,
//                                                const dtype* __restrict__ category_location,
//                                                bool* mask, uint32_t local_samples_size,
//                                                uint32_t num_instances) {
//   for (uint32_t i = blockIdx.x * blockDim.x + threadIdx.x; i < local_samples_size;
//        i += gridDim.x * blockDim.x) {
//     dtype category = local_samples[i];
//     uint32_t model_id = static_cast<uint32_t>(category_location[2 * category]);
//     for (uint32_t section_id = 0; section_id < num_instances; section_id++) {
//       mask[local_samples_size * section_id + i] = (model_id == section_id);
//     }
//   }
// }

template <typename LambdaPtr>
static __global__ void offsets_to_sizes(size_t* sizes, LambdaPtr get_offsets_ptr,
                                        size_t element_size, uint32_t num_instances) {
  uint32_t* offsets = get_offsets_ptr();
  for (int t = blockIdx.x * blockDim.x + threadIdx.x; t < num_instances;
       t += gridDim.x * blockDim.x) {
    sizes[t] = (offsets[t + 1] - offsets[t]) * element_size;
  }
}

}  // namespace infrequent_embedding_kernels

template <typename dtype>
InfrequentEmbeddingBase<dtype>::InfrequentEmbeddingBase() {}

template <typename dtype>
InfrequentEmbeddingBase<dtype>::~InfrequentEmbeddingBase() {}

template <typename dtype>
InfrequentEmbeddingBase<dtype>::InfrequentEmbeddingBase(const InfrequentEmbeddingBase& other) {
  HCTR_LIB_THROW(hipMalloc(&indices_view_, sizeof(*indices_view_)));

  HCTR_LIB_THROW(hipMemcpy(indices_view_, other.indices_view_, sizeof(*indices_view_),
                            hipMemcpyDeviceToDevice));
}

template <typename dtype>
void InfrequentEmbeddingBase<dtype>::set_current_indices(
    InfrequentEmbeddingSelection<dtype>* indices) {
  indices_ = indices;
  data_ = indices->get_data();
  indices_view_ = indices->get_device_view();
}

template <typename dtype, typename emtype>
InfrequentEmbedding_NVLink_SingleNode<dtype, emtype>::InfrequentEmbedding_NVLink_SingleNode(
    Model<dtype>& model, GPUResource& gpu_resource, size_t embedding_vec_size)
    : model_(model), gpu_resource_(gpu_resource), embedding_vec_size_(embedding_vec_size) {
  auto buf = GeneralBuffer2<CudaAllocator>::create();
  buf->reserve({ceildiv<size_t>(model.num_categories, model.num_instances), embedding_vec_size_},
               &infrequent_embedding_vectors_);
  buf->reserve({model.num_instances, 1}, &interaction_layer_input_pointers_train_);
  buf->reserve({model.num_instances, 1}, &interaction_layer_input_pointers_eval_);
  buf->reserve({model.num_instances, 1}, &gradients_pointers_);
  buf->allocate();
}

template <typename dtype, typename emtype>
void InfrequentEmbedding_NVLink_SingleNode<dtype, emtype>::init_pointers(
    int local_gpu_count, const hipStream_t stream,
    std::vector<emtype*>& interaction_layer_input_pointers_train,
    std::vector<emtype*>& interaction_layer_input_pointers_eval,
    std::vector<const emtype*>& gradients_pointers) {
  HCTR_LIB_THROW(hipMemcpyAsync(interaction_layer_input_pointers_train_.get_ptr(),
                                 interaction_layer_input_pointers_train.data(),
                                 local_gpu_count * sizeof(emtype*), hipMemcpyHostToDevice,
                                 stream));
  HCTR_LIB_THROW(hipMemcpyAsync(interaction_layer_input_pointers_eval_.get_ptr(),
                                 interaction_layer_input_pointers_eval.data(),
                                 local_gpu_count * sizeof(emtype*), hipMemcpyHostToDevice,
                                 stream));
  HCTR_LIB_THROW(hipMemcpyAsync(gradients_pointers_.get_ptr(), gradients_pointers.data(),
                                 local_gpu_count * sizeof(emtype*), hipMemcpyHostToDevice,
                                 stream));
}

/** Forward network for single GPU (no communications) */
template <typename dtype, typename emtype>
void InfrequentEmbedding_NVLink_SingleNode<dtype, emtype>::forward_network_direct(
    bool is_train, hipStream_t stream) {
  const uint32_t num_instances = model_.num_instances;
  const uint32_t model_id = model_.global_instance_id;
  uint32_t local_samples_size =
      ceildiv<uint32_t>(data_->batch_size, num_instances) * data_->table_sizes.size();

  auto interaction_layer_input_pointers = is_train
                                              ? interaction_layer_input_pointers_train_.get_ptr()
                                              : interaction_layer_input_pointers_eval_.get_ptr();
  auto indices = this->indices_view_;
  auto category_location = model_.category_location.get_ptr();
  auto model_table = infrequent_embedding_vectors_.get_ptr();
  auto embedding_vec_size = embedding_vec_size_;

  auto copy_desc = CopyDescriptors::make_OneToOne<float, emtype, 1>(
      embedding_vec_size,
      [=] __device__() { return indices->model_indices_offsets[num_instances]; },
      [=] __device__(size_t i) -> CopyDescriptors::CopyDetails<float, emtype, 1> {
        const uint32_t offset = indices->model_indices_offsets[model_id + 1];
        const uint32_t num_model_indices = indices->model_indices_offsets[num_instances];
        const uint32_t vid = (i + offset) % num_model_indices;
        const uint32_t index = indices->model_indices[vid];

        const dtype category = indices->samples[index];
        const dtype location = category_location[2 * category + 1];

        const uint32_t network_id = index / local_samples_size;
        const uint32_t local_index = index % local_samples_size;

        emtype* interaction_layer_input = interaction_layer_input_pointers[network_id];

        return {model_table + location * embedding_vec_size,
                {interaction_layer_input + local_index * embedding_vec_size},
                {true}};
      });

  shuffle(copy_desc, stream, local_samples_size / 10);
  HCTR_LIB_THROW(hipPeekAtLastError());
}

template <typename dtype, typename emtype>
void InfrequentEmbedding_NVLink_SingleNode<dtype, emtype>::update_model_direct(
    float* dev_lr, float scale, hipStream_t stream) {
  const uint32_t& num_instances = model_.num_instances;
  uint32_t local_samples_size =
      ceildiv<uint32_t>(data_->batch_size, num_instances) * data_->table_sizes.size();

  int num_sm = gpu_resource_.get_sm_count();
  int n_blocks = 8 * num_sm;  // TODO: better heuristics

  /* Each model reads from the gradients of each network */
  infrequent_embedding_kernels::
      infrequent_update_model_direct<<<n_blocks, embedding_vec_size_, 0, stream>>>(
          gradients_pointers_.get_ptr(), infrequent_embedding_vectors_.get_ptr(),
          this->indices_view_, model_.category_location.get_ptr(), model_.num_instances,
          model_.global_instance_id, embedding_vec_size_, local_samples_size, dev_lr, scale);
  HCTR_LIB_THROW(hipPeekAtLastError());
}

template <typename dtype, typename emtype>
InfrequentEmbedding_IB_NVLINK<dtype, emtype>::InfrequentEmbedding_IB_NVLINK(
    Model<dtype>& model, GPUResource& gpu_resource, size_t embedding_vec_size)
    : model_(model), gpu_resource_(gpu_resource), embedding_vec_size_(embedding_vec_size) {
  auto buf = GeneralBuffer2<CudaAllocator>::create();

  buf->reserve({ceildiv<size_t>(model.num_categories, model.num_instances), embedding_vec_size_},
               &infrequent_embedding_vectors_);
  buf->allocate();

  auto managed_buf = GeneralBuffer2<CudaManagedAllocator>::create();
  managed_buf->reserve({model.num_instances + 1, 1}, &model_indices_offsets_);
  managed_buf->reserve({model.num_instances + 1, 1}, &network_indices_offsets_);
  managed_buf->allocate();
  // int current_device;
  // HCTR_LIB_THROW(hipGetDevice(&current_device));
  // HCTR_LIB_THROW(hipMemAdvise(managed_buf->get_ptr(), managed_buf->get_size_in_bytes(),
  // hipMemAdviseSetReadMostly, current_device));
}

template <typename dtype, typename emtype>
void InfrequentEmbedding_IB_NVLINK<dtype, emtype>::init_comms(size_t embedding_vec_size,
                                                              const GPUResource* gpu_resource,
                                                              GeneralBuffer2<CudaAllocator>* i_buf,
                                                              size_t max_buf_size) {
  infrequent_forward_comm_buffers_ = std::make_unique<AllToAllStorage<emtype>>(i_buf, max_buf_size);
  infrequent_backward_comm_buffers_ =
      std::make_unique<AllToAllStorage<emtype>>(i_buf, max_buf_size);
  infrequent_forward_comms_ = std::make_unique<AllToAll_Multi_NCCL<emtype>>(
      infrequent_forward_comm_buffers_->send_buffer, infrequent_forward_comm_buffers_->recv_buffer,
      get_model_indices_offsets_ptr(), get_network_indices_offsets_ptr(), gpu_resource,
      embedding_vec_size);
  infrequent_backward_comms_ = std::make_unique<AllToAll_Multi_NCCL<emtype>>(
      infrequent_backward_comm_buffers_->send_buffer,
      infrequent_backward_comm_buffers_->recv_buffer, get_network_indices_offsets_ptr(),
      get_model_indices_offsets_ptr(), gpu_resource, embedding_vec_size);
}

template <typename dtype, typename emtype>
void InfrequentEmbedding_IB_NVLINK<dtype, emtype>::forward_model(emtype* message_buffer,
                                                                 hipStream_t stream) {
  HCTR_LIB_THROW(hipMemcpyAsync(
      model_indices_offsets_.get_ptr(), this->indices_->model_indices_offsets_.get_ptr(),
      model_indices_offsets_.get_size_in_bytes(), hipMemcpyDeviceToDevice, stream));

  HCTR_LIB_THROW(hipMemcpyAsync(
      network_indices_offsets_.get_ptr(), this->indices_->network_indices_offsets_.get_ptr(),
      network_indices_offsets_.get_size_in_bytes(), hipMemcpyDeviceToDevice, stream));

  HCTR_LIB_THROW(hipStreamSynchronize(stream));

  auto indices = this->indices_view_;
  auto category_location = model_.category_location.get_ptr();
  auto infrequent_embedding_vectors = infrequent_embedding_vectors_.get_ptr();
  auto embedding_vec_size = embedding_vec_size_;
  auto num_instances = model_.num_instances;

  auto copy_desc = CopyDescriptors::make_OneToOne<float, emtype, 1>(
      embedding_vec_size,
      [=] __device__() { return indices->model_indices_offsets[num_instances]; },
      [=] __device__(size_t i) -> CopyDescriptors::CopyDetails<float, emtype, 1> {
        uint32_t index = indices->model_indices[i];
        dtype category = indices->samples[index];
        dtype location = category_location[2 * category + 1];

        return {infrequent_embedding_vectors + location * embedding_vec_size,
                {message_buffer + i * embedding_vec_size},
                {true}};
      });

  shuffle(copy_desc, stream, data_->samples.get_num_elements() / model_.num_instances / 8);
  HCTR_LIB_THROW(hipPeekAtLastError());
}

template <typename dtype, typename emtype>
void InfrequentEmbedding_IB_NVLINK<dtype, emtype>::forward_network(const emtype* message_buffer,
                                                                   emtype* output_ptr,
                                                                   hipStream_t stream) {
  auto indices = this->indices_view_;
  auto embedding_vec_size = embedding_vec_size_;
  auto num_instances = model_.num_instances;

  auto copy_desc = CopyDescriptors::make_OneToOne<emtype, emtype, 1>(
      embedding_vec_size,
      [=] __device__() { return indices->network_indices_offsets[num_instances]; },
      [=] __device__(size_t i) -> CopyDescriptors::CopyDetails<emtype, emtype, 1> {
        uint32_t index = indices->network_indices[i];
        return {message_buffer + i * embedding_vec_size,
                {output_ptr + index * embedding_vec_size},
                {true}};
      });

  shuffle(copy_desc, stream, data_->samples.get_num_elements() / model_.num_instances / 8);
  HCTR_LIB_THROW(hipPeekAtLastError());
}

template <typename dtype, typename emtype>
void InfrequentEmbedding_IB_NVLINK<dtype, emtype>::update_network(const emtype* gradients,
                                                                  emtype* message_buffer,
                                                                  hipStream_t stream) {
  auto indices = this->indices_view_;
  auto embedding_vec_size = embedding_vec_size_;
  auto num_instances = model_.num_instances;

  auto copy_desc = CopyDescriptors::make_OneToOne<emtype, emtype, 1>(
      embedding_vec_size,
      [=] __device__() { return indices->network_indices_offsets[num_instances]; },
      [=] __device__(size_t i) -> CopyDescriptors::CopyDetails<emtype, emtype, 1> {
        uint32_t index = indices->network_indices[i];

        return {gradients + index * embedding_vec_size,
                {message_buffer + i * embedding_vec_size},
                {true}};
      });

  shuffle(copy_desc, stream, data_->samples.get_num_elements() / model_.num_instances / 8);
  HCTR_LIB_THROW(hipPeekAtLastError());
}

template <typename dtype, typename emtype>
void InfrequentEmbedding_IB_NVLINK<dtype, emtype>::update_model(const emtype* message_buffer,
                                                                float* dev_lr, float scale,
                                                                hipStream_t stream) {
  auto indices = this->indices_view_;
  const dtype* __restrict__ category_location = model_.category_location.get_ptr();
  auto num_instances = model_.num_instances;

  uint32_t n_blocks = gpu_resource_.get_sm_count();

  sgd_atomic_update(
      message_buffer, infrequent_embedding_vectors_.get_ptr(),
      [indices, num_instances] __device__() {
        return indices->model_indices_offsets[num_instances];
      },
      [indices, category_location] __device__(uint32_t i) {
        uint32_t index = indices->model_indices[i];
        dtype category = indices->samples[index];
        return category_location[2 * category + 1];
      },
      n_blocks, embedding_vec_size_, dev_lr, scale, stream);
}

template <typename dtype, typename emtype>
InfrequentEmbedding_IB_NVLink_Hier<dtype, emtype>::InfrequentEmbedding_IB_NVLink_Hier(
    Model<dtype>& model, GPUResource& gpu_resource, size_t embedding_vec_size)
    : model_(model), gpu_resource_(gpu_resource), embedding_vec_size_(embedding_vec_size) {
  auto buf = GeneralBuffer2<CudaAllocator>::create();
  buf->reserve({ceildiv<size_t>(model.num_categories, model.num_instances), embedding_vec_size_},
               &infrequent_embedding_vectors_);
  buf->reserve({model_.num_instances}, &model_indices_sizes_);
  buf->reserve({model_.num_instances}, &model_indices_sizes_ptrs_);
  buf->reserve({model_.num_instances}, &network_indices_sizes_);
  buf->reserve({model_.num_instances}, &network_indices_sizes_ptrs_);
  buf->allocate();
}

template <typename dtype, typename emtype>
void InfrequentEmbedding_IB_NVLink_Hier<dtype, emtype>::init_comms(
    int64_t max_num_infrequent_samples, size_t slot_num, size_t embedding_vec_size,
    GeneralBuffer2<CudaAllocator>* buf_ptr, size_t batch_size_true, size_t batch_size_false,
    size_t local_gpu_count) {
  double p_infrequent_samples = 1.0;
  if (max_num_infrequent_samples >= 0) {
    p_infrequent_samples =
        (double)max_num_infrequent_samples / ((double)batch_size_true * slot_num);
  }
  auto align = [this](size_t val) {
    auto alignment = model_.num_instances;
    return ((val + alignment - 1) / alignment) * alignment;
  };

  max_num_infrequent_per_batch_ =
      align(std::max(batch_size_true, batch_size_false) * slot_num * p_infrequent_samples);

  max_num_infrequent_per_train_batch_ = align(batch_size_true * slot_num * p_infrequent_samples);

  size_t max_buf_size = embedding_vec_size * max_num_infrequent_per_batch_;
  size_t max_back_buf_size = embedding_vec_size * max_num_infrequent_per_train_batch_;

  HCTR_LOG_S(INFO, ROOT) << "Allocating A2A buffers for infrequent categories. For training : "
                         << max_num_infrequent_per_train_batch_
                         << ", for evaluation:  " << max_num_infrequent_per_batch_ << std::endl;

  infrequent_backward_comm_buffers_ =
      std::make_unique<AllToAllStorage<emtype>>(buf_ptr, max_back_buf_size);
  infrequent_forward_comm_buffers_ =
      std::make_unique<AllToAllStorage<emtype>>(buf_ptr, max_buf_size);
  // TODO: need to check the correctness
  buf_ptr->reserve({local_gpu_count}, &infrequent_forward_comm_buffers_->send_buffer_ptrs);
  buf_ptr->reserve({local_gpu_count}, &infrequent_backward_comm_buffers_->send_buffer_ptrs);
}

template <typename dtype, typename emtype>
void InfrequentEmbedding_IB_NVLink_Hier<dtype, emtype>::fused_intra_forward_model(
    emtype** message_buffer, hipStream_t stream) {
  auto indices = this->indices_view_;
  auto category_location = model_.category_location.get_ptr();
  auto infrequent_embedding_vectors = infrequent_embedding_vectors_.get_ptr();
  size_t embedding_vec_size = embedding_vec_size_;
  auto local_instance_id = model_.instance_id;
  auto num_instances = model_.num_instances;
  auto per_node_instances = num_instances / model_.h_num_instances_per_node.size();
  uint32_t local_samples_size =
      ceildiv<uint32_t>(data_->batch_size, num_instances) * data_->table_sizes.size();

  uint32_t local_comm_buff_size =
      ceildiv<uint32_t>(max_num_infrequent_per_batch_, model_.num_instances);

  auto copy_desc = CopyDescriptors::make_OneToOne<float, emtype, 1>(
      embedding_vec_size,
      [=] __device__() { return indices->model_indices_offsets[num_instances]; },
      [=] __device__(size_t i) -> CopyDescriptors::CopyDetails<float, emtype, 1> {
        uint32_t num_selected = indices->model_indices_offsets[num_instances];
        uint32_t vid =
            (i + indices->model_indices_offsets[(local_instance_id + 1) % per_node_instances]) %
            num_selected;
        uint32_t index = indices->model_indices[vid];
        uint32_t network_id = (index / local_samples_size);
        dtype category = indices->samples[index];
        dtype location = category_location[2 * category + 1];
        uint32_t local_network_id = (network_id % per_node_instances);
        emtype* output_ptr =
            &message_buffer[local_network_id][(network_id - local_network_id + local_instance_id) *
                                              local_comm_buff_size * embedding_vec_size];

        return {
            infrequent_embedding_vectors + location * embedding_vec_size,
            {output_ptr + (vid - indices->model_indices_offsets[network_id]) * embedding_vec_size},
            {true}};
      });

  shuffle(copy_desc, stream, data_->samples.get_num_elements() / model_.num_instances / 8);
  HCTR_LIB_THROW(hipPeekAtLastError());
}

template <typename dtype, typename emtype>
void InfrequentEmbedding_IB_NVLink_Hier<dtype, emtype>::hier_forward_network(
    const emtype* message_buffer, emtype* output_ptr, hipStream_t stream) {
  auto indices = this->indices_view_;
  auto embedding_vec_size = embedding_vec_size_;
  auto num_instances = model_.num_instances;
  uint32_t local_samples_size =
      ceildiv<uint32_t>(data_->batch_size, model_.num_instances) * data_->table_sizes.size();
  uint32_t local_comm_buff_size =
      ceildiv<uint32_t>(max_num_infrequent_per_batch_, model_.num_instances);

  auto copy_desc = CopyDescriptors::make_OneToOne<emtype, emtype, 1>(
      embedding_vec_size,
      [=] __device__() { return indices->network_indices_offsets[num_instances]; },
      [=] __device__(size_t i) -> CopyDescriptors::CopyDetails<emtype, emtype, 1> {
        uint32_t index = indices->network_indices[i];
        uint32_t model_id = indices->network_indices_src_model_id[i];
        uint32_t offset = indices->network_indices_offsets[model_id];

        return {
            message_buffer + (model_id * local_comm_buff_size + i - offset) * embedding_vec_size,
            {output_ptr + index * embedding_vec_size},
            {true}};
      });

  shuffle(copy_desc, stream, data_->samples.get_num_elements() / model_.num_instances / 8);
  HCTR_LIB_THROW(hipPeekAtLastError());
}

template <typename dtype, typename emtype>
void InfrequentEmbedding_IB_NVLink_Hier<dtype, emtype>::fused_intra_update_network(
    const emtype* gradients, emtype** message_buffer, hipStream_t stream) {
  auto indices = this->indices_view_;
  size_t embedding_vec_size = embedding_vec_size_;
  auto local_instance_id = model_.instance_id;
  auto num_instances = model_.num_instances;
  auto per_node_instances = num_instances / model_.h_num_instances_per_node.size();
  uint32_t local_comm_buff_size =
      ceildiv<uint32_t>(max_num_infrequent_per_train_batch_, model_.num_instances);

  auto copy_desc = CopyDescriptors::make_OneToOne<emtype, emtype, 1>(
      embedding_vec_size,
      [=] __device__() { return indices->network_indices_offsets[num_instances]; },
      [=] __device__(size_t i) -> CopyDescriptors::CopyDetails<emtype, emtype, 1> {
        uint32_t num_selected = indices->network_indices_offsets[num_instances];
        uint32_t vid =
            (i + indices->network_indices_offsets[(local_instance_id + 1) % per_node_instances]) %
            num_selected;
        uint32_t index = indices->network_indices[vid];

        uint32_t model_id = indices->network_indices_src_model_id[vid];

        uint32_t local_model_id = (model_id % per_node_instances);
        emtype* output_ptr =
            &message_buffer[local_model_id][(model_id - local_model_id + local_instance_id) *
                                            local_comm_buff_size * embedding_vec_size];

        return {
            gradients + index * embedding_vec_size,
            {output_ptr + (vid - indices->network_indices_offsets[model_id]) * embedding_vec_size},
            {true}};
      });

  shuffle(copy_desc, stream, data_->samples.get_num_elements() / model_.num_instances / 8);
  HCTR_LIB_THROW(hipPeekAtLastError());
}

template <typename dtype, typename emtype>
void InfrequentEmbedding_IB_NVLink_Hier<dtype, emtype>::hier_update_model(
    const emtype* message_buffer, float* dev_lr, float scale, hipStream_t stream) {
  const uint32_t& num_instances = model_.num_instances;
  uint32_t local_samples_size =
      ceildiv<uint32_t>(data_->batch_size, num_instances) * data_->table_sizes.size();
  uint32_t local_comm_buff_size =
      ceildiv<uint32_t>(max_num_infrequent_per_train_batch_, model_.num_instances);

  int num_sm = gpu_resource_.get_sm_count();
  int n_blocks = 16 * num_sm;  // TODO: better heuristics

  infrequent_embedding_kernels::hier_update_model<<<n_blocks, embedding_vec_size_, 0, stream>>>(
      this->indices_view_, model_.category_location.get_ptr(),
      // infrequent_backward_comm_buffers_.back().recv_buffer.get_ptr(),
      message_buffer, infrequent_embedding_vectors_.get_ptr(), embedding_vec_size_,
      model_.num_instances, local_samples_size, local_comm_buff_size, dev_lr, scale);
  HCTR_LIB_THROW(hipPeekAtLastError());
}

template <typename dtype, typename emtype>
void InfrequentEmbedding_IB_NVLink_Hier<dtype, emtype>::calculate_model_indices_sizes_from_offsets(
    hipStream_t stream) {
  auto indices = this->indices_view_;
  constexpr size_t TPB = 256;
  const size_t n_blocks = ceildiv<size_t>(model_.num_instances, TPB);
  infrequent_embedding_kernels::offsets_to_sizes<<<n_blocks, TPB, 0, stream>>>(
      model_indices_sizes_.get_ptr(), [=] __device__() { return indices->model_indices_offsets; },
      embedding_vec_size_ * sizeof(emtype), model_.num_instances);
}

template <typename dtype, typename emtype>
void InfrequentEmbedding_IB_NVLink_Hier<
    dtype, emtype>::calculate_network_indices_sizes_from_offsets(hipStream_t stream) {
  auto indices = this->indices_view_;
  constexpr size_t TPB = 256;
  const size_t n_blocks = ceildiv<size_t>(model_.num_instances, TPB);
  infrequent_embedding_kernels::offsets_to_sizes<<<n_blocks, TPB, 0, stream>>>(
      network_indices_sizes_.get_ptr(),
      [=] __device__() { return indices->network_indices_offsets; },
      embedding_vec_size_ * sizeof(emtype), model_.num_instances);
}

template <typename dtype, typename emtype>
void InfrequentEmbedding_NVLink_SingleNode<dtype, emtype>::initialize_embedding_vectors(
    const std::vector<size_t>& table_sizes) {
  CudaDeviceContext context(gpu_resource_.get_device_id());

  const size_t num_tables = table_sizes.size();
  for (size_t i = 0; i < num_tables; i++) {
    float up_bound = sqrt(1.f / table_sizes[i]);

    const size_t offset = embedding_vec_size_ * model_.h_infrequent_model_table_offsets[i];
    const size_t number_of_vectors =
        model_.h_infrequent_model_table_offsets[i + 1] - model_.h_infrequent_model_table_offsets[i];
    UniformGenerator::fill(
        infrequent_embedding_vectors_.get_ptr() + offset, embedding_vec_size_ * number_of_vectors,
        -up_bound, up_bound, gpu_resource_.get_sm_count(),
        gpu_resource_.get_replica_variant_curand_generator(), gpu_resource_.get_stream());
  }
}

template <typename dtype, typename emtype>
void InfrequentEmbedding_IB_NVLINK<dtype, emtype>::initialize_embedding_vectors(
    const std::vector<size_t>& table_sizes) {
  CudaDeviceContext context(gpu_resource_.get_device_id());

  const size_t num_tables = table_sizes.size();
  for (size_t i = 0; i < num_tables; i++) {
    float up_bound = sqrt(1.f / table_sizes[i]);

    const size_t offset = embedding_vec_size_ * model_.h_infrequent_model_table_offsets[i];
    const size_t number_of_vectors =
        model_.h_infrequent_model_table_offsets[i + 1] - model_.h_infrequent_model_table_offsets[i];
    UniformGenerator::fill(
        infrequent_embedding_vectors_.get_ptr() + offset, embedding_vec_size_ * number_of_vectors,
        -up_bound, up_bound, gpu_resource_.get_sm_count(),
        gpu_resource_.get_replica_variant_curand_generator(), gpu_resource_.get_stream());
  }
}

template <typename dtype, typename emtype>
void InfrequentEmbedding_IB_NVLink_Hier<dtype, emtype>::initialize_embedding_vectors(
    const std::vector<size_t>& table_sizes) {
  CudaDeviceContext context(gpu_resource_.get_device_id());

  const size_t num_tables = table_sizes.size();
  for (size_t i = 0; i < num_tables; i++) {
    float up_bound = sqrt(1.f / table_sizes[i]);

    const size_t offset = embedding_vec_size_ * model_.h_infrequent_model_table_offsets[i];
    const size_t number_of_vectors =
        model_.h_infrequent_model_table_offsets[i + 1] - model_.h_infrequent_model_table_offsets[i];
    UniformGenerator::fill(
        infrequent_embedding_vectors_.get_ptr() + offset, embedding_vec_size_ * number_of_vectors,
        -up_bound, up_bound, gpu_resource_.get_sm_count(),
        gpu_resource_.get_replica_variant_curand_generator(), gpu_resource_.get_stream());
  }
}

template class InfrequentEmbeddingBase<uint32_t>;
template class InfrequentEmbeddingBase<long long>;

// NVLink_SingleNode
template class InfrequentEmbedding_NVLink_SingleNode<uint32_t, __half>;
template class InfrequentEmbedding_NVLink_SingleNode<uint32_t, float>;
template class InfrequentEmbedding_NVLink_SingleNode<long long, __half>;
template class InfrequentEmbedding_NVLink_SingleNode<long long, float>;

// IB_NVLINK
template class InfrequentEmbedding_IB_NVLINK<uint32_t, __half>;
template class InfrequentEmbedding_IB_NVLINK<uint32_t, float>;
template class InfrequentEmbedding_IB_NVLINK<long long, __half>;
template class InfrequentEmbedding_IB_NVLINK<long long, float>;

// IB_NVLink_Hier
template class InfrequentEmbedding_IB_NVLink_Hier<uint32_t, __half>;
template class InfrequentEmbedding_IB_NVLink_Hier<uint32_t, float>;
template class InfrequentEmbedding_IB_NVLink_Hier<long long, __half>;
template class InfrequentEmbedding_IB_NVLink_Hier<long long, float>;

}  // namespace hybrid_embedding

}  // namespace HugeCTR
