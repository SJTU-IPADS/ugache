#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_runtime.h>

#include <algorithm>
#include <hipcub/hipcub.hpp>
#include <iostream>
#include <vector>

#include "HugeCTR/include/common.hpp"
#include "HugeCTR/include/data_simulator.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/data.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/frequent_embedding.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/model.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/update.cuh"
#include "HugeCTR/include/embeddings/hybrid_embedding/utils.cuh"
#include "HugeCTR/include/embeddings/hybrid_embedding/utils.hpp"
#include "HugeCTR/include/shuffle/shuffle.cuh"
#include "HugeCTR/include/tensor2.hpp"
#include "HugeCTR/include/utils.cuh"
#include "HugeCTR/include/utils.hpp"

namespace HugeCTR {

namespace hybrid_embedding {

namespace frequent_embedding_kernels {

template <typename dtype>
__global__ void reset_relevant_gradients(float* __restrict__ gradients, uint32_t embedding_vec_size,
                                         FrequentEmbeddingCompressionView<dtype>* indices,
                                         uint32_t num_instances) {
  const uint32_t num_network_cache_indices = indices->network_cache_indices_offsets[num_instances];
  for (uint32_t i = blockIdx.x; i < num_network_cache_indices; i += gridDim.x)
    gradients[indices->network_cache_indices[i] * embedding_vec_size + threadIdx.x] = 0.0f;
}

template <typename dtype, typename emtype>
__global__ void frequent_local_reduce(const emtype* __restrict__ gradients_in,
                                      float* __restrict__ gradients_out,
                                      size_t local_samples_offset,
                                      const dtype* __restrict__ category_location,
                                      uint32_t embedding_vec_size,
                                      FrequentEmbeddingCompressionView<dtype>* indices) {
  const uint32_t num_frequent_sample_indices = *indices->d_num_frequent_sample_indices;

  for (uint32_t i = blockIdx.x; i < num_frequent_sample_indices; i += gridDim.x) {
    uint32_t local_sample_index = indices->frequent_sample_indices[i];
    dtype category = indices->samples[local_samples_offset + local_sample_index];
    dtype frequent_index = category_location[2 * category + 1];

    atomicAdd(gradients_out + frequent_index * embedding_vec_size + threadIdx.x,
              TypeConvertFunc<float, emtype>::convert(
                  gradients_in[local_sample_index * embedding_vec_size + threadIdx.x]));
  }
}

template <typename emtype>
__forceinline__ __device__ void update_model_direct_common(
    const emtype* const* __restrict__ gradients_pointers, float* __restrict__ embedding_vectors,
    const uint32_t* __restrict__ model_cache_indices,
    const uint32_t* __restrict__ model_cache_indices_offsets, uint32_t num_instances,
    uint32_t model_id, uint32_t num_frequent_per_model, uint32_t embedding_vec_size, float lr) {}

template <typename dtype, typename emtype>
__global__ void update_model_direct(const emtype* const* __restrict__ gradients_pointers,
                                    float* __restrict__ embedding_vectors,
                                    FrequentEmbeddingCompressionView<dtype>* indices,
                                    uint32_t num_instances, uint32_t model_id,
                                    uint32_t num_frequent_per_model, uint32_t embedding_vec_size,
                                    const float* __restrict__ lr_ptr, const float scale) {
  float lr = __ldg(lr_ptr) / scale;
  const uint32_t offset = indices->model_cache_indices_offsets[model_id + 1];
  const uint32_t num_model_cache_indices = indices->model_cache_indices_offsets[num_instances];

  for (uint32_t i = blockIdx.x; i < num_model_cache_indices; i += gridDim.x) {
    int vid = (i + offset) % num_model_cache_indices;

    uint32_t frequent_index = indices->model_cache_indices[vid];
    uint32_t network_id;
    for (network_id = 0;
         network_id < num_instances && indices->model_cache_indices_offsets[network_id + 1] <= vid;
         network_id++)
      ;

    const emtype* gradients = gradients_pointers[network_id];

    uint32_t cache_location = frequent_index * embedding_vec_size + threadIdx.x;
    atomicAdd(embedding_vectors + cache_location,
              -lr * TypeConvertFunc<float, emtype>::convert(gradients[cache_location]));
  }
}

}  // namespace frequent_embedding_kernels

template <typename dtype>
FrequentEmbeddingBase<dtype>::FrequentEmbeddingBase() {}

template <typename dtype>
FrequentEmbeddingBase<dtype>::~FrequentEmbeddingBase() {}

template <typename dtype>
void FrequentEmbeddingBase<dtype>::set_current_indices(
    FrequentEmbeddingCompression<dtype>* indices) {
  indices_ = indices;
  data_ = indices->get_data();
  indices_view_ = indices->get_device_view();
}

template <typename dtype, typename emtype>
FrequentEmbeddingData<dtype, emtype>::FrequentEmbeddingData(const Model<dtype>& model,
                                                            const GPUResource& gpu_resource,
                                                            BuffPtr<emtype>& grouped_wgrad_buff,
                                                            uint32_t embedding_vec_size,
                                                            size_t max_num_frequent_categories)
    : model_(model),
      gpu_resource_(gpu_resource),
      grouped_wgrad_buff_(grouped_wgrad_buff),
      embedding_vec_size_(embedding_vec_size),
      max_num_frequent_categories_(max_num_frequent_categories) {
  std::shared_ptr<GeneralBuffer2<CudaAllocator>> buf = GeneralBuffer2<CudaAllocator>::create();
  buf->reserve({max_num_frequent_categories, embedding_vec_size_}, &frequent_embedding_vectors_);
  if (sizeof(emtype) != sizeof(float)) {
    buf->reserve({max_num_frequent_categories, embedding_vec_size_}, &float_frequent_gradients_);
  }

  auto& gradients = get_gradients();
  if (grouped_wgrad_buff == NULL) {
    buf->reserve({max_num_frequent_categories, embedding_vec_size_}, &gradients);
  } else {
    grouped_wgrad_buff->reserve({max_num_frequent_categories, embedding_vec_size_}, &gradients);
  }

  buf->allocate();
}

template <typename dtype, typename emtype>
FrequentEmbeddingSingleNode<dtype, emtype>::FrequentEmbeddingSingleNode(
    const Model<dtype>& model, const GPUResource& gpu_resource, BuffPtr<emtype>& grouped_wgrad_buff,
    uint32_t embedding_vec_size, size_t max_num_frequent_categories)
    : frequent_data_(model, gpu_resource, grouped_wgrad_buff, embedding_vec_size,
                     max_num_frequent_categories) {
  std::shared_ptr<GeneralBuffer2<CudaAllocator>> buf = GeneralBuffer2<CudaAllocator>::create();

  buf->reserve({model.num_instances, 1}, &embedding_vectors_cache_pointers_);
  buf->reserve({model.num_instances, 1}, &partial_gradients_pointers_);
  if (sizeof(emtype) != sizeof(float)) {
    buf->reserve({max_num_frequent_categories, embedding_vec_size},
                 &frequent_embedding_vectors_cache_);
  }
  buf->allocate();
}

template <typename dtype, typename emtype>
void FrequentEmbeddingMultiNode<dtype, emtype>::init_ar_comm(AllReduceInPlaceComm* ar_comm,
                                                             AllReduceInPlaceComm::Handle& handle,
                                                             int local_id) {
  auto& local_gpu = frequent_data_.gpu_resource_;
  CudaDeviceContext context(local_gpu.get_device_id());

  auto& gradients = frequent_data_.get_gradients();
  ar_comm->set_coll_buf(handle, gradients.get_ptr(), gradients.get_size_in_bytes(), local_id);
  ar_comm_ = std::make_unique<AllReduceComm<emtype>>(ar_comm, handle, &local_gpu);
}

template <typename dtype, typename emtype>
void FrequentEmbeddingData<dtype, emtype>::initialize_embedding_vectors(
    const std::vector<size_t>& table_sizes, size_t grouped_wgrad_offset_in_bytes) {
  CudaDeviceContext context(gpu_resource_.get_device_id());

  const size_t num_tables = table_sizes.size();
  for (size_t model_id = 0; model_id < model_.num_instances; ++model_id) {
    for (size_t embedding = 0; embedding < num_tables; embedding++) {
      float up_bound = sqrt(1.f / table_sizes[embedding]);
      size_t offset =
          embedding_vec_size_ *
          model_.h_frequent_model_table_offsets[model_id * (num_tables + 1) + embedding];
      size_t num_elements =
          embedding_vec_size_ *
          (model_.h_frequent_model_table_offsets[model_id * (num_tables + 1) + embedding + 1] -
           model_.h_frequent_model_table_offsets[model_id * (num_tables + 1) + embedding]);
      UniformGenerator::fill(frequent_embedding_vectors_.get_ptr() + offset, num_elements,
                             -up_bound, up_bound, gpu_resource_.get_sm_count(),
                             gpu_resource_.get_replica_uniform_curand_generator(),
                             gpu_resource_.get_stream());
    }
  }
  if (grouped_wgrad_buff_ != NULL) {
    // update wgrad tensors
    size_t grad_size = model_.num_frequent * embedding_vec_size_;
    if (sizeof(float) != sizeof(emtype)) {
      auto buf = std::make_shared<ExternalManagedBuffer>(
          (char*)grouped_wgrad_buff_->as_tensor().get_ptr() + grouped_wgrad_offset_in_bytes);
      frequent_gradients_ = Tensor2<emtype>({grad_size}, buf);
    } else {
      auto buf = std::make_shared<ExternalManagedBuffer>(
          (char*)grouped_wgrad_buff_->as_tensor().get_ptr() + grouped_wgrad_offset_in_bytes);
      float_frequent_gradients_ = Tensor2<float>({grad_size}, buf);
    }
  }
}

/* Single-node: refresh needed vectors in the cache of each network
 * Note: each network pulls from the models */
template <typename dtype, typename emtype>
void FrequentEmbeddingSingleNode<dtype, emtype>::forward_model(hipStream_t stream) {
  const uint32_t num_instances = frequent_data_.model_.num_instances;
  const uint32_t model_id = frequent_data_.model_.global_instance_id;

  auto embedding_vectors_cache_pointers = embedding_vectors_cache_pointers_.get_ptr();
  auto frequent_embedding_vectors = frequent_data_.frequent_embedding_vectors_.get_ptr();
  auto indices = this->indices_view_;
  auto embedding_vec_size = frequent_data_.embedding_vec_size_;

  auto copy_desc = CopyDescriptors::make_OneToOne<float, emtype, 1>(
      embedding_vec_size,
      [=] __device__() { return indices->model_cache_indices_offsets[num_instances]; },
      [=] __device__(size_t i) -> CopyDescriptors::CopyDetails<float, emtype, 1> {
        const uint32_t offset = indices->model_cache_indices_offsets[model_id + 1];
        const uint32_t num_model_cache_indices =
            indices->model_cache_indices_offsets[num_instances];
        int vid = (i + offset) % num_model_cache_indices;
        uint32_t frequent_index = indices->model_cache_indices[vid];

        uint32_t network_id;
        for (network_id = 0; network_id < num_instances &&
                             indices->model_cache_indices_offsets[network_id + 1] <= vid;
             network_id++)
          ;
        emtype* embedding_vectors_out = embedding_vectors_cache_pointers[network_id];

        const float* src_ptr = frequent_embedding_vectors + frequent_index * embedding_vec_size;
        emtype* dst_ptr = embedding_vectors_out + frequent_index * embedding_vec_size;

        return {
            src_ptr, {dst_ptr}, {static_cast<const void*>(src_ptr) != static_cast<void*>(dst_ptr)}};
      });

  shuffle(copy_desc, stream, frequent_data_.model_.num_frequent / 4);
  HCTR_LIB_THROW(hipPeekAtLastError());
}

/* Single-node: refresh all vectors in the cache of each network */
template <typename dtype, typename emtype>
void FrequentEmbeddingSingleNode<dtype, emtype>::forward_model_eval(hipStream_t stream) {
  const uint32_t num_instances = frequent_data_.model_.num_instances;
  const uint32_t model_id = frequent_data_.model_.global_instance_id;

  emtype** embedding_vectors_cache_pointers = embedding_vectors_cache_pointers_.get_ptr();
  const float* frequent_embedding_vectors = frequent_data_.frequent_embedding_vectors_.get_ptr();
  size_t embedding_vec_size = frequent_data_.embedding_vec_size_;
  const uint32_t num_frequent = frequent_data_.model_.num_frequent;
  const uint32_t num_frequent_per_model = num_frequent / num_instances;

  auto copy_desc = CopyDescriptors::make_OneToOne<float, emtype, 1>(
      embedding_vec_size, [=] __device__() { return num_frequent; },
      [=] __device__(size_t i) -> CopyDescriptors::CopyDetails<float, emtype, 1> {
        // Shift pattern
        uint32_t shifted_i = (i + (model_id + 1) * num_frequent_per_model) % num_frequent;
        uint32_t network_id = shifted_i / num_frequent_per_model;
        uint32_t frequent_index =
            model_id * num_frequent_per_model + shifted_i % num_frequent_per_model;

        emtype* embedding_vectors_out = embedding_vectors_cache_pointers[network_id];

        const float* src_ptr = frequent_embedding_vectors + frequent_index * embedding_vec_size;
        emtype* dst_ptr = embedding_vectors_out + frequent_index * embedding_vec_size;

        return {
            src_ptr, {dst_ptr}, {static_cast<const void*>(src_ptr) != static_cast<void*>(dst_ptr)}};
      });

  shuffle(copy_desc, stream, num_frequent);
  HCTR_LIB_THROW(hipPeekAtLastError());
}

template <typename dtype, typename emtype>
template <typename vectype>
void FrequentEmbeddingData<dtype, emtype>::forward_network<vectype>(
    const vectype* embedding_vectors, emtype* interaction_layer_input,
    FrequentEmbeddingBase<dtype>* base, hipStream_t stream) {
  uint32_t samples_per_instance =
      base->data_->samples.get_num_elements() / this->model_.num_instances;
  uint32_t global_sample_index_base = model_.global_instance_id * samples_per_instance;

  auto indices = base->indices_view_;
  auto category_location = this->model_.category_location.get_ptr();
  auto embedding_vec_size = this->embedding_vec_size_;

  auto copy_desc = CopyDescriptors::make_OneToOne<vectype, emtype, 1>(
      embedding_vec_size,
      [=] __device__() -> size_t { return *indices->d_num_frequent_sample_indices; },
      [=] __device__(size_t i) -> CopyDescriptors::CopyDetails<vectype, emtype, 1> {
        auto index = indices->frequent_sample_indices[i];
        auto category = indices->samples[index + global_sample_index_base];
        auto frequent_index = category_location[2 * category + 1];

        return {
            embedding_vectors + frequent_index * embedding_vec_size,
            {interaction_layer_input + indices->frequent_sample_indices[i] * embedding_vec_size},
            {true}};
      });

  shuffle(copy_desc, stream, samples_per_instance);
  HCTR_LIB_THROW(hipPeekAtLastError());
}

/* Concatenate the embedding vectors into the buffer for top-mlp input */
template <typename dtype, typename emtype>
void FrequentEmbeddingSingleNode<dtype, emtype>::forward_network(emtype* interaction_layer_input,
                                                                 hipStream_t stream) {
  frequent_data_.forward_network(get_embedding_vectors_cache().get_ptr(), interaction_layer_input,
                                 this, stream);
}

template <typename dtype, typename emtype>
void FrequentEmbeddingMultiNode<dtype, emtype>::forward_network(emtype* interaction_layer_input,
                                                                hipStream_t stream) {
  frequent_data_.forward_network(frequent_data_.frequent_embedding_vectors_.get_ptr(),
                                 interaction_layer_input, this, stream);
}

/* Reduce gradients on each network */
template <typename dtype, typename emtype>
void FrequentEmbeddingData<dtype, emtype>::local_reduce(const emtype* gradients,
                                                        FrequentEmbeddingBase<dtype>* base,
                                                        hipStream_t stream) {
  const auto num_instances = model_.num_instances;
  const auto network_id = model_.global_instance_id;
  size_t local_samples_size =
      ceildiv<size_t>(base->data_->batch_size, num_instances) * base->data_->table_sizes.size();

  int n_blocks = 16 * gpu_resource_.get_sm_count();
  auto embedding_vec_size = embedding_vec_size_;

  frequent_embedding_kernels::frequent_local_reduce<<<n_blocks, embedding_vec_size, 0, stream>>>(
      gradients, float_frequent_gradients_.get_ptr(), network_id * local_samples_size,
      model_.category_location.get_ptr(), embedding_vec_size, base->indices_view_);
  HCTR_LIB_THROW(hipPeekAtLastError());

  if (sizeof(emtype) != sizeof(float)) {
    convert_array<<<1000, 128, 0, stream>>>(frequent_gradients_.get_ptr(),
                                            float_frequent_gradients_.get_ptr(),
                                            model_.num_frequent * embedding_vec_size);
    HCTR_LIB_THROW(hipPeekAtLastError());
  }
}

template <typename dtype, typename emtype>
void FrequentEmbeddingSingleNode<dtype, emtype>::local_reduce(const emtype* gradients,
                                                              hipStream_t stream) {
  auto num_instances = frequent_data_.model_.num_instances;
  int n_blocks = 16 * frequent_data_.gpu_resource_.get_sm_count();
  auto embedding_vec_size = frequent_data_.embedding_vec_size_;

  /* Set to zero the gradients of categories that appear in the batch */
  frequent_embedding_kernels::reset_relevant_gradients<<<n_blocks, embedding_vec_size, 0, stream>>>(
      frequent_data_.float_frequent_gradients_.get_ptr(), embedding_vec_size, this->indices_view_,
      num_instances);
  HCTR_LIB_THROW(hipPeekAtLastError());

  frequent_data_.local_reduce(gradients, this, stream);
}

template <typename dtype, typename emtype>
void FrequentEmbeddingMultiNode<dtype, emtype>::local_reduce(const emtype* gradients,
                                                             hipStream_t stream) {
  /* Set to zero all the gradients */
  if (frequent_data_.model_.num_frequent > 0) {
    HCTR_LIB_THROW(hipMemsetAsync(
        frequent_data_.float_frequent_gradients_.get_ptr(), 0,
        frequent_data_.model_.num_frequent * frequent_data_.embedding_vec_size_ * sizeof(float),
        stream));
  }

  frequent_data_.local_reduce(gradients, this, stream);
}

template <typename dtype, typename emtype>
void FrequentEmbeddingMultiNode<dtype, emtype>::update_model(float* dev_lr, float scale,
                                                             hipStream_t stream) {
  sgd_global_update(frequent_data_.get_gradients().get_ptr(),
                    frequent_data_.frequent_embedding_vectors_.get_ptr(),
                    frequent_data_.model_.num_frequent, frequent_data_.embedding_vec_size_, dev_lr,
                    scale, stream);
}

/* Update model for single-node: direct write in category "owner"'s table, lr is a device variable
 */
template <typename dtype, typename emtype>
void FrequentEmbeddingSingleNode<dtype, emtype>::update_model_direct(float* dev_lr, float scale,
                                                                     hipStream_t stream) {
  const uint32_t& num_instances = frequent_data_.model_.num_instances;
  const uint32_t& model_id = frequent_data_.model_.global_instance_id;
  const uint32_t num_frequent_per_model = frequent_data_.model_.num_frequent / num_instances;

  int num_sm = frequent_data_.gpu_resource_.get_sm_count();
  int n_blocks = 8 * num_sm;  // TODO: better heuristics

  /* Update models */
  frequent_embedding_kernels::
      update_model_direct<<<n_blocks, frequent_data_.embedding_vec_size_, 0, stream>>>(
          partial_gradients_pointers_.get_ptr(),
          frequent_data_.frequent_embedding_vectors_.get_ptr(), this->indices_view_, num_instances,
          model_id, num_frequent_per_model, frequent_data_.embedding_vec_size_, dev_lr, scale);
  HCTR_LIB_THROW(hipPeekAtLastError());
}

template <typename dtype, typename emtype>
void FrequentEmbeddingMultiNode<dtype, emtype>::communicate(hipStream_t stream) {
  ar_comm_->communicate(stream);
}

template class FrequentEmbeddingBase<uint32_t>;
template class FrequentEmbeddingBase<long long>;

template class FrequentEmbeddingData<uint32_t, __half>;
template class FrequentEmbeddingData<uint32_t, float>;
template class FrequentEmbeddingData<long long, __half>;
template class FrequentEmbeddingData<long long, float>;

template class FrequentEmbeddingSingleNode<uint32_t, __half>;
template class FrequentEmbeddingSingleNode<uint32_t, float>;
template class FrequentEmbeddingSingleNode<long long, __half>;
template class FrequentEmbeddingSingleNode<long long, float>;

template class FrequentEmbeddingMultiNode<uint32_t, __half>;
template class FrequentEmbeddingMultiNode<uint32_t, float>;
template class FrequentEmbeddingMultiNode<long long, __half>;
template class FrequentEmbeddingMultiNode<long long, float>;

template void FrequentEmbeddingData<uint32_t, __half>::forward_network<__half>(
    const __half*, __half*, FrequentEmbeddingBase<uint32_t>*, hipStream_t);
template void FrequentEmbeddingData<uint32_t, __half>::forward_network<float>(
    const float*, __half*, FrequentEmbeddingBase<uint32_t>*, hipStream_t);
template void FrequentEmbeddingData<uint32_t, float>::forward_network<float>(
    const float*, float*, FrequentEmbeddingBase<uint32_t>*, hipStream_t);
template void FrequentEmbeddingData<long long, __half>::forward_network<__half>(
    const __half*, __half*, FrequentEmbeddingBase<long long>*, hipStream_t);
template void FrequentEmbeddingData<long long, __half>::forward_network<float>(
    const float*, __half*, FrequentEmbeddingBase<long long>*, hipStream_t);
template void FrequentEmbeddingData<long long, float>::forward_network<float>(
    const float*, float*, FrequentEmbeddingBase<long long>*, hipStream_t);
}  // namespace hybrid_embedding

}  // namespace HugeCTR
