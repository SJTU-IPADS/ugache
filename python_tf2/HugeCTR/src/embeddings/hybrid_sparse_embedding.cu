#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_runtime.h>

#include <collectives/all_reduce_comm.hpp>
#include <vector>

#include "HugeCTR/include/common.hpp"
#include "HugeCTR/include/embedding.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/calibration_data.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/data.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/frequent_embedding.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/indices_container.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/infrequent_embedding.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/model.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/statistics.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/utils.hpp"
#include "HugeCTR/include/embeddings/hybrid_sparse_embedding.hpp"
#include "HugeCTR/include/tensor2.hpp"

namespace HugeCTR {
template <typename dtype, typename emtype>
HybridSparseEmbedding<dtype, emtype>::HybridSparseEmbedding(
    const SparseTensors<dtype> &train_input_tensors,
    const SparseTensors<dtype> &evaluate_input_tensors,
    const HybridSparseEmbeddingParams &embedding_params,
    const std::vector<BuffPtr<emtype>> &grouped_wgrad_buff,
    const GpuLearningRateSchedulers lr_scheds, bool graph_mode,
    const std::shared_ptr<ResourceManager> &resource_manager)
    : embedding_params_(embedding_params),
      resource_manager_(resource_manager),
      grouped_wgrad_buff_(grouped_wgrad_buff),
      grouped_all_reduce_(grouped_wgrad_buff[0] != NULL),
      lr_scheds_(lr_scheds),
      graph_mode_(graph_mode),
      current_train_batch_size_(get_batch_size(true)),
      current_eval_batch_size_(get_batch_size(false)) {
  try {
    // 0. Error check
    if (embedding_params_.train_batch_size < 1 || embedding_params_.evaluate_batch_size < 1 ||
        embedding_params_.slot_num < 1 || embedding_params_.embedding_vec_size < 1) {
      HCTR_OWN_THROW(Error_t::WrongInput,
                     "batchsize < 1 || slot_num < 1 || embedding_vec_size < 1");
    }

    if (embedding_params_.embedding_vec_size > 1024) {
      HCTR_OWN_THROW(Error_t::WrongInput,
                     "the embedding_vec_size can not be more than 1024 in embedding layer");
    }

    size_t total_gpu_count = resource_manager_->get_global_gpu_count();
    size_t local_gpu_count = resource_manager_->get_local_gpu_count();

    if (train_input_tensors.size() != local_gpu_count ||
        evaluate_input_tensors.size() != local_gpu_count) {
      HCTR_OWN_THROW(Error_t::WrongInput,
                     "either train_input_tensors.size() or evaluate_input_tensors.size() isn't "
                     "local_gpu_count_");
    }

    HCTR_LOG_S(INFO, ROOT) << "Using Hybrid Embedding with train batch " << get_batch_size(true)
                           << " and eval batch " << get_batch_size(false) << std::endl;

    // 1. initialize optimizer
    for (size_t id = 0; id < local_gpu_count; id++) {
      OptParams opt_params;
      opt_params.optimizer = embedding_params_.opt_params.optimizer;
      opt_params.lr = embedding_params_.opt_params.lr;
      opt_params.update_type = embedding_params_.opt_params.update_type;
      opt_params.scaler = embedding_params_.opt_params.scaler;
      opt_params_.emplace_back(opt_params);
    }
    // 2. reserve buffers for different tensors
    data_statistics_.reserve(local_gpu_count);
    model_.reserve(local_gpu_count);
    calibration_.reserve(local_gpu_count);
    statistics_.reserve(local_gpu_count);
    train_output_tensors_.reserve(local_gpu_count);
    evaluate_output_tensors_.reserve(local_gpu_count);
    if (embedding_params_.communication_type == CommunicationType::NVLink_SingleNode) {
      frequent_embeddings_single_node_.reserve(local_gpu_count);
    } else {
      frequent_embeddings_multi_node_.reserve(local_gpu_count);
    }

    infrequent_embeddings_single_node_.reserve(local_gpu_count);
    infrequent_embeddings_ib_nvlink_.reserve(local_gpu_count);
    infrequent_embeddings_ib_nvlink_hier_.reserve(local_gpu_count);

    assert(bufs_.empty());
    CudaDeviceContext context;
    // 2.1. construct data
    for (uint32_t i = 0; i < local_gpu_count; i++) {
      int cur_device = get_local_gpu(i).get_device_id();
      context.set_device(cur_device);

      data_statistics_.emplace_back(embedding_params_.slot_size_array, get_batch_size(true),
                                    embedding_params_.num_iterations_statistics);
    }

    // 2.2 construct model
    for (uint32_t i = 0; i < local_gpu_count; i++) {
      int cur_device = get_local_gpu(i).get_device_id();
      context.set_device(cur_device);

      std::vector<uint32_t> num_instances_per_node(resource_manager_->get_num_process(), 0);
      get_num_instances_per_node(num_instances_per_node);
      model_.emplace_back(embedding_params_.communication_type,
                          resource_manager_->get_local_gpu(i)->get_global_id(),
                          num_instances_per_node, get_categories_num());
    }

    // 2.3 construct calibration
    for (uint32_t i = 0; i < local_gpu_count; i++) {
      int cur_device = get_local_gpu(i).get_device_id();
      context.set_device(cur_device);
      calibration_.emplace_back(resource_manager_->get_num_process(), embedding_params_.p_dup_max,
                                embedding_params_.max_all_reduce_bandwidth,
                                embedding_params_.max_all_to_all_bandwidth,
                                embedding_params_.efficiency_bandwidth_ratio);
    }

    // 2.4 construct Statistics
    for (uint32_t i = 0; i < local_gpu_count; i++) {
      int cur_device = get_local_gpu(i).get_device_id();
      context.set_device(cur_device);
      const size_t num_samples_statistics = embedding_params_.num_iterations_statistics *
                                            get_batch_size(true) * embedding_params_.slot_num;
      statistics_.emplace_back((dtype)num_samples_statistics, embedding_params_.slot_num,
                               model_[i].num_instances, get_categories_num());
    }

    for (uint32_t i = 0; i < local_gpu_count; i++) {
      int cur_device = get_local_gpu(i).get_device_id();
      context.set_device(cur_device);
      std::shared_ptr<GeneralBuffer2<CudaAllocator>> buf = GeneralBuffer2<CudaAllocator>::create();
      bufs_.emplace_back(buf);
      // 2.5. reserve for train output/ evaluate output tensors
      Tensor2<emtype> tensor;
      buf->reserve({get_batch_size_per_gpu(true), get_slot_num(), get_embedding_vec_size()},
                   &tensor);
      train_output_tensors_.emplace_back(tensor);
      buf->reserve({get_batch_size_per_gpu(false), get_slot_num(), get_embedding_vec_size()},
                   &tensor);
      evaluate_output_tensors_.emplace_back(tensor);

      // 2.6 construct frequent embedding
      if (embedding_params_.communication_type == CommunicationType::NVLink_SingleNode) {
        frequent_embeddings_single_node_.emplace_back(
            model_[i], get_local_gpu(i), grouped_wgrad_buff_[i], get_embedding_vec_size(),
            embedding_params_.max_num_frequent_categories);
      } else {
        frequent_embeddings_multi_node_.emplace_back(
            model_[i], get_local_gpu(i), grouped_wgrad_buff_[i], get_embedding_vec_size(),
            embedding_params_.max_num_frequent_categories);
      }

      // 2.7 construct infrequent embedding
      if (embedding_params_.communication_type == CommunicationType::NVLink_SingleNode) {
        infrequent_embeddings_single_node_.emplace_back(model_[i], get_local_gpu(i),
                                                        get_embedding_vec_size());
      }
      if (embedding_params_.communication_type == CommunicationType::IB_NVLink) {
        infrequent_embeddings_ib_nvlink_.emplace_back(model_[i], get_local_gpu(i),
                                                      get_embedding_vec_size());
      }
      if (embedding_params_.communication_type == CommunicationType::IB_NVLink_Hier) {
        infrequent_embeddings_ib_nvlink_hier_.emplace_back(model_[i], get_local_gpu(i),
                                                           get_embedding_vec_size());
      }

      // 2.8 construct communication
      if (embedding_params_.communication_type == CommunicationType::IB_NVLink) {
        size_t max_buf_size = embedding_params_.embedding_vec_size *
                              std::max(get_batch_size(true), get_batch_size(false)) *
                              embedding_params_.slot_num;
        infrequent_embeddings_ib_nvlink_.back().init_comms(
            embedding_params_.embedding_vec_size, &get_local_gpu(i), buf.get(), max_buf_size);
      }

      // Construct comm buffers
      if (embedding_params_.communication_type == CommunicationType::IB_NVLink_Hier) {
        infrequent_embeddings_ib_nvlink_hier_[i].init_comms(
            embedding_params_.max_num_infrequent_samples, embedding_params_.slot_num,
            embedding_params_.embedding_vec_size, buf.get(), get_batch_size(true),
            get_batch_size(false), local_gpu_count);
      }

      // For global barrier in eval
      {
        Tensor2<uint32_t> tensor;
        buf->reserve({1}, &tensor);
        d_barrier_store_.push_back(tensor);
      }
      buf->allocate();
    }

    // Frequent AR comm init
    if ((embedding_params_.communication_type == CommunicationType::IB_NVLink_Hier) ||
        (embedding_params_.communication_type == CommunicationType::IB_NVLink)) {
      if (!grouped_all_reduce_) {
        // Do your own all-reduce
        auto ar_comm = resource_manager_->get_ar_comm();
        frequent_embedding_handle_ = ar_comm->register_coll();
        // Frequent all reduce comm
        for (uint32_t i = 0; i < local_gpu_count; i++) {
          frequent_embeddings_multi_node_[i].init_ar_comm(ar_comm, frequent_embedding_handle_, i);
        }
        ar_comm->register_coll_buf(frequent_embedding_handle_);
      }
    }

    // Init after buffer allocation
    if (embedding_params_.communication_type == CommunicationType::IB_NVLink_Hier) {
#ifdef ENABLE_MPI
      ib_comm_ = resource_manager_->get_ib_comm();
      if (!ib_comm_) {
        resource_manager_->init_ib_comm();
        ib_comm_ = resource_manager_->get_ib_comm();
      }
      comm_stream_.resize(local_gpu_count);

      std::vector<size_t *> h_model_indices_sizes_ptrs(local_gpu_count);
      std::vector<size_t *> h_network_indices_sizes_ptrs(local_gpu_count);
      std::vector<emtype *> h_fwd_send_buffer_ptrs(local_gpu_count);
      std::vector<emtype *> h_bwd_send_buffer_ptrs(local_gpu_count);
      for (uint32_t i = 0; i < local_gpu_count; i++) {
        h_model_indices_sizes_ptrs[i] =
            infrequent_embeddings_ib_nvlink_hier_[i].model_indices_sizes_.get_ptr();
        h_network_indices_sizes_ptrs[i] =
            infrequent_embeddings_ib_nvlink_hier_[i].network_indices_sizes_.get_ptr();
        h_fwd_send_buffer_ptrs[i] = infrequent_embeddings_ib_nvlink_hier_[i]
                                        .infrequent_forward_comm_buffers_->send_buffer.get_ptr();
        h_bwd_send_buffer_ptrs[i] = infrequent_embeddings_ib_nvlink_hier_[i]
                                        .infrequent_backward_comm_buffers_->send_buffer.get_ptr();
      }

      // Forward coll init
      auto infrequent_forward_coll_handle = ib_comm_->register_hier_a2a_v_coll(true);
      for (uint32_t i = 0; i < local_gpu_count; i++) {
        int cur_device = get_local_gpu(i).get_device_id();
        context.set_device(cur_device);

        // download pointers
        HCTR_LIB_THROW(hipMemcpyAsync(
            infrequent_embeddings_ib_nvlink_hier_[i].model_indices_sizes_ptrs_.get_ptr(),
            h_model_indices_sizes_ptrs.data(), sizeof(size_t *) * local_gpu_count,
            hipMemcpyHostToDevice, get_local_gpu(i).get_stream()));

        HCTR_LIB_THROW(hipMemcpyAsync(
            infrequent_embeddings_ib_nvlink_hier_[i].network_indices_sizes_ptrs_.get_ptr(),
            h_network_indices_sizes_ptrs.data(), sizeof(size_t *) * local_gpu_count,
            hipMemcpyHostToDevice, get_local_gpu(i).get_stream()));

        HCTR_LIB_THROW(
            hipMemcpyAsync(infrequent_embeddings_ib_nvlink_hier_[i]
                                .infrequent_forward_comm_buffers_->send_buffer_ptrs.get_ptr(),
                            h_fwd_send_buffer_ptrs.data(), sizeof(emtype *) * local_gpu_count,
                            hipMemcpyHostToDevice, get_local_gpu(i).get_stream()));

        HCTR_LIB_THROW(
            hipMemcpyAsync(infrequent_embeddings_ib_nvlink_hier_[i]
                                .infrequent_backward_comm_buffers_->send_buffer_ptrs.get_ptr(),
                            h_bwd_send_buffer_ptrs.data(), sizeof(emtype *) * local_gpu_count,
                            hipMemcpyHostToDevice, get_local_gpu(i).get_stream()));

        HCTR_LIB_THROW(hipStreamSynchronize(get_local_gpu(i).get_stream()));

        // Initialize IB comm
        HCTR_LIB_THROW(hipStreamCreateWithPriority(&comm_stream_[i], hipStreamNonBlocking, -100));
        ib_comm_->set_a2a_coll_stream(infrequent_forward_coll_handle, comm_stream_[i], i);

        ib_comm_->set_a2a_coll_buf(
            infrequent_forward_coll_handle,
            infrequent_embeddings_ib_nvlink_hier_[i]
                .infrequent_forward_comm_buffers_->send_buffer.get_ptr(),
            infrequent_embeddings_ib_nvlink_hier_[i]
                .infrequent_forward_comm_buffers_->send_buffer.get_size_in_bytes(),
            infrequent_embeddings_ib_nvlink_hier_[i]
                .infrequent_forward_comm_buffers_->recv_buffer.get_ptr(),
            infrequent_embeddings_ib_nvlink_hier_[i]
                .infrequent_forward_comm_buffers_->recv_buffer.get_size_in_bytes(),
            i);

        infrequent_embeddings_ib_nvlink_hier_[i].infrequent_forward_comms_ =
            std::make_unique<HierAll2Allv_Multi_IB<emtype>>(
                i, infrequent_forward_coll_handle,
                infrequent_embeddings_ib_nvlink_hier_[i].model_indices_sizes_ptrs_.get_ptr(),
                &get_local_gpu(i), ib_comm_, comm_stream_[i]);
      }
      ib_comm_->register_a2a_coll_buf(infrequent_forward_coll_handle);

      // Backward coll init
      auto infrequent_backward_coll_handle = ib_comm_->register_hier_a2a_v_coll(true);
      for (uint32_t i = 0; i < local_gpu_count; i++) {
        int cur_device = get_local_gpu(i).get_device_id();
        context.set_device(cur_device);

        ib_comm_->set_a2a_coll_stream(infrequent_backward_coll_handle, comm_stream_[i], i);
        ib_comm_->set_a2a_coll_buf(
            infrequent_backward_coll_handle,
            infrequent_embeddings_ib_nvlink_hier_[i]
                .infrequent_backward_comm_buffers_->send_buffer.get_ptr(),
            infrequent_embeddings_ib_nvlink_hier_[i]
                .infrequent_backward_comm_buffers_->send_buffer.get_size_in_bytes(),
            infrequent_embeddings_ib_nvlink_hier_[i]
                .infrequent_backward_comm_buffers_->recv_buffer.get_ptr(),
            infrequent_embeddings_ib_nvlink_hier_[i]
                .infrequent_backward_comm_buffers_->recv_buffer.get_size_in_bytes(),
            i);

        infrequent_embeddings_ib_nvlink_hier_[i].infrequent_backward_comms_ =
            std::make_unique<HierAll2Allv_Multi_IB<emtype>>(
                i, infrequent_backward_coll_handle,
                infrequent_embeddings_ib_nvlink_hier_[i].network_indices_sizes_ptrs_.get_ptr(),
                &get_local_gpu(i), ib_comm_, comm_stream_[i]);
      }
      ib_comm_->register_a2a_coll_buf(infrequent_backward_coll_handle);
#else
      HCTR_OWN_THROW(Error_t::WrongInput, "MPI is not enabled but trying to use IB_NVLink_Hier");
#endif
    }

    // 2.9 Single-node: copy some pointers arrays to device
    if (embedding_params_.communication_type == CommunicationType::NVLink_SingleNode) {
      // Initialize GPU barrier
      gpu_barrier_ = std::make_unique<GPUBarrier>(resource_manager_->get_local_gpu_count(),
                                                  resource_manager_->get_local_gpu_device_id_list(),
                                                  graph_mode_);

      std::vector<const emtype *> frequent_vectors_cache_pointers(local_gpu_count);
      std::vector<emtype *> interaction_layer_input_pointers_train(local_gpu_count);
      std::vector<emtype *> interaction_layer_input_pointers_eval(local_gpu_count);
      std::vector<const emtype *> gradients_pointers(local_gpu_count);
      std::vector<const emtype *> frequent_partial_gradients_pointers(local_gpu_count);

      for (uint32_t i = 0; i < local_gpu_count; i++) {
        frequent_vectors_cache_pointers[i] =
            frequent_embeddings_single_node_[i].get_embedding_vectors_cache().get_ptr();
        interaction_layer_input_pointers_train[i] = train_output_tensors_[i].get_ptr();
        gradients_pointers[i] = train_output_tensors_[i].get_ptr();
        interaction_layer_input_pointers_eval[i] = evaluate_output_tensors_[i].get_ptr();
        frequent_partial_gradients_pointers[i] =
            frequent_embeddings_single_node_[i].frequent_data_.get_gradients().get_ptr();
      }

      for (uint32_t i = 0; i < local_gpu_count; i++) {
        int cur_device = get_local_gpu(i).get_device_id();
        context.set_device(cur_device);

        HCTR_LIB_THROW(hipMemcpyAsync(
            frequent_embeddings_single_node_[i].embedding_vectors_cache_pointers_.get_ptr(),
            frequent_vectors_cache_pointers.data(), local_gpu_count * sizeof(float *),
            hipMemcpyHostToDevice, get_local_gpu(i).get_stream()));

        infrequent_embeddings_single_node_[i].init_pointers(
            local_gpu_count, get_local_gpu(i).get_stream(), interaction_layer_input_pointers_train,
            interaction_layer_input_pointers_eval, gradients_pointers);
        HCTR_LIB_THROW(hipMemcpyAsync(
            frequent_embeddings_single_node_[i].partial_gradients_pointers_.get_ptr(),
            frequent_partial_gradients_pointers.data(), local_gpu_count * sizeof(emtype *),
            hipMemcpyHostToDevice, get_local_gpu(i).get_stream()));
      }
    }

    // Setup default indices
    train_batch_indices_.emplace_back(model_, train_input_tensors, resource_manager_,
                                      get_batch_size(true), embedding_params_.slot_size_array,
                                      embedding_params_.max_num_frequent_categories,
                                      embedding_params_.communication_type);

    eval_batch_indices_.emplace_back(model_, evaluate_input_tensors, resource_manager_,
                                     get_batch_size(false), embedding_params_.slot_size_array,
                                     embedding_params_.max_num_frequent_categories,
                                     embedding_params_.communication_type);

  } catch (const std::runtime_error &rt_err) {
    HCTR_LOG_S(ERROR, WORLD) << rt_err.what() << std::endl;
    throw;
  }
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::init_model(const SparseTensors<dtype> &data,
                                                      size_t &wgrad_offset_in_bytes) {
  size_t local_gpu_count = resource_manager_->get_local_gpu_count();
  HCTR_LOG(INFO, ROOT, "Initializing Hybrid Embedding\n");
#pragma omp parallel for num_threads(local_gpu_count)
  for (size_t id = 0; id < local_gpu_count; ++id) {
    int cur_device = get_local_gpu(id).get_device_id();
    CudaDeviceContext context(cur_device);
    auto stream = get_local_gpu(id).get_stream();
    data_statistics_[id].data_to_unique_categories(data[id].get_value_tensor(), stream);
    model_[id].init_hybrid_model(calibration_[id], statistics_[id], data_statistics_[id], stream);

    get_frequent_embedding_data(id).initialize_embedding_vectors(data_statistics_[id].table_sizes,
                                                                 wgrad_offset_in_bytes);

    if (embedding_params_.communication_type == CommunicationType::NVLink_SingleNode) {
      infrequent_embeddings_single_node_[id].initialize_embedding_vectors(
          data_statistics_[id].table_sizes);
    }
    if (embedding_params_.communication_type == CommunicationType::IB_NVLink) {
      infrequent_embeddings_ib_nvlink_[id].initialize_embedding_vectors(
          data_statistics_[id].table_sizes);
    }
    if (embedding_params_.communication_type == CommunicationType::IB_NVLink_Hier) {
      infrequent_embeddings_ib_nvlink_hier_[id].initialize_embedding_vectors(
          data_statistics_[id].table_sizes);
    }

    if (embedding_params_.max_num_frequent_categories < (size_t)model_[id].num_frequent) {
      HCTR_OWN_THROW(
          Error_t::WrongInput,
          "Found too many frequent categories, please increase 'max_num_frequent_categories'");
    }
  }

  HCTR_LOG_S(INFO, ROOT) << "Initialized hybrid model with " << model_[0].num_frequent
                         << " frequent categories, probability of being frequent is "
                         << model_[0].frequent_probability << std::endl;

  size_t avg_train_infrequent = (1 - model_[0].frequent_probability) *
                                embedding_params_.slot_size_array.size() * get_batch_size(true);
  size_t avg_evaluate_infrequent = (1 - model_[0].frequent_probability) *
                                   embedding_params_.slot_size_array.size() * get_batch_size(false);

  HCTR_LOG_S(INFO, ROOT) << "Estimated number of infrequent categories per train batch: "
                         << avg_train_infrequent << ", eval batch: " << avg_evaluate_infrequent
                         << std::endl;

  if ((embedding_params_.communication_type == CommunicationType::IB_NVLink_Hier) ||
      (embedding_params_.communication_type == CommunicationType::IB_NVLink)) {
    size_t wgrad_size =
        model_[0].num_frequent * embedding_params_.embedding_vec_size * sizeof(emtype);

    if (!grouped_all_reduce_) {
      // Manage your own all-reduce
      auto ar_comm = resource_manager_->get_ar_comm();
      ar_comm->update_size(frequent_embedding_handle_, wgrad_size);
    } else {
      wgrad_offset_in_bytes += wgrad_size;
    }
  }
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::setup_buffered_indices(bool is_train,
                                                                  AsyncReader<dtype> *data_reader) {
  if (is_train) {
    // Double buffering for overlapping indices calculation between iterations
    data_reader->set_tensor_buffering(2);
  } else {
    // If get_max_batches_inflight() is > than the number of eval batches in the dataset,
    // this will cause the batch tensors to be cached. We need the tensors to be cached in order
    // for the indices to be cached because the index calculation is done in place in these
    // tensors.
    // TODO: if OOM then eval_data_reader->set_tensor_buffering(2)
    data_reader->set_tensor_buffering(data_reader->get_max_batches_inflight());
  }

  const auto data_tensors = data_reader->get_value_tensor_buffers();
  auto &batch_indices = is_train ? train_batch_indices_ : eval_batch_indices_;
  batch_indices.clear();  // remove default
  for (size_t i = 0; i < data_tensors.size(); ++i) {
    batch_indices.emplace_back(model_, data_tensors.at(i), resource_manager_,
                               get_batch_size(is_train), embedding_params_.slot_size_array,
                               embedding_params_.max_num_frequent_categories,
                               embedding_params_.communication_type);
  }
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::forward(bool is_train) {
  size_t local_gpu_count = resource_manager_->get_local_gpu_count();

// Index calculations
#pragma omp parallel for num_threads(local_gpu_count)
  for (size_t i = 0; i < local_gpu_count; i++) {
    auto &gpu = get_local_gpu(i);
    CudaDeviceContext context(gpu.get_device_id());

    index_calculation(is_train, i);
    infreq_model_forward(i);
    freq_forward(is_train, i, true);
    infreq_network_forward(is_train, i);
  }
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::backward() {
  size_t local_gpu_count = resource_manager_->get_local_gpu_count();

#pragma omp parallel for num_threads(local_gpu_count)
  for (size_t i = 0; i < local_gpu_count; i++) {
    auto cur_device = get_local_gpu(i).get_device_id();
    CudaDeviceContext context(cur_device);

    freq_backward(i);
    infreq_network_backward(i);
    infreq_model_backward(i);
  }
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::update_params() {
  size_t local_gpu_count = resource_manager_->get_local_gpu_count();

#pragma omp parallel for num_threads(local_gpu_count)
  for (size_t i = 0; i < local_gpu_count; i++) {
    auto cur_device = get_local_gpu(i).get_device_id();
    CudaDeviceContext context(cur_device);

    freq_update_params(i);
  }
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::init_params() {
  // TODO: create init_params()
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::load_parameters(
    std::string sparse_model, const DataSourceParams &data_source_params) {
  // TODO: create load_parameters()
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::dump_parameters(
    std::string sparse_model, const DataSourceParams &data_source_params) const {
  // TODO: create dump_parameters()
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::set_learning_rate(float lr) {
  HCTR_OWN_THROW(Error_t::WrongInput, "HybridSparseEmbedding only supports GPU LR scheduler");
}

template <typename dtype, typename emtype>
GpuLearningRateSchedulers HybridSparseEmbedding<dtype, emtype>::get_learning_rate_schedulers()
    const {
  return lr_scheds_;
}

template <typename dtype, typename emtype>
size_t HybridSparseEmbedding<dtype, emtype>::get_params_num() const {
  return 0;
}

template <typename dtype, typename emtype>
size_t HybridSparseEmbedding<dtype, emtype>::get_vocabulary_size() const {
  // TODO: create get_vocabulary_size()
  return 0;
}

template <typename dtype, typename emtype>
size_t HybridSparseEmbedding<dtype, emtype>::get_max_vocabulary_size() const {
  // TODO: create get_max_vocabulary_size()
  return 0;
}

template <typename dtype, typename emtype>
std::vector<TensorBag2> HybridSparseEmbedding<dtype, emtype>::get_train_output_tensors() const {
  return tensors_to_bags(train_output_tensors_);
}

template <typename dtype, typename emtype>
std::vector<TensorBag2> HybridSparseEmbedding<dtype, emtype>::get_evaluate_output_tensors() const {
  return tensors_to_bags(evaluate_output_tensors_);
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::assign_input_tensors(bool is_train, size_t batch_size,
                                                                size_t inflight_id, bool cached) {
  if (is_train) {
    train_inflight_id_ = inflight_id;
    current_train_batch_size_ = batch_size;
    current_train_batch_cached_ = cached;
  } else {
    eval_inflight_id_ = inflight_id;
    current_eval_batch_size_ = batch_size;
    current_eval_batch_cached_ = cached;
  }
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::index_calculation(bool is_train, int i) {
  int cur_device = get_local_gpu(i).get_device_id();
  CudaDeviceContext context(cur_device);
  auto &gpu = get_local_gpu(i);
  hipStream_t stream = gpu.get_stream();

  auto &batch_indices = is_train ? train_batch_indices_.at(train_inflight_id_)
                                 : eval_batch_indices_.at(eval_inflight_id_);

  if (is_train) {
    if (!current_train_batch_cached_) {
      batch_indices.compute(i, current_train_batch_size_, stream);
    }
  } else {  // eval
    if (!current_eval_batch_cached_) {
      batch_indices.compute(i, current_eval_batch_size_, stream);
    }
  }

  // We don't copy the sparse tensor since all the required data are already in the
  // Data type and indices
  get_frequent_embedding(i).set_current_indices(&batch_indices.get_frequent(i));
  get_infrequent_embedding(i).set_current_indices(&batch_indices.get_infrequent(i));
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::freq_forward(bool is_train, int i,
                                                        bool is_first_eval_batch) {
  int cur_device = get_local_gpu(i).get_device_id();
  auto &gpu = get_local_gpu(i);
  CudaDeviceContext context(cur_device);
  hipStream_t stream = gpu.get_stream();

  auto &output = (is_train) ? train_output_tensors_[i] : evaluate_output_tensors_[i];
  if (embedding_params_.communication_type == CommunicationType::NVLink_SingleNode) {
    if (is_train) {
      frequent_embeddings_single_node_[i].forward_model(stream);
    } else {
      if (is_first_eval_batch) {
        frequent_embeddings_single_node_[i].forward_model_eval(stream);
      }
    }
    gpu_barrier_->sync_all_gpus(stream, i);

    frequent_embeddings_single_node_[i].forward_network(output.get_ptr(), stream);
  }
  if (embedding_params_.communication_type == CommunicationType::IB_NVLink ||
      embedding_params_.communication_type == CommunicationType::IB_NVLink_Hier) {
    frequent_embeddings_multi_node_[i].forward_network(output.get_ptr(), stream);
  }
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::freq_backward(int i) {
  int cur_device = get_local_gpu(i).get_device_id();
  CudaDeviceContext context(cur_device);
  auto &gpu = get_local_gpu(i);
  hipStream_t stream = gpu.get_stream();

  if (frequent_embeddings_single_node_.size()) {
    frequent_embeddings_single_node_[i].local_reduce(train_output_tensors_[i].get_ptr(), stream);
  } else {
    frequent_embeddings_multi_node_[i].local_reduce(train_output_tensors_[i].get_ptr(), stream);
    if (!grouped_all_reduce_) {
      frequent_embeddings_multi_node_[i].communicate(stream);
    }
  }
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::freq_update_params(int i) {
  int cur_device = get_local_gpu(i).get_device_id();
  CudaDeviceContext context(cur_device);
  float *dev_lr = lr_scheds_[i]->get_learning_rate();
  float scale = opt_params_[i].scaler;
  auto &gpu = get_local_gpu(i);
  hipStream_t stream = gpu.get_stream();

  if (embedding_params_.communication_type != CommunicationType::NVLink_SingleNode) {
    frequent_embeddings_multi_node_[i].update_model(dev_lr, scale, stream);
  }
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::infreq_model_forward(int i) {
  int cur_device = get_local_gpu(i).get_device_id();
  auto &gpu = get_local_gpu(i);
  CudaDeviceContext context(cur_device);
  hipStream_t stream = gpu.get_stream();

  if (embedding_params_.communication_type == CommunicationType::IB_NVLink) {
    infrequent_embeddings_ib_nvlink_[i].forward_model(
        infrequent_embeddings_ib_nvlink_[i].infrequent_forward_comm_buffers_->send_buffer.get_ptr(),
        stream);
  } else if (embedding_params_.communication_type == CommunicationType::IB_NVLink_Hier) {
    infrequent_embeddings_ib_nvlink_hier_[i].calculate_model_indices_sizes_from_offsets(stream);
    infrequent_embeddings_ib_nvlink_hier_[i].calculate_network_indices_sizes_from_offsets(stream);
    infrequent_embeddings_ib_nvlink_hier_[i].infrequent_forward_comms_->update_sizes(stream);
    infrequent_embeddings_ib_nvlink_hier_[i].fused_intra_forward_model(
        infrequent_embeddings_ib_nvlink_hier_[i]
            .infrequent_forward_comm_buffers_->send_buffer_ptrs.get_ptr(),
        stream);
    infrequent_embeddings_ib_nvlink_hier_[i].infrequent_forward_comms_->initiate_communication(
        stream);
  }
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::infreq_network_forward(bool is_train, int i) {
  int cur_device = get_local_gpu(i).get_device_id();
  auto &gpu = get_local_gpu(i);
  CudaDeviceContext context(cur_device);
  hipStream_t stream = gpu.get_stream();

  auto &output = (is_train) ? train_output_tensors_[i] : evaluate_output_tensors_[i];
  if (embedding_params_.communication_type == CommunicationType::IB_NVLink) {
    infrequent_embeddings_ib_nvlink_[i].infrequent_forward_comms_->communicate(stream);
    infrequent_embeddings_ib_nvlink_[i].forward_network(
        infrequent_embeddings_ib_nvlink_[i].infrequent_forward_comm_buffers_->recv_buffer.get_ptr(),
        output.get_ptr(), stream);
  } else if (embedding_params_.communication_type == CommunicationType::IB_NVLink_Hier) {
    infrequent_embeddings_ib_nvlink_hier_[i].infrequent_forward_comms_->wait_completion(stream);
    infrequent_embeddings_ib_nvlink_hier_[i].hier_forward_network(
        infrequent_embeddings_ib_nvlink_hier_[i]
            .infrequent_forward_comm_buffers_->recv_buffer.get_ptr(),
        output.get_ptr(), stream);
  } else {
    infrequent_embeddings_single_node_[i].forward_network_direct(is_train, stream);
  }
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::global_barrier(bool is_train, int i) {
  int cur_device = get_local_gpu(i).get_device_id();
  auto &gpu = get_local_gpu(i);
  CudaDeviceContext context(cur_device);
  hipStream_t stream = gpu.get_stream();

  if (embedding_params_.communication_type == CommunicationType::IB_NVLink_Hier) {
    if (!is_train) {
      HCTR_LIB_THROW(ncclAllReduce((const void *)d_barrier_store_[i].get_ptr(),
                                   d_barrier_store_[i].get_ptr(), sizeof(uint32_t),
                                   NcclDataType<uint32_t>::getType(), ncclSum,
                                   get_local_gpu(i).get_nccl(), stream));
    }
  }
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::infreq_network_backward(int i) {
  int cur_device = get_local_gpu(i).get_device_id();
  CudaDeviceContext context(cur_device);
  auto &gpu = get_local_gpu(i);
  hipStream_t stream = gpu.get_stream();

  if (embedding_params_.communication_type == CommunicationType::IB_NVLink) {
    infrequent_embeddings_ib_nvlink_[i].update_network(
        train_output_tensors_[i].get_ptr(),
        infrequent_embeddings_ib_nvlink_[i]
            .infrequent_backward_comm_buffers_->send_buffer.get_ptr(),
        stream);
  }
  if (embedding_params_.communication_type == CommunicationType::IB_NVLink_Hier) {
    infrequent_embeddings_ib_nvlink_hier_[i].infrequent_backward_comms_->update_sizes(stream);
    infrequent_embeddings_ib_nvlink_hier_[i].fused_intra_update_network(
        train_output_tensors_[i].get_ptr(),
        infrequent_embeddings_ib_nvlink_hier_[i]
            .infrequent_backward_comm_buffers_->send_buffer_ptrs.get_ptr(),
        stream);
  }
}

// Everything that involves network and can be better overlapped with compute
template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::infreq_model_backward(int i) {
  int cur_device = get_local_gpu(i).get_device_id();
  CudaDeviceContext context(cur_device);
  auto &gpu = get_local_gpu(i);
  hipStream_t stream = gpu.get_stream();
  float *dev_lr = lr_scheds_[i]->get_learning_rate();
  float scale = opt_params_[i].scaler;

  if (embedding_params_.communication_type == CommunicationType::IB_NVLink) {
    infrequent_embeddings_ib_nvlink_[i].infrequent_backward_comms_->communicate(stream);
    infrequent_embeddings_ib_nvlink_[i].update_model(
        infrequent_embeddings_ib_nvlink_[i]
            .infrequent_backward_comm_buffers_->recv_buffer.get_ptr(),
        dev_lr, scale, stream);
  }

  if (embedding_params_.communication_type == CommunicationType::IB_NVLink_Hier) {
    infrequent_embeddings_ib_nvlink_hier_[i].infrequent_backward_comms_->communicate(stream);

    infrequent_embeddings_ib_nvlink_hier_[i].hier_update_model(
        infrequent_embeddings_ib_nvlink_hier_[i]
            .infrequent_backward_comm_buffers_->recv_buffer.get_ptr(),
        dev_lr, scale, stream);
  }
  if (embedding_params_.communication_type == CommunicationType::NVLink_SingleNode) {
    // Synchronize all GPUs before pulling the reduced gradients
    gpu_barrier_->sync_all_gpus(stream, i);

    float *dev_lr = lr_scheds_[i]->get_learning_rate();
    float scale = opt_params_[i].scaler;
    frequent_embeddings_single_node_[i].update_model_direct(dev_lr, scale, stream);

    infrequent_embeddings_single_node_[i].update_model_direct(dev_lr, scale, stream);
  }
}

template class HybridSparseEmbedding<uint32_t, __half>;
template class HybridSparseEmbedding<uint32_t, float>;
template class HybridSparseEmbedding<long long, __half>;
template class HybridSparseEmbedding<long long, float>;
}  // namespace HugeCTR
