/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "HugeCTR/include/embeddings/sparse_embedding_functors.hpp"
#include "HugeCTR/include/utils.hpp"

namespace HugeCTR {

template <typename TypeHashKey>
void SparseEmbeddingFunctors::get_update_params_results(
    size_t embedding_vec_size, size_t vocabulary_size,
    const Tensors2<float> &hash_table_value_tensors,
    const std::vector<std::shared_ptr<HashTable<TypeHashKey, size_t>>> &hash_tables,
    Tensor2<TypeHashKey> &hash_table_key, Tensor2<float> &hash_table_value,
    const ResourceManager &resource_manager) {
  CudaDeviceContext context;

  size_t local_gpu_count = resource_manager.get_local_gpu_count();

  // memory allocation
  std::unique_ptr<size_t[]> count(new size_t[local_gpu_count]);
  size_t total_count = 0;
  for (size_t id = 0; id < local_gpu_count; id++) {
    const auto &local_gpu = resource_manager.get_local_gpu(id);
    context.set_device(local_gpu->get_device_id());
    if ((count[id] = hash_tables[id]->get_value_head(local_gpu->get_stream())) !=
        hash_tables[id]->get_size(local_gpu->get_stream())) {
      HCTR_LOG_S(ERROR, WORLD) << "hashtable: get_value_head()="
                               << hash_tables[id]->get_value_head(local_gpu->get_stream())
                               << ", get_size()="
                               << hash_tables[id]->get_size(local_gpu->get_stream()) << std::endl;
      HCTR_OWN_THROW(Error_t::WrongInput,
                     "Error: hash_table get_value_head() size not equal to get_size()");
    }
    total_count += count[id];

#ifndef NDEBUG
    HCTR_LOG_S(DEBUG, WORLD) << "GPU[" << id << "]: number of <key,value> pairs:" << count[id]
                             << std::endl;
#endif
  }

#ifndef NDEBUG
  HCTR_LOG_S(DEBUG, WORLD) << "Total number of <key,value> pairs:" << total_count << std::endl;
#endif

  if (total_count > (size_t)vocabulary_size) {
    HCTR_OWN_THROW(Error_t::WrongInput,
                   "Error: required download size is larger than hash table vocabulary_size");
  }

  std::unique_ptr<TypeHashKey *[]> d_hash_table_key(new TypeHashKey *[local_gpu_count]);
  std::unique_ptr<size_t *[]> d_hash_table_value_index(new size_t *[local_gpu_count]);
  std::unique_ptr<float *[]> d_hash_table_value(new float *[local_gpu_count]);
  std::unique_ptr<size_t *[]> d_dump_counter(new size_t *[local_gpu_count]);
  for (size_t id = 0; id < local_gpu_count; id++) {
    if (count[id] == 0) {
      continue;
    }

    context.set_device(resource_manager.get_local_gpu(id)->get_device_id());

    HCTR_LIB_THROW(hipMalloc(&d_hash_table_key[id], count[id] * sizeof(TypeHashKey)));
    HCTR_LIB_THROW(hipMalloc(&d_hash_table_value_index[id], count[id] * sizeof(size_t)));
    HCTR_LIB_THROW(
        hipMalloc(&d_hash_table_value[id], count[id] * embedding_vec_size * sizeof(float)));
    HCTR_LIB_THROW(hipMalloc(&d_dump_counter[id], count[id] * sizeof(size_t)));
  }

  // dump hash table on GPU
  for (size_t id = 0; id < local_gpu_count; id++) {
    if (count[id] == 0) {
      continue;
    }

    const auto &local_gpu = resource_manager.get_local_gpu(id);
    context.set_device(local_gpu->get_device_id());

    hash_tables[id]->dump(d_hash_table_key[id], d_hash_table_value_index[id], d_dump_counter[id],
                          local_gpu->get_stream());

    get_hash_value(count[id], embedding_vec_size, d_hash_table_value_index[id],
                   hash_table_value_tensors[id].get_ptr(), d_hash_table_value[id],
                   local_gpu->get_stream());
  }

  // sync wait
  sync_all_gpus(resource_manager);

  // memcpy from GPU to CPU memory
  size_t key_offset = 0;
  size_t value_offset = 0;
  for (size_t id = 0; id < local_gpu_count; id++) {
    if (count[id] == 0) {
      continue;
    }

    context.set_device(resource_manager.get_local_gpu(id)->get_device_id());

    HCTR_LIB_THROW(hipMemcpy(hash_table_key.get_ptr() + key_offset, d_hash_table_key[id],
                              count[id] * sizeof(TypeHashKey), hipMemcpyDeviceToHost));
    key_offset += count[id];

    HCTR_LIB_THROW(hipMemcpy(hash_table_value.get_ptr() + value_offset, d_hash_table_value[id],
                              count[id] * embedding_vec_size * sizeof(float),
                              hipMemcpyDeviceToHost));
    value_offset += count[id] * embedding_vec_size;
  }

  for (size_t id = 0; id < local_gpu_count; id++) {
    if (count[id] == 0) {
      continue;
    }

    context.set_device(resource_manager.get_local_gpu(id)->get_device_id());

    HCTR_LIB_THROW(hipFree(d_hash_table_key[id]));
    HCTR_LIB_THROW(hipFree(d_hash_table_value_index[id]));
    HCTR_LIB_THROW(hipFree(d_hash_table_value[id]));
    HCTR_LIB_THROW(hipFree(d_dump_counter[id]));
  }

#ifdef ENABLE_MPI

  if (resource_manager.get_num_process() > 1) {
    std::unique_ptr<int> displs(new int(resource_manager.get_num_process()));
    std::unique_ptr<int> recv_count(new int(resource_manager.get_num_process()));
    HCTR_MPI_THROW(
        MPI_Gather(&total_count, 1, MPI_INT, recv_count.get(), 1, MPI_INT, 0, MPI_COMM_WORLD));

    if (resource_manager.is_master_process()) {
      displs.get()[0] = 0;
      for (int i = 1; i < resource_manager.get_num_process(); i++) {
        displs.get()[i] = displs.get()[i - 1] + recv_count.get()[i - 1];
      }
    }

    std::unique_ptr<int> displs_key(new int(resource_manager.get_num_process()));
    std::unique_ptr<int> recv_count_key(new int(resource_manager.get_num_process()));
    if (resource_manager.is_master_process()) {
      for (int i = 0; i < resource_manager.get_num_process(); i++) {
        recv_count_key.get()[i] = recv_count.get()[i] * sizeof(TypeHashKey);
        displs_key.get()[i] = displs.get()[i] * sizeof(TypeHashKey);
      }
    }

    HCTR_MPI_THROW(MPI_Gatherv(hash_table_key.get_ptr(), total_count * sizeof(TypeHashKey),
                               MPI_CHAR, hash_table_key.get_ptr(), recv_count_key.get(),
                               displs_key.get(), MPI_CHAR, 0, MPI_COMM_WORLD));

    std::unique_ptr<int> displs_value(new int(resource_manager.get_num_process()));
    std::unique_ptr<int> recv_count_value(new int(resource_manager.get_num_process()));
    if (resource_manager.is_master_process()) {
      for (int i = 0; i < resource_manager.get_num_process(); i++) {
        recv_count_value.get()[i] = recv_count.get()[i] * embedding_vec_size * sizeof(float);
        displs_value.get()[i] = displs.get()[i] * embedding_vec_size * sizeof(float);
      }
    }

    HCTR_MPI_THROW(MPI_Gatherv(hash_table_value.get_ptr(),
                               total_count * embedding_vec_size * sizeof(float), MPI_CHAR,
                               hash_table_value.get_ptr(), recv_count_value.get(),
                               displs_value.get(), MPI_CHAR, 0, MPI_COMM_WORLD));
  }
#endif

  return;
}

template void SparseEmbeddingFunctors::get_update_params_results<unsigned int>(
    size_t embedding_vec_size, size_t vocabulary_size,
    const Tensors2<float> &hash_table_value_tensors,
    const std::vector<std::shared_ptr<HashTable<unsigned int, size_t>>> &hash_tables,
    Tensor2<unsigned int> &hash_table_key, Tensor2<float> &hash_table_value,
    const ResourceManager &resource_manager);

template void SparseEmbeddingFunctors::get_update_params_results<long long>(
    size_t embedding_vec_size, size_t vocabulary_size,
    const Tensors2<float> &hash_table_value_tensors,
    const std::vector<std::shared_ptr<HashTable<long long, size_t>>> &hash_tables,
    Tensor2<long long> &hash_table_key, Tensor2<float> &hash_table_value,
    const ResourceManager &resource_manager);

}  // namespace HugeCTR