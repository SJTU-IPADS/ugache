#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <base/debug/logger.hpp>
#include <embeddings/sparse_embedding_functors.hpp>
#include <utils.hpp>

namespace HugeCTR {
template <typename TypeEmbeddingComp>
std::vector<Tensors2<TypeEmbeddingComp>> SparseEmbeddingFunctors::get_opt_states(
    const std::vector<OptimizerTensor<TypeEmbeddingComp>>& opt_tensors_, Optimizer_t optimizer_type,
    size_t local_gpu_count) {
  std::vector<Tensors2<TypeEmbeddingComp>> opt_states;
  opt_states.resize(local_gpu_count);

  for (size_t i = 0; i < local_gpu_count; ++i) {
    switch (optimizer_type) {
      case Optimizer_t::Adam:  // adam
      {
        opt_states[i].push_back(opt_tensors_[i].opt_m_tensors_);
        opt_states[i].push_back(opt_tensors_[i].opt_v_tensors_);
        break;
      }

      case Optimizer_t::AdaGrad:  // nesterov
      {
        opt_states[i].push_back(opt_tensors_[i].opt_accm_tensors_);
        break;
      }
      case Optimizer_t::MomentumSGD:  // momentum_sgd
      {
        opt_states[i].push_back(opt_tensors_[i].opt_momentum_tensors_);
        break;
      }

      case Optimizer_t::Nesterov:  // nesterov
      {
        opt_states[i].push_back(opt_tensors_[i].opt_accm_tensors_);
        break;
      }

      case Optimizer_t::SGD:
        break;

      default:
        throw std::runtime_error(
            std::string("[HCDEBUG][ERROR] Runtime error: Invalid optimizer type\n"));
    }
  }

  std::vector<Tensors2<TypeEmbeddingComp>> transpose_opt_states;
  if (opt_states[0].size() > 0) {
    transpose_opt_states.resize(opt_states[0].size());
    for (size_t i = 0; i < opt_states[0].size(); ++i) {
      transpose_opt_states[i].resize(opt_states.size());
      for (size_t j = 0; j < opt_states.size(); ++j) {
        transpose_opt_states[i][j] = opt_states[j][i];
      }
    }
  }
  return transpose_opt_states;
}

template <typename TypeEmbeddingComp>
void SparseEmbeddingFunctors::dump_opt_states(
    std::ofstream& stream, std::string& write_path, const DataSourceParams& data_source_params,
    const ResourceManager& resource_manager, std::vector<Tensors2<TypeEmbeddingComp>>& opt_states) {
  size_t local_gpu_count = resource_manager.get_local_gpu_count();

  bool hdfs_append_flag = false;
  CudaDeviceContext context;
  for (auto& opt_state : opt_states) {
    size_t total_size = 0;
    for (size_t id = 0; id < local_gpu_count; id++) {
      total_size += opt_state[id].get_size_in_bytes();
    }
    size_t max_size = total_size;

#ifdef ENABLE_MPI
    bool is_master_process = resource_manager.is_master_process();
    HCTR_MPI_THROW(MPI_Reduce(is_master_process ? MPI_IN_PLACE : &max_size, &max_size,
                              sizeof(size_t), MPI_CHAR, MPI_MAX,
                              resource_manager.get_master_process_id(), MPI_COMM_WORLD));
#endif

    std::unique_ptr<char[]> h_opt_state(new char[max_size]);
    size_t offset = 0;
    for (size_t id = 0; id < local_gpu_count; id++) {
      size_t local_size = opt_state[id].get_size_in_bytes();
      auto& local_gpu = resource_manager.get_local_gpu(id);
      context.set_device(local_gpu->get_device_id());
      HCTR_LIB_THROW(hipMemcpyAsync(h_opt_state.get() + offset, opt_state[id].get_ptr(),
                                     local_size, hipMemcpyDeviceToHost, local_gpu->get_stream()));
      offset += local_size;
    }
    sync_all_gpus(resource_manager);
    int pid = resource_manager.get_process_id();
    if (resource_manager.is_master_process()) {
      HCTR_LOG_S(INFO, WORLD) << "Rank" << pid << ": Write optimzer state to file" << std::endl;
      if (data_source_params.type == DataSourceType_t::HDFS) {
        auto hs = data_source_params.create_unique();
        if (!hdfs_append_flag) {
          hs->write(write_path, h_opt_state.get(), total_size, true);
          hdfs_append_flag = true;
        } else {
          hs->write(write_path, h_opt_state.get(), total_size, false);
        }
      } else if (data_source_params.type == DataSourceType_t::Local) {
        // TODO: Move to self-contained DataSourceBackend implementation.
        stream.write(h_opt_state.get(), total_size);
      } else {
        HCTR_OWN_THROW(Error_t::WrongInput, "Filesystem not supported yet.");
      }
    }
#ifdef ENABLE_MPI
    else {
      HCTR_LOG_S(INFO, WORLD) << "Rank" << pid << ": Send optimzer state to master node"
                              << std::endl;
      int tag = (pid << 8) | 0xBA;
      HCTR_MPI_THROW(MPI_Send(h_opt_state.get(), total_size, MPI_CHAR,
                              resource_manager.get_master_process_id(), tag, MPI_COMM_WORLD));
    }

    if (resource_manager.is_master_process()) {
      for (int r = 1; r < resource_manager.get_num_process(); r++) {
        HCTR_LOG_S(INFO, WORLD) << "Rank" << pid << ": Recv optimzer state from rank" << r
                                << ", and write to file" << std::endl;
        int tag = (r << 8) | 0xBA;
        int recv_size = 0;
        MPI_Status status;
        HCTR_MPI_THROW(MPI_Probe(r, tag, MPI_COMM_WORLD, &status));
        HCTR_MPI_THROW(MPI_Get_count(&status, MPI_CHAR, &recv_size));
        HCTR_MPI_THROW(MPI_Recv(h_opt_state.get(), recv_size, MPI_CHAR, r, tag, MPI_COMM_WORLD,
                                MPI_STATUS_IGNORE));
        if (data_source_params.type == DataSourceType_t::HDFS) {
          auto hs = data_source_params.create_unique();
          if (!hdfs_append_flag) {
            hs->write(write_path, h_opt_state.get(), recv_size, true);
            hdfs_append_flag = true;
          } else {
            hs->write(write_path, h_opt_state.get(), recv_size, false);
          }
        } else if (data_source_params.type == DataSourceType_t::Local) {
          // TODO: Move to self-contained DataSourceBackend implementation.
          stream.write(h_opt_state.get(), recv_size);
        } else {
          HCTR_OWN_THROW(Error_t::WrongInput, "Filesystem not supported yet.");
        }
      }
    }
#endif
    HCTR_LOG(INFO, ROOT, "Done\n");
  }
}

template <typename TypeEmbeddingComp>
void SparseEmbeddingFunctors::load_opt_states(std::ifstream& stream, std::string& read_path,
                                              const ResourceManager& resource_manager,
                                              std::vector<Tensors2<TypeEmbeddingComp>>& opt_states,
                                              const DataSourceParams& data_source_params) {
  size_t local_gpu_count = resource_manager.get_local_gpu_count();

  CudaDeviceContext context;
  size_t hdfs_cursor = 0;
  for (auto& opt_state : opt_states) {
    size_t total_size = 0;
    for (size_t id = 0; id < local_gpu_count; id++) {
      total_size += opt_state[id].get_size_in_bytes();
    }
    int pid = resource_manager.get_process_id();

    auto h2d_op = [&opt_state, &resource_manager, &context](char* h_opt_state) {
      size_t offset = 0;
      for (size_t id = 0; id < resource_manager.get_local_gpu_count(); id++) {
        size_t local_size = opt_state[id].get_size_in_bytes();
        auto& local_gpu = resource_manager.get_local_gpu(id);
        context.set_device(local_gpu->get_device_id());
        HCTR_LIB_THROW(hipMemcpyAsync(opt_state[id].get_ptr(), h_opt_state + offset, local_size,
                                       hipMemcpyHostToDevice, local_gpu->get_stream()));
        offset += local_size;
      }
    };

    std::unique_ptr<size_t[]> proc_sizes(new size_t[resource_manager.get_num_process()]);
    proc_sizes[0] = total_size;
#ifdef ENABLE_MPI
    HCTR_MPI_THROW(MPI_Gather(&total_size, sizeof(size_t), MPI_CHAR, proc_sizes.get(),
                              sizeof(size_t), MPI_CHAR, 0, MPI_COMM_WORLD));
#endif

    if (resource_manager.is_master_process()) {
      size_t sum_sizes = 0;
      size_t max_size = 0;
      for (int i = 0; i < resource_manager.get_num_process(); ++i) {
        sum_sizes += proc_sizes[i];
        if (proc_sizes[i] > max_size) {
          max_size = proc_sizes[i];
        }
      }
      std::unique_ptr<char[]> h_opt_state(new char[max_size]);
      HCTR_LOG_S(INFO, WORLD) << "Rank" << pid << ": Read optimzer state from file" << std::endl;
      if (data_source_params.type == DataSourceType_t::HDFS) {
        auto hs = data_source_params.create_unique();
        hs->read(read_path, h_opt_state.get(), total_size, hdfs_cursor);
        hdfs_cursor += total_size;
      } else if (data_source_params.type == DataSourceType_t::Local) {
        // TODO: Move to self-contained DataSourceBackend implementation.
        size_t cur_pos = stream.tellg();
        stream.seekg(0, stream.end);
        size_t remaining_file_size = stream.tellg() - cur_pos;
        if (remaining_file_size < sum_sizes) {
          HCTR_OWN_THROW(Error_t::WrongInput,
                         "optimizer state file size is incompatible with the embedding!");
        }
        stream.seekg(cur_pos);
        stream.read(h_opt_state.get(), total_size);
      } else {
        HCTR_OWN_THROW(Error_t::WrongInput, "Filesystem not supported yet.");
      }

      h2d_op(h_opt_state.get());
      sync_all_gpus(resource_manager);

#ifdef ENABLE_MPI
      for (int r = 1; r < resource_manager.get_num_process(); r++) {
        HCTR_LOG_S(INFO, WORLD) << "Rank" << pid << ": Read from file"
                                << ", and send optimzer state to rank" << r << std::endl;
        if (data_source_params.type == DataSourceType_t::HDFS) {
          auto hs = data_source_params.create_unique();
          hs->read(read_path, h_opt_state.get(), proc_sizes[r], hdfs_cursor);
          hdfs_cursor += proc_sizes[r];
        } else if (data_source_params.type == DataSourceType_t::Local) {
          // TODO: Move to self-contained DataSourceBackend implementation.
          stream.read(h_opt_state.get(), proc_sizes[r]);
        } else {
          HCTR_OWN_THROW(Error_t::WrongInput, "Filesystem not supported yet.");
        }
        int tag = (r << 8) | 0xAB;
        HCTR_MPI_THROW(
            MPI_Send(h_opt_state.get(), proc_sizes[r], MPI_CHAR, r, tag, MPI_COMM_WORLD));
      }
#endif
    }
#ifdef ENABLE_MPI
    else {
      HCTR_LOG_S(INFO, WORLD) << "Rank" << pid
                              << ": Recv optimzer state from master node, and write to GPUs"
                              << std::endl;
      int mid = resource_manager.get_master_process_id();
      int tag = (pid << 8) | 0xAB;
      int recv_size = 0;
      MPI_Status status;
      HCTR_MPI_THROW(MPI_Probe(mid, tag, MPI_COMM_WORLD, &status));
      HCTR_MPI_THROW(MPI_Get_count(&status, MPI_CHAR, &recv_size));
      std::unique_ptr<char[]> h_opt_state(new char[recv_size]);
      if (data_source_params.type == DataSourceType_t::HDFS) {
        auto hs = data_source_params.create_unique();
        hs->read(read_path, h_opt_state.get(), recv_size, hdfs_cursor);
        hdfs_cursor += recv_size;
      } else if (data_source_params.type == DataSourceType_t::Local) {
        // TODO: Move to self-contained DataSourceBackend implementation.
        stream.read(h_opt_state.get(), recv_size);
      } else {
        HCTR_OWN_THROW(Error_t::WrongInput, "Filesystem not supported yet.");
      }
      HCTR_MPI_THROW(MPI_Recv(h_opt_state.get(), recv_size, MPI_CHAR, mid, tag, MPI_COMM_WORLD,
                              MPI_STATUS_IGNORE));
      h2d_op(h_opt_state.get());
      sync_all_gpus(resource_manager);
    }
#endif
    HCTR_LOG(INFO, ROOT, "Done\n");
  }
}
template std::vector<Tensors2<float>> SparseEmbeddingFunctors::get_opt_states(
    const std::vector<OptimizerTensor<float>>& opt_tensors_, Optimizer_t optimizer_type,
    size_t local_gpu_count);

template std::vector<Tensors2<__half>> SparseEmbeddingFunctors::get_opt_states(
    const std::vector<OptimizerTensor<__half>>& opt_tensors_, Optimizer_t optimizer_type,
    size_t local_gpu_count);

template void SparseEmbeddingFunctors::dump_opt_states<float>(
    std::ofstream& stream, std::string& write_path, const DataSourceParams& data_source_params,
    const ResourceManager& resource_manager, std::vector<Tensors2<float>>& opt_states);

template void SparseEmbeddingFunctors::dump_opt_states<__half>(
    std::ofstream& stream, std::string& write_path, const DataSourceParams& data_source_params,
    const ResourceManager& resource_manager, std::vector<Tensors2<__half>>& opt_states);

template void SparseEmbeddingFunctors::load_opt_states<float>(
    std::ifstream& stream, std::string& read_path, const ResourceManager& resource_manager,
    std::vector<Tensors2<float>>& opt_states, const DataSourceParams& data_source_params);

template void SparseEmbeddingFunctors::load_opt_states<__half>(
    std::ifstream& stream, std::string& read_path, const ResourceManager& resource_manager,
    std::vector<Tensors2<__half>>& opt_states, const DataSourceParams& data_source_params);

}  // namespace HugeCTR
