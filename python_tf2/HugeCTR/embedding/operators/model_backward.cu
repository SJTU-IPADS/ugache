/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "HugeCTR/include/utils.hpp"
#include "generic_lookup.cuh"
#include "model_backward.hpp"
#include "multi_to_one_reduce.cuh"
#include "utils.cuh"
namespace embedding {

ModelBackward::ModelBackward(std::shared_ptr<CoreResourceManager> core, int num_gpus,
                             int num_local_embedding, const std::vector<int>& h_local_hotness_list,
                             const std::vector<int>& h_local_ev_size_list, int universal_batch_size,
                             int max_ev_size, int num_sms)
    : core_(core),
      num_gpus_(num_gpus),
      num_local_embedding_(num_local_embedding),
      max_ev_size_(max_ev_size),
      num_sms_(num_sms) {
  std::vector<int> num_unique_key_list;
  for (int i = 0; i < num_local_embedding; ++i) {
    num_unique_key_list.push_back(h_local_hotness_list[i] * h_local_ev_size_list[i]);
  }

  int max_unique_key_ev_buffer_size =
      std::accumulate(num_unique_key_list.begin(), num_unique_key_list.end(), 0);
  HugeCTR::CudaDeviceContext ctx(core_->get_device_id());

  auto buffer_ptr = GetBuffer(core);
  grad_ev_ = buffer_ptr->reserve({universal_batch_size, max_unique_key_ev_buffer_size},
                                 DeviceType::GPU, TensorScalarType::Float32);
  partial_grad_ev_ = buffer_ptr->reserve({num_sms_ * 4 * max_ev_size_}, DeviceType::GPU,
                                         TensorScalarType::Float32);
  partial_key_ = buffer_ptr->reserve({num_sms_ * 4}, DeviceType::GPU, TensorScalarType::UInt32);
  partial_ev_length_ =
      buffer_ptr->reserve({num_sms_ * 4}, DeviceType::GPU, TensorScalarType::Int32);
  partial_dst_offset_array_ =
      buffer_ptr->reserve({num_sms_ * 4}, DeviceType::GPU, TensorScalarType::UInt32);

  buffer_ptr->allocate();
}

void ModelBackward::compute(const TensorList& model_comm_buffer, const Tensor& unique_dst_idx,
                            const Tensor& sorted_bucket_id_list,
                            const Tensor& sorted_bucket_id_offset, size_t num_unique_key,
                            const Tensor& corrdinate_key, const Tensor& coordinate_wgrad_dst_idx,
                            const Tensor& d_local_ev_size_offset, int batch_size, int max_ev_size,
                            size_t num_model_key, Tensor* grad_ev) {
  HugeCTR::CudaDeviceContext ctx(core_->get_device_id());
  auto stream = core_->get_local_gpu()->get_stream();
  int batch_size_per_gpu = batch_size / num_gpus_;

  hipMemsetAsync(grad_ev_.get(), 0, grad_ev_.nbytes(), core_->get_local_gpu()->get_stream());
  DISPATCH_FLOAT_AND_HALF_FUNCTION(model_comm_buffer.dtype().type(), emb_t, [&] {
    const uint32_t* unique_dst_idx_ptr = unique_dst_idx.get<uint32_t>();
    const emb_t** model_comm_buffer_ptr = model_comm_buffer.get<emb_t>();
    const int* local_ev_offset_list_ptr = d_local_ev_size_offset.get<int>();
    auto stream = core_->get_local_gpu()->get_stream();
    uint32_t* corrdinate_key_ptr = (uint32_t*)corrdinate_key.get();
    uint32_t* sorted_bucket_id_list_ptr = (uint32_t*)sorted_bucket_id_list.get();
    uint32_t* coordinate_wgrad_dst_idx_ptr = (uint32_t*)coordinate_wgrad_dst_idx.get();
    auto partial_grad_ev_ptr = (float*)partial_grad_ev_.get();
    auto partial_key_ptr = (uint32_t*)partial_key_.get();
    auto partial_ev_length_ptr = (int*)partial_ev_length_.get();
    auto partial_dst_offset_array_ptr = (uint32_t*)partial_dst_offset_array_.get();
    float* grad_ev_ptr = grad_ev_.get<float>();

    auto multi_to_one_desc_first_stage = make_MultiToOne_reduce<emb_t, float>(
        num_model_key, [=] __device__(int i) { return corrdinate_key_ptr[i]; },
        [=] __device__(int i) {
          uint32_t src_index = sorted_bucket_id_list_ptr[i];
          int embedding_id = src_index / batch_size;
          return local_ev_offset_list_ptr[embedding_id + 1] -
                 local_ev_offset_list_ptr[embedding_id];
        },
        [=] __device__(int i) {
          auto tmp_index = coordinate_wgrad_dst_idx_ptr[i];
          return unique_dst_idx_ptr[tmp_index + 1] - unique_dst_idx_ptr[tmp_index];
        },
        [=] __device__(int i) { return coordinate_wgrad_dst_idx_ptr[i]; },

        [=] __device__(int i) {
          uint32_t src_index = sorted_bucket_id_list_ptr[i];
          int embedding_id = src_index / batch_size;
          int batch_id = src_index % batch_size;
          int gpu_id = batch_id / batch_size_per_gpu;
          int local_batch_id = batch_id % batch_size_per_gpu;
          int ev_size =
              local_ev_offset_list_ptr[embedding_id + 1] - local_ev_offset_list_ptr[embedding_id];
          return model_comm_buffer_ptr[gpu_id] +
                 batch_size_per_gpu * local_ev_offset_list_ptr[embedding_id] +
                 local_batch_id * ev_size;
        },

        [=] __device__(int i) {
          auto tmp_index = coordinate_wgrad_dst_idx_ptr[i];
          return grad_ev_ptr + unique_dst_idx_ptr[tmp_index];
        });

    auto multi_to_one_desc_second_stage = make_MultiToOne_reduce<float, float>(
        num_model_key, [=] __device__(int i) { return partial_key_ptr[i]; },
        [=] __device__(int i) { return partial_ev_length_ptr[i]; },
        [=] __device__(int i) {
          auto tmp_index = partial_dst_offset_array_ptr[i];
          return unique_dst_idx_ptr[tmp_index + 1] - unique_dst_idx_ptr[tmp_index];
        },
        [=] __device__(int i) { return 1; },

        [=] __device__(int i) { return partial_grad_ev_ptr + i * max_ev_size; },

        [=] __device__(int i) {
          auto tmp_index = partial_dst_offset_array_ptr[i];
          return grad_ev_ptr + unique_dst_idx_ptr[tmp_index];
        });

    multi_to_one_reduce(multi_to_one_desc_first_stage, multi_to_one_desc_second_stage,
                        (float*)partial_grad_ev_.get(), (uint32_t*)partial_key_.get(),
                        (int*)partial_ev_length_.get(), (uint32_t*)partial_dst_offset_array_.get(),
                        num_sms_, max_ev_size, stream);
  });
  *grad_ev = grad_ev_;
}

DPLocalReduce::DPLocalReduce(std::shared_ptr<CoreResourceManager> core, int num_gpus,
                             int num_local_embedding, const std::vector<int>& h_local_hotness_list,
                             const std::vector<int>& h_local_ev_size_list, int universal_batch_size)
    : core_(core), num_gpus_(num_gpus), num_local_embedding_(num_local_embedding) {
  HugeCTR::CudaDeviceContext ctx(core_->get_device_id());
  Device device{DeviceType::GPU};

  max_ev_size_ = *std::max_element(h_local_ev_size_list.begin(), h_local_ev_size_list.end());
  std::vector<int> num_unique_key_list;
  for (int i = 0; i < num_local_embedding; ++i) {
    num_unique_key_list.push_back(h_local_hotness_list[i] * h_local_ev_size_list[i]);
  }

  int max_unique_key_ev_buffer_size =
      std::accumulate(num_unique_key_list.begin(), num_unique_key_list.end(), 0);

  auto buffer_ptr = GetBuffer(core);
  grad_ev_ = buffer_ptr->reserve({universal_batch_size, max_unique_key_ev_buffer_size}, device,
                                 TensorScalarType::Float32);
  buffer_ptr->allocate();
}

void DPLocalReduce::compute(const Tensor& top_grad, const Tensor& unique_dst_idx,
                            const Tensor& sorted_bucket_id_list,
                            const Tensor& sorted_bucket_id_offset, size_t num_unique_key,
                            const Tensor& d_ev_size_offset, int batch_size, int max_ev_size,
                            Tensor* grad_ev) {
  HugeCTR::CudaDeviceContext ctx(core_->get_device_id());
  int batch_size_per_gpu = batch_size / num_gpus_;

  DISPATCH_FLOAT_AND_HALF_FUNCTION(top_grad.dtype().type(), emb_t, [&] {
    auto stream = core_->get_local_gpu()->get_stream();
    const uint32_t* sorted_bucket_id_list_ptr = sorted_bucket_id_list.get<uint32_t>();
    const uint32_t* sorted_bucket_id_offset_ptr = sorted_bucket_id_offset.get<uint32_t>();
    const uint32_t* unique_dst_idx_ptr = unique_dst_idx.get<uint32_t>();
    const int* d_ev_size_offset_ptr = d_ev_size_offset.get<int>();
    const emb_t* top_grad_ptr = top_grad.get<emb_t>();
    float* grad_ev_ptr = grad_ev_.get<float>();

    auto multi_to_one_desc = make_MultiToOne<emb_t, float>(
        num_unique_key, [=] __device__(int i) { return sorted_bucket_id_offset_ptr[i]; },
        [=] __device__(int i) { return 1; },
        [=] __device__(int i) { return unique_dst_idx_ptr[i + 1] - unique_dst_idx_ptr[i]; },
        [=] __device__(int i) {
          int bucket_id = sorted_bucket_id_list_ptr[i];
          int i_lookup = bucket_id / batch_size_per_gpu;
          int b = bucket_id % batch_size_per_gpu;
          int ev_size = d_ev_size_offset_ptr[i_lookup + 1] - d_ev_size_offset_ptr[i_lookup];

          return top_grad_ptr + batch_size_per_gpu * d_ev_size_offset_ptr[i_lookup] + b * ev_size;
        },
        [=] __device__(int i) { return grad_ev_ptr + unique_dst_idx_ptr[i]; });
    copy_multi_to_one(multi_to_one_desc, max_ev_size, stream);
  });

  *grad_ev = grad_ev_;
}
}  // namespace embedding
