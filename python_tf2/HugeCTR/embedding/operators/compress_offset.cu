#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "HugeCTR/embedding/common.hpp"
#include "HugeCTR/include/utils.hpp"
#include "compress_offset.hpp"
#include "generic_lookup.cuh"
namespace embedding {

__global__ void compress_offset_kernel(const uint32_t *offset, int num, int stride,
                                       uint32_t *compressed_offset) {
  int thread_cnt = blockDim.x * blockDim.y;

  for (int tid = threadIdx.x + threadIdx.y * blockDim.x; tid < num; tid += thread_cnt) {
    compressed_offset[tid] = offset[tid * stride];
  }
}

CompressOffset::CompressOffset(std::shared_ptr<CoreResourceManager> core, int num_compressed_offset)
    : core_(core), num_compressed_offset_(num_compressed_offset) {
  HugeCTR::CudaDeviceContext ctx(core_->get_device_id());

  auto buffer_ptr = GetBuffer(core);
  compressed_offset_ =
      buffer_ptr->reserve({num_compressed_offset}, DeviceType::GPU, TensorScalarType::UInt32);
  buffer_ptr->allocate();
}

void CompressOffset::compute(const Tensor &offset, int stride, Tensor *compressed_offset) {
  HugeCTR::CudaDeviceContext ctx(core_->get_device_id());
  auto stream = core_->get_local_gpu()->get_stream();

  dim3 block_size(32, 8);

  compress_offset_kernel<<<1, block_size, 0, stream>>>(
      offset.get<uint32_t>(), num_compressed_offset_, stride, compressed_offset_.get<uint32_t>());

  *compressed_offset = compressed_offset_;
}

AverageCominber::AverageCominber(std::shared_ptr<CoreResourceManager> core, int num_gpus,
                                 int num_local_embedding, const std::vector<int> &ev_size_list,
                                 int universal_batch_size)
    : core_(core), num_gpus_(num_gpus), num_local_embedding_(num_local_embedding) {
  HugeCTR::CudaDeviceContext ctx(core_->get_device_id());

  int num_ev_size_sum = std::accumulate(ev_size_list.begin(), ev_size_list.end(), 0);
  auto buffer_ptr = GetBuffer(core);
  // TODO: The float emb vec can be reduced to the scale of num of local embedding
  float_emb_vec_ = buffer_ptr->reserve({universal_batch_size / num_gpus, num_ev_size_sum},
                                       DeviceType::GPU, TensorScalarType::Float32);
  buffer_ptr->allocate();
}

void AverageCominber::forward(const Tensor &bucket_range, Tensor &dst_emb_vec,
                              const Tensor &d_local_embedding_list, const Tensor &d_combiner_list,
                              const Tensor &d_ev_size_offset, int batch_size, int max_ev_size) {
  HugeCTR::CudaDeviceContext ctx(core_->get_device_id());
  int gpu_id = core_->get_global_gpu_id();
  auto stream = core_->get_local_gpu()->get_stream();
  int batch_size_per_gpu = batch_size / num_gpus_;

  DISPATCH_INTEGRAL_FUNCTION(bucket_range.dtype().type(), offset_t, [&] {
    DISPATCH_FLOAT_AND_HALF_FUNCTION(dst_emb_vec.dtype().type(), emb_t, [&] {
      auto get_idx = [local_embedding_ptr = d_local_embedding_list.get<int>(),
                      batch_size_per_gpu] __device__(int index) {
        int batch_id = index % batch_size_per_gpu;
        int embedding_id = local_embedding_ptr[index / batch_size_per_gpu];
        return embedding_id * batch_size_per_gpu + batch_id;
      };
      LambdaIterator<int, int32_t, decltype(get_idx)> src_idx_iter(
          get_idx, batch_size_per_gpu * num_local_embedding_);

      auto scaler_array = [local_embedding_ptr = d_local_embedding_list.get<int>(),
                           combiner_ptr = d_combiner_list.get<char>(),
                           bucket_range_ptr = bucket_range.get<offset_t>(), batch_size_per_gpu,
                           batch_size, gpu_id] __device__(int index) {
        int batch_id = index % batch_size_per_gpu;
        int embedding_id = local_embedding_ptr[index / batch_size_per_gpu];
        if (combiner_ptr[embedding_id] == static_cast<char>(Combiner::Average)) {
          int start = batch_size * embedding_id + gpu_id * batch_size_per_gpu + batch_id;
          return static_cast<int>(bucket_range_ptr[start + 1] - bucket_range_ptr[start]);
        } else {
          return 1;
        }
      };
      LambdaIterator<int, int32_t, decltype(scaler_array)> scaler_arr_iter(
          scaler_array, batch_size_per_gpu * num_local_embedding_);

      LambdaIterator<int, int32_t, decltype(get_idx)> dst_idx_iter(
          get_idx, batch_size_per_gpu * num_local_embedding_);

      RaggedEmbForwardResultView<float, RestrictPtrTraits, int32_t> src_buffer_iter{
          float_emb_vec_.get(), d_ev_size_offset.get<int>(), batch_size_per_gpu};

      RaggedEmbForwardResultView<emb_t, RestrictPtrTraits, int32_t> dst_buffer_iter{
          dst_emb_vec.get(), d_ev_size_offset.get<int>(), batch_size_per_gpu};

      generic_copy(src_idx_iter, scaler_arr_iter, dst_idx_iter, src_buffer_iter, dst_buffer_iter,
                   max_ev_size, stream);
    });
  });
}

void AverageCominber::backward(const Tensor &bucket_range, const Tensor &src_emb_vec,
                               const Tensor &d_local_embedding_list, const Tensor &d_combiner_list,
                               const Tensor &d_ev_size_offset, int batch_size, int max_ev_size) {
  HugeCTR::CudaDeviceContext ctx(core_->get_device_id());
  int gpu_id = core_->get_global_gpu_id();
  auto stream = core_->get_local_gpu()->get_stream();
  int batch_size_per_gpu = batch_size / num_gpus_;

  DISPATCH_INTEGRAL_FUNCTION(bucket_range.dtype().type(), offset_t, [&] {
    DISPATCH_FLOAT_AND_HALF_FUNCTION(src_emb_vec.dtype().type(), emb_t, [&] {
      auto get_idx = [local_embedding_ptr = d_local_embedding_list.get<int>(),
                      batch_size_per_gpu] __device__(int index) {
        int batch_id = index % batch_size_per_gpu;
        int embedding_id = local_embedding_ptr[index / batch_size_per_gpu];
        return embedding_id * batch_size_per_gpu + batch_id;
      };
      LambdaIterator<int, int32_t, decltype(get_idx)> src_idx_iter(
          get_idx, batch_size_per_gpu * num_local_embedding_);

      auto scaler_array = [local_embedding_ptr = d_local_embedding_list.get<int>(),
                           combiner_ptr = d_combiner_list.get<char>(),
                           bucket_range_ptr = bucket_range.get<offset_t>(), batch_size_per_gpu,
                           batch_size, gpu_id] __device__(int index) {
        int batch_id = index % batch_size_per_gpu;
        int embedding_id = local_embedding_ptr[index / batch_size_per_gpu];
        if (combiner_ptr[embedding_id] == static_cast<char>(Combiner::Average)) {
          int start = batch_size * embedding_id + gpu_id * batch_size_per_gpu + batch_id;
          return static_cast<int>(bucket_range_ptr[start + 1] - bucket_range_ptr[start]);
        } else {
          return 1;
        }
      };
      LambdaIterator<int, int32_t, decltype(scaler_array)> scaler_arr_iter(
          scaler_array, batch_size_per_gpu * num_local_embedding_);

      LambdaIterator<int, int32_t, decltype(get_idx)> dst_idx_iter(
          get_idx, batch_size_per_gpu * num_local_embedding_);

      RaggedEmbForwardResultView<emb_t, RestrictPtrTraits, int32_t> src_buffer_iter{
          src_emb_vec.get(), d_ev_size_offset.get<int>(), batch_size_per_gpu};

      RaggedEmbForwardResultView<float, RestrictPtrTraits, int32_t> dst_buffer_iter{
          float_emb_vec_.get(), d_ev_size_offset.get<int>(), batch_size_per_gpu};

      generic_copy(src_idx_iter, scaler_arr_iter, dst_idx_iter, src_buffer_iter, dst_buffer_iter,
                   max_ev_size, stream);
    });
  });
}

}  // namespace embedding