#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <hipcub/hipcub.hpp>

#include "HugeCTR/include/utils.cuh"
#include "HugeCTR/include/utils.hpp"
#include "dp_index_calculation.hpp"
#include "generic_lookup.cuh"
namespace embedding {

namespace {

template <typename offset_t>
__global__ void mask_flag_kernel(int num_local_embedding, int batch_size_per_gpu,
                                 int const* d_local_embedding_list, int batch_size, int gpu_id,
                                 offset_t const* bucket_range, char* flag, uint32_t* dp_offset,
                                 uint32_t* dp_dst) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid < num_local_embedding * batch_size_per_gpu) {
    int batch_id = tid % batch_size_per_gpu;
    int embedding_id = d_local_embedding_list[tid / batch_size_per_gpu];
    int bucket_id = batch_size * embedding_id + batch_size_per_gpu * gpu_id + batch_id;

    int start = bucket_range[bucket_id];
    int end = bucket_range[bucket_id + 1];
    dp_offset[1 + tid] = end - start;
    for (int j = start; j < end; ++j) {
      flag[j] = 1;
    }

    int dst_bucket_id = batch_size_per_gpu * embedding_id + batch_id;
    dp_dst[tid] = dst_bucket_id;
  }
}

template <typename key_t, typename offset_t>
__global__ void fused_select_dp_key_and_bucket_id_kernel(
    const key_t* keys, const offset_t* bucket_range, size_t num_key, int batch_size,
    int num_local_embedding, int num_embedding, int gpu_id, int num_gpu, const int* id_space_list,
    const int* local_embedding_list, key_t* dp_keys, uint32_t* dp_bucket_id,
    const int* segment_start_offsets, int* segment_end_offsets) {
  int local_embedding_id = blockIdx.x;
  int embedding_id = local_embedding_list[local_embedding_id];
  // int id_space = id_space_list[local_embedding_id];
  int batch_size_per_gpu = batch_size / num_gpu;

  uint32_t segment_start = segment_start_offsets[local_embedding_id];
  offset_t bucket_start = bucket_range[batch_size * embedding_id];
  for (int batch_id = threadIdx.x; batch_id < batch_size; batch_id += blockDim.x) {
    uint32_t bucket_id = batch_size * embedding_id + batch_id;

    uint32_t start = bucket_range[bucket_id];
    uint32_t end = bucket_range[bucket_id + 1];

    uint32_t local_bucket_id;
    if (batch_id >= gpu_id * batch_size_per_gpu && batch_id < (gpu_id + 1) * batch_size_per_gpu) {
      local_bucket_id = batch_size_per_gpu * embedding_id + batch_id % batch_size_per_gpu;
    } else {
      local_bucket_id = batch_size * num_embedding;
    }

    for (uint32_t r = start; r < end; ++r) {
      dp_keys[segment_start + (r - bucket_start)] = keys[r];
      dp_bucket_id[segment_start + (r - bucket_start)] = local_bucket_id;
    }
  }
  if (threadIdx.x == 0) {
    offset_t bucket_end = bucket_range[batch_size * embedding_id + batch_size];
    int num_key_in_bucket = static_cast<int>(bucket_end) - static_cast<int>(bucket_start);
    segment_end_offsets[local_embedding_id] = num_key_in_bucket + static_cast<int>(segment_start);
  }
}

template <typename key_t>
class SelectUniqueDPKeyOp {
  const key_t* sorted_dp_keys_;
  const int* segment_start_offsets_;
  const int* segment_end_offsets_;
  int num_embedding_;

 public:
  __host__ __device__ __forceinline__ SelectUniqueDPKeyOp(const key_t* sorted_dp_keys,
                                                          const int* segment_start_offsets,
                                                          const int* segment_end_offsets,
                                                          int num_embedding)
      : sorted_dp_keys_(sorted_dp_keys),
        segment_start_offsets_(segment_start_offsets),
        segment_end_offsets_(segment_end_offsets),
        num_embedding_(num_embedding) {}

  __device__ __forceinline__ bool operator()(const uint32_t& idx) const {
    int embedding_id = binary_search_index_lower_bound(segment_start_offsets_, num_embedding_ + 1,
                                                       static_cast<int>(idx));
    if (idx >= segment_end_offsets_[embedding_id]) return false;
    if (idx == segment_start_offsets_[embedding_id]) return true;
    return sorted_dp_keys_[idx] != sorted_dp_keys_[idx - 1];
  }
};

class SelectLocalBucketidOP {
  int empty_bucket_id_;

 public:
  __host__ __device__ __forceinline__ SelectLocalBucketidOP(int empty_bucket_id)
      : empty_bucket_id_(empty_bucket_id) {}

  __device__ __forceinline__ bool operator()(const uint32_t& bucket_id) const {
    return bucket_id < empty_bucket_id_;
  }
};

template <typename key_t>
__global__ void fused_compact_unique_key_and_count_bucket_id_offset(
    const key_t* key, const uint32_t* indices, const size_t* num_indices,
    const uint32_t* sorted_dp_bucket_id_list, const int* segment_start_offsets,
    const int* segment_end_offsets, const int* local_ev_size_list, int batch_size,
    int num_local_embedding, int num_embedding, key_t* compact_key, uint32_t* dst_idx,
    uint32_t* bucket_offset, uint32_t* unique_id_space_offset) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < *num_indices) {
    int idx = indices[tid];
    compact_key[tid] = key[idx];
    int embedding_id =
        binary_search_index_lower_bound(segment_start_offsets, num_local_embedding + 1, idx);
    if (segment_start_offsets[embedding_id] == idx) {
      unique_id_space_offset[embedding_id] = tid;
    }

    int ev_size = local_ev_size_list[embedding_id];
    dst_idx[tid + 1] = ev_size;
    int next_idx =
        (tid == *num_indices - 1) ? segment_start_offsets[num_local_embedding] : indices[tid + 1];
    int num_bucket_id = 0;
    for (int i = idx; i < next_idx; ++i) {
      if (sorted_dp_bucket_id_list[i] < batch_size * num_embedding) num_bucket_id += 1;
    }
    bucket_offset[tid + 1] = num_bucket_id;
  }

  if (tid == 0) {
    dst_idx[0] = 0;
    bucket_offset[0] = 0;
    unique_id_space_offset[num_local_embedding] = static_cast<uint32_t>(*num_indices);
  }
}

__global__ void memset_kernel(uint32_t* arr, int num, uint32_t val) {
  for (int tid = threadIdx.x + blockIdx.x * blockDim.x; tid < num; tid += blockDim.x * gridDim.x) {
    arr[tid] = val;
  }
}
}  // namespace

DPIndexCalculation::DPIndexCalculation(std::shared_ptr<CoreResourceManager> core, int num_gpus,
                                       int num_local_embedding, int local_hotness_sum,
                                       int hotness_sum, int universal_batch_size, DataType key_type,
                                       DataType offset_type)
    : core_(core),
      num_gpus_(num_gpus),
      num_local_embedding_(num_local_embedding),
      universal_batch_size_(universal_batch_size),
      universal_batch_size_per_gpu_(universal_batch_size / num_gpus),
      local_hotness_sum_(local_hotness_sum),
      hotness_sum_(hotness_sum),
      key_type_(key_type),
      offset_type_(offset_type) {
  HugeCTR::CudaDeviceContext ctx(core->get_device_id());
  Device device{DeviceType::GPU, core->get_device_id()};

  core::BufferPtr buffer_ptr = GetBuffer(core_);

  // reserve and allocate tensors for index calculation on GPU
  num_dp_key_ = buffer_ptr->reserve({1}, DeviceType::CPU, TensorScalarType::Size_t);
  flag_ =
      buffer_ptr->reserve({universal_batch_size_ * hotness_sum_}, device, TensorScalarType::Char);

  size_t temp_storage_bytes_category = 0;
  DISPATCH_INTEGRAL_FUNCTION(key_type_.type(), key_t, ([&] {
                               hipcub::DeviceSelect::Flagged(nullptr, temp_storage_bytes_category,
                                                          (key_t*)nullptr, (char*)nullptr,
                                                          (key_t*)nullptr, (size_t*)nullptr,
                                                          universal_batch_size_ * hotness_sum_);
                             }));
  d_temp_storage_category_ =
      buffer_ptr->reserve({temp_storage_bytes_category}, device, TensorScalarType::Void);

  size_t temp_storage_bytes_offset = 0;
  hipcub::DeviceScan::InclusiveSum(nullptr, temp_storage_bytes_offset, (uint32_t*)nullptr,
                                (uint32_t*)nullptr,
                                universal_batch_size_per_gpu_ * num_local_embedding_ + 1);
  d_temp_storage_offset_ =
      buffer_ptr->reserve({temp_storage_bytes_offset}, device, TensorScalarType::Void);

  // allocate output memory
  dp_key_ =
      buffer_ptr->reserve({universal_batch_size_per_gpu_ * local_hotness_sum_}, device, key_type_);
  dp_offset_ = buffer_ptr->reserve({universal_batch_size_per_gpu_ * num_local_embedding_ + 1},
                                   device, TensorScalarType::UInt32);
  dp_dst_ = buffer_ptr->reserve({universal_batch_size_per_gpu_ * num_local_embedding_}, device,
                                TensorScalarType::UInt32);
  buffer_ptr->allocate();
}

void DPIndexCalculation::compute(const Tensor& key, const Tensor& bucket_range, size_t num_keys,
                                 const Tensor& d_local_embedding_list, int batch_size,
                                 Tensor* dp_key, Tensor* dp_offset, size_t* num_dp_key,
                                 Tensor* dp_dst) {
  HugeCTR::CudaDeviceContext ctx(core_->get_device_id());

  int batch_size_per_gpu = batch_size / num_gpus_;

  int gpu_id = core_->get_global_gpu_id();
  auto stream = core_->get_local_gpu()->get_stream();

  DISPATCH_INTEGRAL_FUNCTION(key.dtype().type(), key_t, [&] {
    DISPATCH_INTEGRAL_FUNCTION(bucket_range.dtype().type(), offset_t, [&] {
      HCTR_LIB_THROW(hipMemsetAsync(dp_key_.get<key_t>(), 0, dp_key_.nbytes(), stream));
      HCTR_LIB_THROW(hipMemsetAsync(dp_offset_.get<uint32_t>(), 0, dp_offset_.nbytes(), stream));
      HCTR_LIB_THROW(hipMemsetAsync(dp_dst_.get<uint32_t>(), 0, dp_dst_.nbytes(), stream));
      HCTR_LIB_THROW(hipMemsetAsync(flag_.get<char>(), 0, flag_.nbytes(), stream));
      HCTR_LIB_THROW(hipMemsetAsync(num_dp_key_.get<size_t>(), 0, num_dp_key_.nbytes(), stream));

      // mask_flag
      constexpr int blockDim = 1024;
      int gridDim = (num_local_embedding_ * batch_size_per_gpu - 1) / blockDim + 1;
      mask_flag_kernel<<<gridDim, blockDim, 0, stream>>>(
          num_local_embedding_, batch_size_per_gpu, d_local_embedding_list.get<int>(), batch_size,
          gpu_id, bucket_range.get<offset_t>(), flag_.get<char>(), dp_offset_.get<uint32_t>(),
          dp_dst_.get<uint32_t>());

      // select key
      size_t temp_storage_category_bytes = d_temp_storage_category_.nbytes();
      hipcub::DeviceSelect::Flagged(d_temp_storage_category_.get(), temp_storage_category_bytes,
                                 key.get<key_t>(), flag_.get<char>(), dp_key_.get<key_t>(),
                                 num_dp_key_.get<size_t>(), key.get_num_elements(), stream);
      HCTR_LIB_THROW(hipPeekAtLastError());
      // inclusive sum for offset
      size_t temp_storage_offset_bytes = d_temp_storage_offset_.nbytes();
      hipcub::DeviceScan::InclusiveSum(d_temp_storage_offset_.get(), temp_storage_offset_bytes,
                                    dp_offset_.get<uint32_t>(), dp_offset_.get<uint32_t>(),
                                    dp_offset_.get_num_elements(), stream);
      HCTR_LIB_THROW(hipPeekAtLastError());
      // sync with cpu to get sum flag
      HCTR_LIB_THROW(hipStreamSynchronize(stream));
    });
  });

  *dp_key = dp_key_;
  *dp_offset = dp_offset_;
  *num_dp_key = num_dp_key_.get<size_t>()[0];
  *dp_dst = dp_dst_;
}

DPLocalReduceIndexCalculation::DPLocalReduceIndexCalculation(
    std::shared_ptr<CoreResourceManager> core, int num_embedding, int num_local_embedding,
    const std::vector<int>& h_local_hotness_list, int universal_batch_size, DataType key_type)
    : core_(core), num_embedding_(num_embedding), num_local_embedding_(num_local_embedding) {
  HugeCTR::CudaDeviceContext ctx(core_->get_device_id());
  Device device{DeviceType::GPU, core_->get_device_id()};

  int local_hotness_sum =
      std::accumulate(h_local_hotness_list.begin(), h_local_hotness_list.end(), 0);

  auto buffer_ptr = GetBuffer(core);

  segment_start_offsets_ =
      buffer_ptr->reserve(num_local_embedding + 1, device, TensorScalarType::Int32);
  segment_end_offsets_ = buffer_ptr->reserve(num_local_embedding, device, TensorScalarType::Int32);
  dp_keys_ = buffer_ptr->reserve({universal_batch_size, local_hotness_sum}, device, key_type);
  dp_bucket_id_ = buffer_ptr->reserve({universal_batch_size, local_hotness_sum}, device,
                                      TensorScalarType::UInt32);
  sorted_dp_keys_ =
      buffer_ptr->reserve({universal_batch_size, local_hotness_sum}, device, key_type);
  unique_dp_keys_indices_ = buffer_ptr->reserve({universal_batch_size, local_hotness_sum}, device,
                                                TensorScalarType::UInt32);
  sorted_dp_bucket_id_ = buffer_ptr->reserve({universal_batch_size, local_hotness_sum}, device,
                                             TensorScalarType::UInt32);

  unique_dp_keys_ =
      buffer_ptr->reserve({universal_batch_size, local_hotness_sum}, device, key_type);
  num_unique_key_ = buffer_ptr->reserve(1, DeviceType::CPU, TensorScalarType::Size_t);

  sorted_bucket_id_list_ = buffer_ptr->reserve({universal_batch_size, local_hotness_sum}, device,
                                               TensorScalarType::UInt32);
  num_sorted_bucket_id_ = buffer_ptr->reserve(1, DeviceType::CPU, TensorScalarType::Size_t);
  unique_dst_idx_ = buffer_ptr->reserve(1 + universal_batch_size * local_hotness_sum, device,
                                        TensorScalarType::UInt32);
  sorted_bucket_id_offset_ = buffer_ptr->reserve(1 + universal_batch_size * local_hotness_sum,
                                                 device, TensorScalarType::UInt32);
  unique_id_space_offset_ =
      buffer_ptr->reserve(1 + num_local_embedding_, device, TensorScalarType::UInt32);

  {
    size_t temp_bytes = 0;
    DISPATCH_INTEGRAL_FUNCTION(key_type.type(), key_t, [&] {
      hipcub::DeviceSegmentedRadixSort::SortPairs(
          nullptr, temp_bytes, (key_t*)nullptr, (key_t*)nullptr, (uint32_t*)nullptr,
          (uint32_t*)nullptr, universal_batch_size * local_hotness_sum, num_local_embedding_,
          (int*)nullptr, (int*)nullptr);
    });
    d_temp_segmented_sort_storage_ =
        buffer_ptr->reserve({temp_bytes}, device, TensorScalarType::Void);
  }
  {
    size_t temp_bytes = 0;
    DISPATCH_INTEGRAL_FUNCTION(key_type.type(), key_t, [&] {
      hipcub::CountingInputIterator<uint32_t> counting(0);
      SelectUniqueDPKeyOp<key_t> select_unique_dp_key_op{nullptr, nullptr, nullptr,
                                                         num_local_embedding_};
      hipcub::DeviceSelect::If(nullptr, temp_bytes, counting, (uint32_t*)nullptr, (size_t*)nullptr,
                            universal_batch_size * local_hotness_sum, select_unique_dp_key_op);
    });
    d_temp_if_storage_ = buffer_ptr->reserve({temp_bytes}, device, TensorScalarType::Void);
  }
  {
    size_t temp_bytes = 0;
    DISPATCH_INTEGRAL_FUNCTION(key_type.type(), key_t, [&] {
      SelectLocalBucketidOP select_unique_dp_key_op{num_local_embedding_ * universal_batch_size};
      hipcub::DeviceSelect::If(nullptr, temp_bytes, (uint32_t*)nullptr, (uint32_t*)nullptr,
                            (size_t*)nullptr, universal_batch_size * local_hotness_sum,
                            select_unique_dp_key_op);
    });
    d_temp_select_bucket_id_storage_ =
        buffer_ptr->reserve({temp_bytes}, device, TensorScalarType::Void);
  }
  {
    size_t temp_bytes = 0;
    hipcub::DeviceScan::InclusiveSum(nullptr, temp_bytes, (uint32_t*)nullptr, (uint32_t*)nullptr,
                                  universal_batch_size * local_hotness_sum + 1);
    d_scan_storage_ = buffer_ptr->reserve({temp_bytes}, device, TensorScalarType::Void);
  }
  buffer_ptr->allocate();
  std::vector<int> cpu_segments_start_offset{0};
  for (int embedding_id = 0; embedding_id < num_local_embedding_; ++embedding_id) {
    cpu_segments_start_offset.push_back(h_local_hotness_list[embedding_id] * universal_batch_size);
  }
  std::partial_sum(cpu_segments_start_offset.begin(), cpu_segments_start_offset.end(),
                   cpu_segments_start_offset.begin());

  segment_start_offsets_.copy_from(cpu_segments_start_offset);
}

void DPLocalReduceIndexCalculation::compute(
    const Tensor& key, size_t num_key, const Tensor& bucket_range,
    const Tensor& d_local_embedding_list, const Tensor& id_space_list,
    const Tensor& d_local_ev_size_list, int batch_size, Tensor* unique_key, size_t* num_unique_key,
    Tensor* unique_dst_idx, Tensor* sorted_bucket_id_list, Tensor* sorted_bucket_id_offset,
    Tensor* unique_id_space_offset) {
  HugeCTR::CudaDeviceContext ctx(core_->get_device_id());

  DISPATCH_INTEGRAL_FUNCTION(key.dtype().type(), key_t, [&] {
    DISPATCH_INTEGRAL_FUNCTION(bucket_range.dtype().type(), offset_t, [&] {
      auto stream = core_->get_local_gpu()->get_stream();
      int gpu_id = core_->get_global_gpu_id();
      int num_gpus = core_->get_global_gpu_count();

      HCTR_LIB_THROW(hipMemsetAsync(segment_end_offsets_.get<int>(), 0,
                                     segment_end_offsets_.nbytes(), stream));
      HCTR_LIB_THROW(hipMemsetAsync(dp_keys_.get<key_t>(), 0, dp_keys_.nbytes(), stream));
      HCTR_LIB_THROW(
          hipMemsetAsync(dp_bucket_id_.get<uint32_t>(), 0, dp_bucket_id_.nbytes(), stream));
      HCTR_LIB_THROW(
          hipMemsetAsync(sorted_dp_keys_.get<key_t>(), 0, sorted_dp_keys_.nbytes(), stream));
      HCTR_LIB_THROW(
          hipMemsetAsync(unique_dp_keys_.get<key_t>(), 0, unique_dp_keys_.nbytes(), stream));
      HCTR_LIB_THROW(hipMemsetAsync(unique_dp_keys_indices_.get<uint32_t>(), 0,
                                     unique_dp_keys_indices_.nbytes(), stream));

      HCTR_LIB_THROW(hipMemsetAsync(sorted_bucket_id_list_.get<uint32_t>(), 0,
                                     sorted_bucket_id_list_.nbytes(), stream));
      HCTR_LIB_THROW(
          hipMemsetAsync(unique_dst_idx_.get<uint32_t>(), 0, unique_dst_idx_.nbytes(), stream));
      HCTR_LIB_THROW(hipMemsetAsync(sorted_bucket_id_offset_.get<uint32_t>(), 0,
                                     sorted_bucket_id_offset_.nbytes(), stream));

      {
        fused_select_dp_key_and_bucket_id_kernel<<<num_local_embedding_, 256, 0, stream>>>(
            key.get<key_t>(), bucket_range.get<offset_t>(), num_key, batch_size,
            num_local_embedding_, num_embedding_, gpu_id, num_gpus, id_space_list.get<int>(),
            d_local_embedding_list.get<int>(), dp_keys_.get<key_t>(), dp_bucket_id_.get<uint32_t>(),
            segment_start_offsets_.get<int>(), segment_end_offsets_.get<int>());
      }
      {
        memset_kernel<<<128, 1024, 0, stream>>>(sorted_dp_bucket_id_.get<uint32_t>(),
                                                sorted_dp_bucket_id_.get_num_elements(),
                                                batch_size * num_embedding_);
        size_t nbytes = d_temp_segmented_sort_storage_.nbytes();
        hipcub::DeviceSegmentedRadixSort::SortPairs(
            d_temp_segmented_sort_storage_.get(), nbytes, dp_keys_.get<key_t>(),
            sorted_dp_keys_.get<key_t>(), dp_bucket_id_.get<uint32_t>(),
            sorted_dp_bucket_id_.get<uint32_t>(), dp_keys_.get_num_elements(), num_local_embedding_,
            segment_start_offsets_.get<int>(), segment_end_offsets_.get<int>(), 0,
            sizeof(key_t) * 8, stream);
      }
      {
        hipcub::CountingInputIterator<uint32_t> counting(0);
        SelectUniqueDPKeyOp<key_t> select_unique_dp_key_op{
            sorted_dp_keys_.get<key_t>(), segment_start_offsets_.get<int>(),
            segment_end_offsets_.get<int>(), num_local_embedding_};
        size_t nbytes = d_temp_if_storage_.nbytes();
        hipcub::DeviceSelect::If(d_temp_if_storage_.get(), nbytes, counting,
                              unique_dp_keys_indices_.get<uint32_t>(),
                              num_unique_key_.get<size_t>(), sorted_dp_keys_.get_num_elements(),
                              select_unique_dp_key_op, stream);
      }
      {
        hipcub::CountingInputIterator<uint32_t> counting(0);
        SelectLocalBucketidOP select_unique_dp_key_op{num_embedding_ * batch_size};
        size_t nbytes = d_temp_if_storage_.nbytes();
        hipcub::DeviceSelect::If(
            d_temp_if_storage_.get(), nbytes, sorted_dp_bucket_id_.get<uint32_t>(),
            sorted_bucket_id_list_.get<uint32_t>(), num_sorted_bucket_id_.get<size_t>(),
            sorted_dp_bucket_id_.get_num_elements(), select_unique_dp_key_op, stream);

        fused_compact_unique_key_and_count_bucket_id_offset<<<(num_key - 1) / 256 + 1, 256, 0,
                                                              stream>>>(
            sorted_dp_keys_.get<key_t>(), unique_dp_keys_indices_.get<uint32_t>(),
            num_unique_key_.get<size_t>(), sorted_dp_bucket_id_.get<uint32_t>(),
            segment_start_offsets_.get<int>(), segment_end_offsets_.get<int>(),
            d_local_ev_size_list.get<int>(), batch_size, num_local_embedding_, num_embedding_,
            unique_dp_keys_.get<key_t>(), unique_dst_idx_.get<uint32_t>(),
            sorted_bucket_id_offset_.get<uint32_t>(), unique_id_space_offset_.get<uint32_t>());
      }
      {
        size_t nbytes = d_scan_storage_.nbytes();
        hipcub::DeviceScan::InclusiveSum(
            d_scan_storage_.get(), nbytes, unique_dst_idx_.get<uint32_t>(),
            unique_dst_idx_.get<uint32_t>(), unique_dst_idx_.get_num_elements(), stream);
        hipcub::DeviceScan::InclusiveSum(d_scan_storage_.get(), nbytes,
                                      sorted_bucket_id_offset_.get<uint32_t>(),
                                      sorted_bucket_id_offset_.get<uint32_t>(),
                                      sorted_bucket_id_offset_.get_num_elements(), stream);
      }
      HCTR_LIB_THROW(hipStreamSynchronize(stream));
    });
  });
  *unique_key = unique_dp_keys_;
  *num_unique_key = *num_unique_key_.get<size_t>();
  *unique_dst_idx = unique_dst_idx_;
  *sorted_bucket_id_list = sorted_bucket_id_list_;
  *sorted_bucket_id_offset = sorted_bucket_id_offset_;
  *unique_id_space_offset = unique_id_space_offset_;
}
}  // namespace embedding